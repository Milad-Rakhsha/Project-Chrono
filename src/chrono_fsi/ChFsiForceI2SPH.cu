#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <sstream>

#include "chrono_fsi/ChFsiForceI2SPH.cuh"

//==========================================================================================================================================
namespace chrono {
namespace fsi {
// extern __constant__ SimParams paramsD;
// extern __constant__ NumberOfObjects numObjectsD;

struct compare_Real3_mag {
    __host__ __device__ bool operator()(Real3 lhs, Real3 rhs) { return length(lhs) < length(rhs); }
};

// double precision atomic add function
__device__ inline double datomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;

    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void V_star_Predictor(Real4* sortedPosRad,  // input: sorted positions
                                 Real3* sortedVelMas,
                                 Real4* sortedRhoPreMu,
                                 Real* A_Matrix,
                                 Real3* b,
                                 Real* vx,
                                 Real* vy,
                                 Real* vz,
                                 Real3* v_old,

                                 const Real* A_L,
                                 const Real3* A_G,
                                 const Real* A_f,

                                 const Real* sumWij_inv,
                                 Real3* normals,

                                 const uint* csrColInd,
                                 const uint* numContacts,

                                 Real4* velMassRigid_fsiBodies_D,
                                 Real3* accRigid_fsiBodies_D,
                                 uint* rigidIdentifierD,

                                 Real3* pos_fsi_fea_D,
                                 Real3* vel_fsi_fea_D,
                                 Real3* acc_fsi_fea_D,
                                 uint* FlexIdentifierD,
                                 int numFlex1D,
                                 uint2* CableElementsNodes,
                                 uint4* ShellelementsNodes,

                                 int4 updatePortion,
                                 uint* gridMarkerIndexD,

                                 int numAllMarkers,
                                 volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1;
        b[i_idx] = mR3(0.0);
        vx[i_idx] = b[i_idx].x;
        vy[i_idx] = b[i_idx].y;
        vz[i_idx] = b[i_idx].z;
        return;
    }

    Real rho0 = paramsD.rho0;
    Real nu0 = paramsD.mu0 / rho0;
    Real dt = paramsD.dT;

    if (Fluid_Marker) {
        Real3 rhs = mR3(0.0);
        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            int j = csrColInd[count];
            A_Matrix[count] = -nu0 / 2.0 * A_L[count];  // viscouse term

            rhs +=  //
                nu0 / 2.0 * A_L[count] *
                sortedVelMas[j];  // viscous term
                                  //                - 1.0 / sortedRhoPreMu[i_idx].x * A_G[count] *
                                  //                      sortedRhoPreMu[j].y  // pressure gradient
        }
        A_Matrix[csrStartIdx] += 1 / dt;
        b[i_idx] = rhs + sortedVelMas[i_idx] / dt  //forward euler term from lhs
                   + paramsD.gravity;              // body force

    } else if (Boundary_Marker) {
        Real h_i = sortedPosRad[i_idx].w;
        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real den = 0.0;
        for (uint count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w != -1)
                continue;
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real h_j = sortedPosRad[j].w;
            Real h_ij = 0.5 * (h_j + h_i);
            Real W3 = W3h(length(rij), h_ij);
            A_Matrix[count] = W3;
            // A_Matrix[count] = A_f[count];
            den = den + W3;
        }

        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);
        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                    accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D,
                    FlexIdentifierD, numFlex1D, CableElementsNodes, ShellelementsNodes);

        if (abs(den) < EPSILON) {
            A_Matrix[csrStartIdx] = 1.0;
            b[i_idx] = V_prescribed;
        } else {
            A_Matrix[csrStartIdx] = den;
            b[i_idx] = 2 * V_prescribed * den;
        }
    }

    v_old[i_idx] = sortedVelMas[i_idx];
    vx[i_idx] = b[i_idx].x;
    vy[i_idx] = b[i_idx].y;
    vz[i_idx] = b[i_idx].z;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Pressure_Equation(Real4* sortedPosRad,  // input: sorted positions
                                  Real3* sortedVelMas,
                                  Real4* sortedRhoPreMu,
                                  Real* A_Matrix,
                                  Real* Bi,
                                  Real3* Vstar,
                                  Real* q_old,

                                  const Real* A_f,
                                  const Real* A_L,
                                  const Real3* A_G,
                                  const Real* sumWij_inv,
                                  Real3* Normals,
                                  const uint* csrColInd,
                                  const uint* numContacts,

                                  Real4* velMassRigid_fsiBodies_D,
                                  Real3* accRigid_fsiBodies_D,
                                  uint* rigidIdentifierD,

                                  Real3* pos_fsi_fea_D,
                                  Real3* vel_fsi_fea_D,
                                  Real3* acc_fsi_fea_D,
                                  uint* FlexIdentifierD,
                                  int numFlex1D,
                                  uint2* CableElementsNodes,
                                  uint4* ShellelementsNodes,

                                  int4 updatePortion,
                                  uint* gridMarkerIndexD,
                                  int numAllMarkers,
                                  int FixedMarker,
                                  volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1.0;
        Bi[i_idx] = 0.0;
        return;
    }

    Real3 gravity = paramsD.gravity;
    Real dt = paramsD.dT;
    Real3 grad_rho_i = mR3(0.0);
    Real div_vi_star = 0;

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        div_vi_star += dot(A_G[count], Vstar[j]);
        grad_rho_i += A_G[count] * sortedRhoPreMu[j].x;
    }
    Real rhoi = sortedRhoPreMu[i_idx].x;
    Real rhoi_star = rhoi - rhoi * div_vi_star * dt;

    //======================== Interior ===========================
    if (Fluid_Marker) {
        //        if (sortedRhoPreMu[i_idx].x < 0.80 * paramsD.rho0 && sortedRhoPreMu[i_idx].x > 0.80 * paramsD.rho0) {
        //            //        if (i_idx == FixedMarker) {
        //            A_Matrix[csrStartIdx] = 1.0;
        //            //
        //            for (int count = csrStartIdx; count < csrEndIdx; count++) {
        //                A_Matrix[count] = -A_f[count];
        //            }
        //        } else if (sortedRhoPreMu[i_idx].x < 0.99 * paramsD.rho0) {
        //            A_Matrix[csrStartIdx] = 1.0;
        //            Bi[i_idx] = 0.0;
        if (i_idx == -1) {
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = 0.0;
        } else {
            for (int count = csrStartIdx; count < csrEndIdx; count++) {
                A_Matrix[count] = 1.0 / rhoi * A_L[count] - 1.0 / (rhoi * rhoi) * dot(grad_rho_i, A_G[count]);
            }
            Bi[i_idx] = 1.0 * div_vi_star / dt + 0.0 * (paramsD.rho0 - rhoi_star) / rhoi_star / (dt * dt);
        }

        //======================= Boundary ===========================
    } else if (Boundary_Marker && paramsD.bceType != ADAMI) {
        Real3 my_normal = Normals[i_idx];
        bool haveFluid = false;
        Real temp = 0;

        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            if (sortedRhoPreMu[csrColInd[count]].w == -1.0)
                haveFluid = true;
        }

        if (!haveFluid) {
            for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
                A_Matrix[count] = 0.0;
            }
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = 1000.0;
        } else {
            for (int count = csrStartIdx; count < csrEndIdx; count++) {
                uint j = csrColInd[count];
                if (sortedRhoPreMu[j].w == -1.0)
                    A_Matrix[count] = dot(A_G[count], my_normal);
            }
            if (A_Matrix[csrStartIdx] < 1e-3) {
                A_Matrix[csrStartIdx] = 1;  // temp / (csrEndIdx - csrStartIdx) / 5;
                Bi[i_idx] = 1e3;            // * dot(paramsD.gravity, my_normal);
            }
        }

        //======================= Boundary Adami===========================
    } else if (Boundary_Marker && paramsD.bceType == ADAMI) {
        Real h_i = sortedPosRad[i_idx].w;
        Real Vi = sumWij_inv[i_idx];
        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);
        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                    accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D,
                    FlexIdentifierD, numFlex1D, CableElementsNodes, ShellelementsNodes);
        Real pRHS = 0.0;
        Real den = 0.0;

        for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w != -1.0)
                continue;
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real h_j = sortedPosRad[j].w;
            Real h_ij = 0.5 * (h_j + h_i);
            Real W3 = W3h(length(rij), h_ij);
            A_Matrix[count] = -W3;
            den += W3;
            pRHS += dot(gravity - myAcc, rij) * sortedRhoPreMu[j].x * W3;
        }

        if (abs(den) > EPSILON) {
            A_Matrix[csrStartIdx] = den;
            Bi[i_idx] = pRHS;
            for (int count = csrStartIdx; count < csrEndIdx; count++) {
                A_Matrix[count] /= den;
            }
            Bi[i_idx] /= den;
        } else {
            A_Matrix[csrStartIdx] = 1;
            Bi[i_idx] = 0;
        }
    }

    //    q_old[i_idx] = sortedRhoPreMu[i_idx].y;
}  // namespace fsi
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Velocity_Correction_and_update(Real4* sortedPosRad,
                                               Real4* sortedRhoPreMu,
                                               Real3* sortedVelMas,
                                               Real3* Vstar,
                                               Real* q_i,  // q=p^(n+1)-p^n
                                               const Real3* A_G,
                                               const uint* csrColInd,
                                               const uint* numContacts,
                                               int numAllMarkers,
                                               const Real MaxVel,
                                               volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];
    Real3 grad_q_i = mR3(0.0);
    Real3 grad_p_nPlus1 = mR3(0.0);
    Real3 inner_sum = mR3(0.0), shift_r = mR3(0.0);
    Real mi_bar = 0.0, r0 = 0.0;
    int Ni = 0;
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        grad_q_i += A_G[count] * q_i[j];
        grad_p_nPlus1 += A_G[count] * (sortedRhoPreMu[j].y + q_i[j]);
        Real3 rij = mR3(sortedPosRad[i_idx] - sortedPosRad[j]);
        Real d = length(rij);
        if (d == 0)
            continue;
        Real m_j = pow(sortedPosRad[j].w * paramsD.MULT_INITSPACE, 3) * paramsD.rho0;
        mi_bar += m_j;
        Ni++;
        r0 += d;
        inner_sum += m_j * rij / (d * d * d);
    }
    r0 /= Ni;
    shift_r = 0.5 * r0 * r0 * length(MaxVel) * paramsD.dT / mi_bar * inner_sum;

    //    if (sortedPosRad[i_idx].x < 0.99 * paramsD.rho0)
    //        grad_q_i = mR3(0.0);

    Real3 V_new = Vstar[i_idx] - paramsD.dT / sortedRhoPreMu[i_idx].x * grad_q_i;

    Real4 x_new = sortedPosRad[i_idx] + mR4(paramsD.dT / 2 * (V_new + sortedVelMas[i_idx]), 0.0);

    sortedVelMas[i_idx] = V_new;
    //    sortedRhoPreMu[i_idx].y = q_i[i_idx];
    //    printf(" %d qi %f\n", i_idx, q_i[i_idx]);

    //    sortedRhoPreMu[i_idx].y += q_i[i_idx] + dot(grad_p_nPlus1, mR3(x_new - sortedPosRad[i_idx]));

    sortedRhoPreMu[i_idx].y = q_i[i_idx];

    if (sortedRhoPreMu[i_idx].w == -1.0) {
        sortedPosRad[i_idx] = x_new;
    }

    Real3 grad_p = mR3(0.0);
    Real3 grad_rho = mR3(0.0);
    Real3 grad_ux = mR3(0.0);
    Real3 grad_uy = mR3(0.0);
    Real3 grad_uz = mR3(0.0);

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        grad_p += A_G[count] * sortedRhoPreMu[i_idx].y;
        grad_rho += A_G[count] * sortedRhoPreMu[i_idx].x;
        grad_ux += A_G[count] * sortedVelMas[i_idx].x;
        grad_uy += A_G[count] * sortedVelMas[i_idx].y;
        grad_uz += A_G[count] * sortedVelMas[i_idx].z;
    }

    if (true && abs(sortedRhoPreMu[i_idx].w - (-1.0)) < EPSILON) {
        sortedPosRad[i_idx] += mR4(shift_r, 0.0);
        sortedRhoPreMu[i_idx].y += dot(shift_r, grad_p);
        sortedRhoPreMu[i_idx].x += dot(shift_r, grad_rho);
        sortedVelMas[i_idx].x += dot(shift_r, grad_ux);
        sortedVelMas[i_idx].y += dot(shift_r, grad_uy);
        sortedVelMas[i_idx].z += dot(shift_r, grad_uz);
    }

    if (!(isfinite(sortedPosRad[i_idx].x) && isfinite(sortedPosRad[i_idx].y) && isfinite(sortedPosRad[i_idx].z))) {
        printf("Error! particle %d position is NAN: thrown from Velocity_Correction_and_update  %f,%f,%f,%f\n", i_idx,
               sortedPosRad[i_idx].x, sortedPosRad[i_idx].y, sortedPosRad[i_idx].z, sortedPosRad[i_idx].w);
    }
    if (!(isfinite(sortedRhoPreMu[i_idx].x) && isfinite(sortedRhoPreMu[i_idx].y) &&
          isfinite(sortedRhoPreMu[i_idx].z))) {
        printf("Error! particle %d rhoPreMu is NAN: thrown from Velocity_Correction_and_update ! %f,%f,%f,%f\n", i_idx,
               sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].y, sortedRhoPreMu[i_idx].z, sortedRhoPreMu[i_idx].w);
    }

    if (!(isfinite(sortedVelMas[i_idx].x) && isfinite(sortedVelMas[i_idx].y) && isfinite(sortedVelMas[i_idx].z))) {
        printf("Error! particle %d velocity is NAN: thrown from Velocity_Correction_and_update !%f,%f,%f\n", i_idx,
               sortedVelMas[i_idx].x, sortedVelMas[i_idx].y, sortedVelMas[i_idx].z);
    }
}
//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Jacobi_SOR_Iter(Real4* sortedRhoPreMu,
                                Real* A_Matrix,
                                Real3* V_old,
                                Real3* V_new,
                                Real3* b3vec,
                                Real* q_old,  // q=p^(n+1)-p^n
                                Real* q_new,  // q=p^(n+1)-p^n
                                Real* b1vec,
                                const uint* csrColInd,
                                const uint* numContacts,
                                int numAllMarkers,
                                bool _3dvector,
                                volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    uint startIdx = numContacts[i_idx] + 1;  // Reserve the starting index for the A_ii
    uint endIdx = numContacts[i_idx + 1];

    if (_3dvector) {
        Real3 aij_vj = mR3(0.0);
        for (int myIdx = startIdx; myIdx < endIdx; myIdx++) {
            aij_vj += A_Matrix[myIdx] * V_old[csrColInd[myIdx]];
        }
        V_new[i_idx] = (b3vec[i_idx] - aij_vj) / A_Matrix[startIdx - 1];
    } else {
        Real aij_pj = 0.0;
        for (int myIdx = startIdx; myIdx < endIdx; myIdx++) {
            aij_pj += A_Matrix[myIdx] * q_old[csrColInd[myIdx]];
        }
        if (A_Matrix[startIdx - 1] == 0.0)
            printf(" %d A_Matrix[startIdx - 1]= %f, type=%f \n", i_idx, A_Matrix[startIdx - 1],
                   sortedRhoPreMu[i_idx].w);

        q_new[i_idx] = (b1vec[i_idx] - aij_pj) / A_Matrix[startIdx - 1];
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Update_AND_Calc_Res(Real4* sortedRhoPreMu,
                                    Real3* V_old,
                                    Real3* V_new,
                                    Real* q_old,
                                    Real* q_new,
                                    Real* Residuals,
                                    const int numAllMarkers,
                                    bool _3dvector,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    Real omega = paramsD.PPE_relaxation;
    Real res = 0;
    if (_3dvector) {
        V_new[i_idx] = (1 - omega) * V_old[i_idx] + omega * V_new[i_idx];
        res = length(V_old[i_idx] - V_new[i_idx]);
        V_old[i_idx] = V_new[i_idx];
        //        if (!(isfinite(V_old[i_idx].x) && isfinite(V_old[i_idx].y) && isfinite(V_old[i_idx].z)))
        //            printf(" %d vel= %f,%f,%f\n", i_idx, V_old[i_idx].x, V_old[i_idx].y, V_old[i_idx].z);

    } else {
        q_new[i_idx] = (1 - omega) * q_old[i_idx] + omega * q_new[i_idx];
        res = abs(q_old[i_idx] - q_new[i_idx]);
        q_old[i_idx] = q_new[i_idx];
        if (!(isfinite(q_old[i_idx])))
            printf(" %d q= %f\n", i_idx, q_old[i_idx]);
    }
    Residuals[i_idx] = res;
}
//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================

ChFsiForceI2SPH::ChFsiForceI2SPH(
    ChBce* otherBceWorker,                   ///< Pointer to the ChBce object that handles BCE markers
    SphMarkerDataD* otherSortedSphMarkersD,  ///< Information of markers in the sorted array on device
    ProximityDataD*
        otherMarkersProximityD,           ///< Pointer to the object that holds the proximity of the markers on device
    FsiGeneralData* otherFsiGeneralData,  ///< Pointer to the sph general data
    SimParams* otherParamsH,              ///< Pointer to the simulation parameters on host
    NumberOfObjects* otherNumObjects      ///< Pointer to number of objects, fluid and boundary markers, etc.
    )
    : ChFsiForceParallel(otherBceWorker,
                         otherSortedSphMarkersD,
                         otherMarkersProximityD,
                         otherFsiGeneralData,
                         otherParamsH,
                         otherNumObjects) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}

ChFsiForceI2SPH::~ChFsiForceI2SPH() {}

void ChFsiForceI2SPH::Finalize() {
    ChFsiForceParallel::Finalize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH, sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH, sizeof(NumberOfObjects));
    hipMemcpyFromSymbol(paramsH, HIP_SYMBOL(paramsD), sizeof(SimParams));
    hipDeviceSynchronize();
    CopyParams_NumberOfObjects(paramsH, numObjectsH);

    //    int numAllMarkers = numObjectsH->numAllMarkers;
    //    _sumWij_inv.resize(numAllMarkers);
    //    Color.resize(numAllMarkers);
    //    G_i.resize(numAllMarkers * 9);
    //    A_i.resize(numAllMarkers * 27);
    //    L_i.resize(numAllMarkers * 6);
    //    Contact_i.resize(numAllMarkers);
    //
    //    thrust::fill(Contact_i.begin(), Contact_i.end(), 1e-3);
    //    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    //    thrust::fill(A_i.begin(), A_i.end(), 0);
    //    thrust::fill(L_i.begin(), L_i.end(), 0);
    //    thrust::fill(G_i.begin(), G_i.end(), 0);
}
//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================
void ChFsiForceI2SPH::ForceImplicitSPH(SphMarkerDataD* otherSphMarkersD,
                                       FsiBodiesDataD* otherFsiBodiesD,
                                       FsiMeshDataD* otherFsiMeshD) {
    std::cout << "dT in ForceImplicitSPH: " << paramsH->dT << "\n";
    CopyParams_NumberOfObjects(paramsH, numObjectsH);

    sphMarkersD = otherSphMarkersD;
    int numAllMarkers = numObjectsH->numAllMarkers;
    int numHelperMarkers = numObjectsH->numHelperMarkers;
    fsiCollisionSystem->ArrangeData(sphMarkersD);
    printf("ForceI2SPH numAllMarkers:%d,numHelperMarkers=%d\n", numAllMarkers, numHelperMarkers);

    bool *isErrorH, *isErrorD, *isErrorD2;

    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    hipMalloc((void**)&isErrorD2, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(isErrorD2, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);

    thrust::device_vector<Real> _sumWij_inv;
    thrust::device_vector<uint> Contact_i;
    thrust::device_vector<Real> G_i;
    thrust::device_vector<Real> A_i;
    thrust::device_vector<Real> L_i;
    thrust::device_vector<Real> Color;
    thrust::device_vector<uint> csrColInd;
    thrust::device_vector<unsigned long int> GlobalcsrColInd;
    thrust::device_vector<Real> csrValLaplacian;
    thrust::device_vector<Real3> csrValGradient;
    thrust::device_vector<Real> csrValFunciton;
    thrust::device_vector<Real> AMatrix;
    thrust::device_vector<Real3> V_Star;
    thrust::device_vector<Real3> Normals;

    _sumWij_inv.resize(numAllMarkers);
    Normals.resize(numAllMarkers);
    Color.resize(numAllMarkers);
    G_i.resize(numAllMarkers * 9);
    A_i.resize(numAllMarkers * 27);
    L_i.resize(numAllMarkers * 6);
    Contact_i.resize(numAllMarkers);
    thrust::fill(Contact_i.begin(), Contact_i.end(), 0);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    thrust::fill(G_i.begin(), G_i.end(), 0);
    fsiCollisionSystem->ArrangeData(sphMarkersD);

    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel");

    uint LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    int NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);

    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices");
    //    //
    //    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_IISPH_D, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);
    //    CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);
    //
    //    return;

    //============================================================================================================
    double A_L_Tensor_GradLaplacian = clock();
    printf(" calc_A_tensor+");

    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor");
    printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor");
    printf("Gradient_Laplacian_Operator: ");
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator");
    double Gradient_Laplacian_Operator = (clock() - A_L_Tensor_GradLaplacian) / (double)CLOCKS_PER_SEC;
    printf("%f (s)\n", Gradient_Laplacian_Operator);
    int numFlexbodies = +numObjectsH->numFlexBodies1D + numObjectsH->numFlexBodies2D;
    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int4 updatePortion =
        mI4(fsiGeneralData->referenceArray[haveGhost + haveHelper + 0].y,  // end of fluid
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1].y,  // end of boundary
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies].y,
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies + numFlexbodies].y);

    thrust::device_vector<Real3> V_star_new(numAllMarkers);
    thrust::device_vector<Real3> V_star_old(numAllMarkers);
    thrust::fill(V_star_old.begin(), V_star_old.end(), mR3(0.0));
    thrust::fill(V_star_new.begin(), V_star_new.end(), mR3(0.0));

    thrust::device_vector<Real> q_new(numAllMarkers);
    thrust::device_vector<Real> q_old(numAllMarkers);
    thrust::fill(q_old.begin(), q_old.end(), 0.0);
    thrust::fill(q_new.begin(), q_new.end(), 0.0);

    thrust::device_vector<Real> b1Vector(numAllMarkers);
    thrust::fill(b1Vector.begin(), b1Vector.end(), 0.0);
    thrust::device_vector<Real3> b3Vector(numAllMarkers);
    thrust::fill(b3Vector.begin(), b3Vector.end(), mR3(0.0));

    thrust::device_vector<Real> Residuals(numAllMarkers);
    thrust::fill(Residuals.begin(), Residuals.end(), 0.0);

    thrust::device_vector<Real> vx(numAllMarkers);
    thrust::device_vector<Real> vy(numAllMarkers);
    thrust::device_vector<Real> vz(numAllMarkers);
    thrust::device_vector<Real> xvec(numAllMarkers);
    //============================================================================================================
    double LinearSystemClock_V = clock();
    V_star_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(b3Vector), R1CAST(vx), R1CAST(vy), R1CAST(vz),
        mR3CAST(V_star_old), R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunciton),
        R1CAST(_sumWij_inv), mR3CAST(Normals), U1CAST(csrColInd), U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "V_star_Predictor");

    int Iteration = 0;
    Real MaxRes = 100;

    while ((MaxRes > paramsH->LinearSolver_Rel_Tol || Iteration < 3) && paramsH->USE_Iterative_solver &&
           Iteration < paramsH->LinearSolver_Max_Iter) {
        Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix),
                                                   mR3CAST(V_star_old), mR3CAST(V_star_new), mR3CAST(b3Vector),
                                                   R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd),
                                                   U1CAST(Contact_i), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");

        Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old),
                                                       mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                       R1CAST(Residuals), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
        Iteration++;
        thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
        unsigned int position = iter - Residuals.begin();
        MaxRes = *iter;

        if (paramsH->Verbose_monitoring)
            printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
    }

    thrust::device_vector<Real3>::iterator iter =
        thrust::max_element(V_star_new.begin(), V_star_new.end(), compare_Real3_mag());
    unsigned int position = iter - V_star_new.begin();
    Real MaxVel = length(*iter);

    uint FixedMarker = 5679;
    //    Real temp = 0;
    //    for (int i = 0; i < numAllMarkers; i++) {
    //        if (Real4(sortedSphMarkersD->rhoPresMuD[i]).w == -1 && Real4(sortedSphMarkersD->posRadD[i]).z > temp) {
    //            FixedMarker = i;
    //            //            break;
    //        }
    //    }
    printf("Fixed marker %d\n", FixedMarker);

    double V_star_Predictor = (clock() - LinearSystemClock_V) / (double)CLOCKS_PER_SEC;
    printf(" V_star_Predictor Equation: %f (sec) - Final Residual=%.3e\n", V_star_Predictor, MaxRes);
    //============================================================================================================
    Iteration = 0;
    MaxRes = 100;
    double LinearSystemClock_p = clock();
    thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
    Pressure_Equation<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), R1CAST(b1Vector), mR3CAST(V_star_new), R1CAST(q_new),
        R1CAST(csrValFunciton), R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(_sumWij_inv), mR3CAST(Normals),
        U1CAST(csrColInd), U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, FixedMarker, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Pressure_Equation");

    if (paramsH->USE_LinearSolver) {
        ChFsiLinearSolver myLS(paramsH->LinearSolver, paramsH->LinearSolver_Abs_Tol, paramsH->LinearSolver_Abs_Tol,
                               paramsH->LinearSolver_Max_Iter, paramsH->Verbose_monitoring);
        if (paramsH->PPE_Solution_type != FORM_SPARSE_MATRIX) {
            printf(
                "You should paramsH->PPE_Solution_type == FORM_SPARSE_MATRIX in order to use the "
                "chrono_fsi linear "
                "solvers\n");
            exit(0);
        }
        myLS.Solve(numAllMarkers, NNZ, R1CAST(AMatrix), U1CAST(Contact_i), U1CAST(csrColInd), (double*)R1CAST(q_new),
                   R1CAST(b1Vector));
        cudaCheckError();
        if (myLS.GetSolverStatus()) {
            std::cout << " Linear solver converged to " << myLS.GetResidual() << " tolerance";
            std::cout << " after " << myLS.GetNumIterations() << " iterations" << std::endl;
        } else {
            std::cout << "Failed to converge after " << myLS.GetIterationLimit() << " iterations";
            std::cout << " (" << myLS.GetResidual() << " final residual)" << std::endl;
        }
    } else {
        while ((MaxRes > paramsH->LinearSolver_Abs_Tol || Iteration < 3) && paramsH->USE_Iterative_solver &&
               Iteration < paramsH->LinearSolver_Max_Iter) {
            Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(
                mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(V_star_old), mR3CAST(V_star_new),
                mR3CAST(b3Vector), R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd), U1CAST(Contact_i),
                numAllMarkers, false, isErrorD);
            ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");

            Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old),
                                                           mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                           R1CAST(Residuals), numAllMarkers, false, isErrorD);
            ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
            Iteration++;
            thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
            unsigned int position = iter - Residuals.begin();
            MaxRes = *iter;

            if (paramsH->Verbose_monitoring)
                printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
        }
    }
    double Pressure_Computation = (clock() - LinearSystemClock_p) / (double)CLOCKS_PER_SEC;
    printf(" Pressure Poisson Equation: %f (sec) - Final Residual=%.3e\n", Pressure_Computation, MaxRes);
    //============================================================================================================
    double updateClock = clock();
    Velocity_Correction_and_update<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(V_star_new), R1CAST(q_new), mR3CAST(csrValGradient),
        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, MaxVel, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Velocity_Correction_and_update");
    double updateComputation = (clock() - updateClock) / (double)CLOCKS_PER_SEC;
    printf(" Pressure Poisson Equation: %f (sec)\n", updateComputation);
    //============================================================================================================

    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_IISPH_D, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);

    _sumWij_inv.clear();
    Color.clear();
    Contact_i.clear();
    AMatrix.clear();
    csrColInd.clear();
    G_i.clear();
    A_i.clear();
    L_i.clear();
    csrValLaplacian.clear();
    csrValGradient.clear();
    V_star_old.clear();
    V_star_new.clear();
    b3Vector.clear();
    q_old.clear();
    q_new.clear();
    b1Vector.clear();

}  // namespace fsi

}  // namespace fsi
}  // namespace chrono
