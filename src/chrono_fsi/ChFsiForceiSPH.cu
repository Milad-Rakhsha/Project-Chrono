#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <sstream>

#include "chrono_fsi/ChFsiForceiSPH.cuh"

//==========================================================================================================================================
namespace chrono {
namespace fsi {

// Note that this function may be used for different cases depending on delta_t
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void V_star(Real4* sortedPosRad,  // input: sorted positions
                       Real3* sortedVelMas,
                       Real4* sortedRhoPreMu,
                       Real* A_Matrix,
                       Real3* Bi,
                       Real3* v_old,

                       const Real* A_L,
                       const Real3* A_G,
                       const Real* A_f,

                       const Real* sumWij_inv,
                       Real3* normals,

                       const uint* csrColInd,
                       const uint* numContacts,

                       Real4* velMassRigid_fsiBodies_D,
                       Real3* accRigid_fsiBodies_D,
                       uint* rigidIdentifierD,

                       Real3* pos_fsi_fea_D,
                       Real3* vel_fsi_fea_D,
                       Real3* acc_fsi_fea_D,
                       uint* FlexIdentifierD,
                       int numFlex1D,
                       uint2* CableElementsNodes,
                       uint4* ShellelementsNodes,

                       int4 updatePortion,
                       uint* gridMarkerIndexD,

                       int numAllMarkers,
                       Real delta_t,
                       volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    Real CN = 0.5;

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1;
        Bi[i_idx] = mR3(0.0);
        return;
    }

    Real rho0 = paramsD.rho0;
    Real rhoi = sortedRhoPreMu[i_idx].x;
    Real3 grad_rho_i = mR3(0.0);

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        grad_rho_i += A_G[count] * sortedRhoPreMu[j].x;
    }
    if (Fluid_Marker) {
        //======================== Interior ===========================
        // Navier-Stokes
        Real3 rhs = mR3(0.0);
        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            int j = csrColInd[count];
            A_Matrix[count] = -delta_t * CN * paramsD.mu0 / rhoi * A_L[count];
            rhs += delta_t * (1 - CN) * paramsD.mu0 / rhoi * A_L[count] * sortedVelMas[j];  // viscous term;
        }
        A_Matrix[csrStartIdx] += 1;
        Bi[i_idx] = rhs + sortedVelMas[i_idx]                                    //forward euler term from lhs
                    + paramsD.gravity * delta_t + paramsD.bodyForce3 * delta_t;  // body force
    } else if (Boundary_Marker) {
        //======================== Boundary ===========================
        Real h_i = sortedPosRad[i_idx].w;
        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real den = 0.0;
        for (uint count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w != -1)
                continue;
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real h_j = sortedPosRad[j].w;
            Real h_ij = 0.5 * (h_j + h_i);
            Real W3 = W3h(length(rij), h_ij);
            A_Matrix[count] = W3;
            // A_Matrix[count] = A_f[count];
            den = den + W3;
        }

        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);
        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                    accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D,
                    FlexIdentifierD, numFlex1D, CableElementsNodes, ShellelementsNodes);

        if (abs(den) < EPSILON) {
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = V_prescribed;
        } else {
            A_Matrix[csrStartIdx] = den;
            Bi[i_idx] = 2 * V_prescribed * den;
        }
    }

    v_old[i_idx] = sortedVelMas[i_idx];
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void update_r(Real4* sortedPosRad,
                         Real3* sortedVelMas,
                         Real4* sortedRhoPreMu,
                         Real* q_i,

                         const Real3* A_G,
                         const uint* csrColInd,
                         const uint* numContacts,
                         int numAllMarkers,
                         Real delta_t,
                         bool UpdateWithPressureGradient) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    sortedRhoPreMu[i_idx].y = q_i[i_idx];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    if (!Fluid_Marker) {
        return;
    }

    if (UpdateWithPressureGradient) {
        uint csrStartIdx = numContacts[i_idx];
        uint csrEndIdx = numContacts[i_idx + 1];
        Real3 grad_p_i = mR3(0.0);
        // Calculating the grad.p
        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            int j = csrColInd[count];
            grad_p_i += A_G[count] * q_i[j];
        }
        sortedPosRad[i_idx] -= mR4(delta_t * delta_t / 2.0 * grad_p_i / sortedRhoPreMu[i_idx].x, 0.0);
    } else {  // simple update from velocity and time step
        sortedPosRad[i_idx] += mR4(sortedVelMas[i_idx] * delta_t, 0.0);
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Pressure_Equation(Real4* sortedPosRad,  // input: sorted positions
                                  Real3* sortedVelMas,
                                  Real4* sortedRhoPreMu,
                                  Real* A_Matrix,
                                  Real* Bi,
                                  Real3* Vstar,
                                  Real* q_old,

                                  const Real* A_f,
                                  const Real* A_L,
                                  const Real3* A_G,
                                  const Real* sumWij_inv,
                                  Real3* Normals,
                                  const uint* csrColInd,
                                  const uint* numContacts,

                                  Real4* velMassRigid_fsiBodies_D,
                                  Real3* accRigid_fsiBodies_D,
                                  uint* rigidIdentifierD,

                                  Real3* pos_fsi_fea_D,
                                  Real3* vel_fsi_fea_D,
                                  Real3* acc_fsi_fea_D,
                                  uint* FlexIdentifierD,
                                  int numFlex1D,
                                  uint2* CableElementsNodes,
                                  uint4* ShellelementsNodes,

                                  int4 updatePortion,
                                  uint* gridMarkerIndexD,
                                  int numAllMarkers,
                                  int FixedMarker,
                                  bool v_star_source,
                                  Real delta_t,
                                  volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1.0;
        Bi[i_idx] = 0.0;
        return;
    }

    Real3 gravity = paramsD.gravity;
    Real3 grad_rho_i = mR3(0.0);
    Real div_vi_star = 0;

    // Calculating the div.v* and grad(rho)
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        div_vi_star += dot(A_G[count], Vstar[j]);
        grad_rho_i += A_G[count] * sortedRhoPreMu[j].x;
    }

    Real rhoi = sortedRhoPreMu[i_idx].x;
    Real rhoi_star = rhoi;

    //    Real rhoi_star = rhoi - rhoi * div_vi_star * dt;
    //    Real rhoi_star = rhoi + dot(grad_rho_i, Vstar[i_idx] * delta_t);

    //======================== Interior ===========================
    if (Fluid_Marker) {
        //        if (sortedRhoPreMu[i_idx].x < 0.90 * paramsD.rho0) {
        //            A_Matrix[csrStartIdx] = 1.0;
        //            Bi[i_idx] = 0.0;
        //        }
        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            A_Matrix[count] = 1.0 / rhoi * A_L[count] - 1.0 / (rhoi * rhoi) * dot(grad_rho_i, A_G[count]);
        }
        if (v_star_source)
            Bi[i_idx] = div_vi_star / delta_t;
        else
            Bi[i_idx] = 2.0 / (delta_t * delta_t) * (paramsD.rho0 - rhoi_star) / paramsD.rho0;

        //        if (abs(Bi[i_idx]) > 10.0)
        //            printf("B[i]=%f\t", Bi[i_idx]);
        //======================= Boundary ===========================
    } else if (Boundary_Marker) {
        Real3 my_normal = Normals[i_idx];
        for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w == -1.0) {
                A_Matrix[count] = -dot(A_G[count], my_normal);
                A_Matrix[csrStartIdx] += +dot(A_G[count], my_normal);
            }
        }
        Bi[i_idx] = 0;
        if (abs(A_Matrix[csrStartIdx]) < EPSILON) {
            clearRow(i_idx, csrStartIdx, csrEndIdx, A_Matrix, Bi);
            A_Matrix[csrStartIdx] = 1;
            Bi[i_idx] = 0.0;
        } else {
            for (int count = csrStartIdx + 1; count < csrEndIdx; count++)
                A_Matrix[count] /= A_Matrix[csrStartIdx];

            A_Matrix[csrStartIdx] = 1.0;
        }
    }
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Velocity_Correction(Real4* sortedPosRad,
                                    Real4* sortedRhoPreMu,
                                    Real3* sortedVelMas,
                                    Real3* sortedVisVel,
                                    Real3* Vstar,
                                    Real* q_i,  // q=p^(n+1)-p^n
                                    const Real* A_f,
                                    const Real3* A_G,
                                    const uint* csrColInd,
                                    const uint* numContacts,
                                    int numAllMarkers,
                                    const Real MaxVel,
                                    const Real delta_t,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];
    Real3 grad_q_i = mR3(0.0);
    Real3 grad_p_nPlus1 = mR3(0.0);

    Real3 inner_sum = mR3(0.0), shift_r = mR3(0.0);
    Real mi_bar = 0.0, r0 = 0.0;

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        grad_q_i += A_G[count] * q_i[j];
        grad_q_i += A_G[count] * q_i[j];
        grad_p_nPlus1 += A_G[count] * (sortedRhoPreMu[j].y + q_i[j]);
        Real3 rij = Distance(mR3(sortedPosRad[i_idx]), mR3(sortedPosRad[j]));
        Real d = length(rij);
        if (count == csrStartIdx)
            continue;
        Real m_j = pow(sortedPosRad[j].w * paramsD.MULT_INITSPACE, 3) * paramsD.rho0;
        mi_bar += m_j;
        r0 += d;
        inner_sum += m_j * rij / (d * d * d);
    }
    if (sortedRhoPreMu[i_idx].w == -1.0)
        r0 /= (csrEndIdx - csrStartIdx - 1);

    shift_r = paramsD.beta_shifting * r0 * r0 * length(MaxVel) * delta_t / mi_bar * inner_sum;

    Real3 V_new = Vstar[i_idx] - delta_t / sortedRhoPreMu[i_idx].x / 2 * grad_q_i;

    sortedVelMas[i_idx] = V_new;
    sortedRhoPreMu[i_idx].y = q_i[i_idx];
    sortedVisVel[i_idx] = sortedVelMas[i_idx];

    Real3 grad_p = mR3(0.0);
    Real3 grad_rho = mR3(0.0);
    Real3 grad_ux = mR3(0.0);
    Real3 grad_uy = mR3(0.0);
    Real3 grad_uz = mR3(0.0);

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        grad_p += A_G[count] * sortedRhoPreMu[i_idx].y;
        grad_rho += A_G[count] * sortedRhoPreMu[i_idx].x;
        grad_ux += A_G[count] * sortedVelMas[i_idx].x;
        grad_uy += A_G[count] * sortedVelMas[i_idx].y;
        grad_uz += A_G[count] * sortedVelMas[i_idx].z;
    }

    if (true && sortedRhoPreMu[i_idx].w == -1.0) {
        sortedPosRad[i_idx] += mR4(shift_r, 0.0);
        sortedRhoPreMu[i_idx].y += dot(shift_r, grad_p);
        sortedRhoPreMu[i_idx].x += dot(shift_r, grad_rho);
        sortedVelMas[i_idx].x += dot(shift_r, grad_ux);
        sortedVelMas[i_idx].y += dot(shift_r, grad_uy);
        sortedVelMas[i_idx].z += dot(shift_r, grad_uz);
    }

    Real3 vis_vel = mR3(0.0);
    Real div_V = 0.0;

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        vis_vel += A_f[count] * (sortedVelMas[j]);
        div_V += dot(A_G[count], sortedVelMas[j]);
    }

    //    sortedRhoPreMu[i_idx].x += -sortedRhoPreMu[i_idx].x * div_V;
    sortedVisVel[i_idx] = sortedVelMas[i_idx];
    sortedVelMas[i_idx] = paramsD.EPS_XSPH * vis_vel + (1 - paramsD.EPS_XSPH) * sortedVelMas[i_idx];
}

//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================
ChFsiForceiSPH::ChFsiForceiSPH(
    ChBce* otherBceWorker,                   ///< Pointer to the ChBce object that handles BCE markers
    SphMarkerDataD* otherSortedSphMarkersD,  ///< Information of markers in the sorted array on device
    ProximityDataD*
        otherMarkersProximityD,           ///< Pointer to the object that holds the proximity of the markers on device
    FsiGeneralData* otherFsiGeneralData,  ///< Pointer to the sph general data
    SimParams* otherParamsH,              ///< Pointer to the simulation parameters on host
    NumberOfObjects* otherNumObjects      ///< Pointer to number of objects, fluid and boundary markers, etc.
    )
    : ChFsiForce(otherBceWorker,
                 otherSortedSphMarkersD,
                 otherMarkersProximityD,
                 otherFsiGeneralData,
                 otherParamsH,
                 otherNumObjects) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}

ChFsiForceiSPH::~ChFsiForceiSPH() {}

void ChFsiForceiSPH::Finalize() {
    ChFsiForce::Finalize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH, sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH, sizeof(NumberOfObjects));
    hipMemcpyFromSymbol(paramsH, HIP_SYMBOL(paramsD), sizeof(SimParams));
    hipDeviceSynchronize();
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
    numAllMarkers = numObjectsH->numAllMarkers;
    _sumWij_inv.resize(numAllMarkers);
    Normals.resize(numAllMarkers);
    G_i.resize(numAllMarkers * 9);
    A_i.resize(numAllMarkers * 27);
    L_i.resize(numAllMarkers * 6);
    Contact_i.resize(numAllMarkers);
    V_star_new.resize(numAllMarkers);
    V_star_old.resize(numAllMarkers);
    q_new.resize(numAllMarkers);
    q_old.resize(numAllMarkers);
    b1Vector.resize(numAllMarkers);
    b3Vector.resize(numAllMarkers);
    Residuals.resize(numAllMarkers);
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    hipMalloc((void**)&isErrorD2, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(isErrorD2, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
}

void ChFsiForceiSPH::PreProcessor(SphMarkerDataD* sortedSphMarkersD, bool print) {
    numAllMarkers = numObjectsH->numAllMarkers;
    Contact_i.resize(numAllMarkers);
    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 128, numBlocks, numThreads);
    thrust::fill(Contact_i.begin(), Contact_i.end(), 0);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    thrust::fill(G_i.begin(), G_i.end(), 0);

    thrust::device_vector<Real3>::iterator iter =
        thrust::max_element(sortedSphMarkersD->velMasD.begin(), sortedSphMarkersD->velMasD.end(), compare_Real3_mag());
    Real MaxVel = length(*iter);

    if (paramsH->Adaptive_time_stepping) {
        Real dt_CFL = paramsH->Co_number * paramsH->HSML / MaxVel;
        Real dt_nu = 0.125 * paramsH->HSML * paramsH->HSML / (paramsH->mu0 / paramsH->rho0);
        Real dt_body = 0.25 * std::sqrt(paramsH->HSML / length(paramsH->bodyForce3 + paramsH->gravity));
        Real dt = std::min(dt_body, std::min(dt_CFL, dt_nu));
        if (dt / paramsH->dT_Max > 0.7 && dt / paramsH->dT_Max < 1)
            paramsH->dT = paramsH->dT_Max * 0.5;
        else
            paramsH->dT = std::min(dt, paramsH->dT_Max);

        printf(" time step=%.3e, dt_Max=%.3e, dt_CFL=%.3e (CFL=%.2g), dt_nu=%.3e, dt_body=%.3e\n", paramsH->dT,
               paramsH->dT_Max, dt_CFL, paramsH->Co_number, dt_nu, dt_body);
    }
    //============================================================================================================
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel");

    uint LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);

    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices");

    //============================================================================================================
    double A_L_Tensor_GradLaplacian = clock();
    if (print)
        printf(" calc_A_tensor+");
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor");
    if (print)
        printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor");
    if (print)
        printf("Gradient_Laplacian_Operator: ");
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator");
    double Gradient_Laplacian_Operator = (clock() - A_L_Tensor_GradLaplacian) / (double)CLOCKS_PER_SEC;
    if (print)
        printf("%f (s)\n", Gradient_Laplacian_Operator);
}

//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================
void ChFsiForceiSPH::ForceImplicitSPH(SphMarkerDataD* otherSphMarkersD,
                                      FsiBodiesDataD* otherFsiBodiesD,
                                      FsiMeshDataD* otherFsiMeshD) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
    SphMarkerDataD sphMarkers_n;

    sphMarkersD = otherSphMarkersD;
    fsiCollisionSystem->ArrangeData(sphMarkersD);

    std::cout << "dT in ForceImplicitSPH: " << paramsH->dT << "\n";

    //=====calcRho_kernel=== calc_A_tensor==calc_L_tensor==Function_Gradient_Laplacian_Operator=================
    ChFsiForceiSPH::PreProcessor(sortedSphMarkersD);
    //==========================================================================================================
    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);

    int numHelperMarkers = numObjectsH->numHelperMarkers;
    int numFlexbodies = +numObjectsH->numFlexBodies1D + numObjectsH->numFlexBodies2D;
    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int4 updatePortion =
        mI4(fsiGeneralData->referenceArray[haveGhost + haveHelper + 0].y,  // end of fluid
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1].y,  // end of boundary
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies].y,
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies + numFlexbodies].y);
    printf("ForceiSPH numAllMarkers:%d,numHelperMarkers=%d\n", numAllMarkers, numHelperMarkers);

    thrust::fill(V_star_old.begin(), V_star_old.end(), mR3(0.0));
    thrust::fill(V_star_new.begin(), V_star_new.end(), mR3(0.0));
    thrust::fill(b3Vector.begin(), b3Vector.end(), mR3(0.0));
    thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
    thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
    //============================================V_star_Predictor===============================================
    double LinearSystemClock_V = clock();
    V_star<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(b3Vector), mR3CAST(V_star_old),
        R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunciton), R1CAST(_sumWij_inv), mR3CAST(Normals),
        U1CAST(csrColInd), U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, paramsH->dT / 2, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "V_star_Predictor");

    int Iteration = 0;
    Real MaxRes = 100;
    while ((MaxRes > paramsH->LinearSolver_Rel_Tol || Iteration < 3) && Iteration < paramsH->LinearSolver_Max_Iter) {
        Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix),
                                                   mR3CAST(V_star_old), mR3CAST(V_star_new), mR3CAST(b3Vector),
                                                   R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd),
                                                   U1CAST(Contact_i), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");
        Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old),
                                                       mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                       R1CAST(Residuals), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
        Iteration++;
        thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
        unsigned int position = iter - Residuals.begin();
        MaxRes = *iter;
        if (paramsH->Verbose_monitoring)
            printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
    }

    thrust::device_vector<Real3> sortedV_nold = sortedSphMarkersD->velMasD;
    sortedSphMarkersD->velMasD = V_star_new;

    update_r<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(V_star_new),
                                        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(q_new), mR3CAST(csrValGradient),
                                        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, paramsH->dT, false);

    double V_star_Predictor = (clock() - LinearSystemClock_V) / (double)CLOCKS_PER_SEC;
    printf(" V_star_Predictor (n+1/2) Equation: %f (sec) - Final Residual=%.3e - #Iter=%d\n", V_star_Predictor, MaxRes,
           Iteration);

    Real Ave_density = 1.0;

    std::cout << " Fixing density variation ...\n";
    int iteration_density = 0;
    while (Ave_density > 0.001 && iteration_density < 1) {
        CopySortedToOriginal_NonInvasive_R4(sphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                            markersProximityD->gridMarkerIndexD);
        CopySortedToOriginal_NonInvasive_R4(sphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                            markersProximityD->gridMarkerIndexD);
        CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                            markersProximityD->gridMarkerIndexD);

        fsiCollisionSystem->ArrangeData(sphMarkersD);
        ChFsiForceiSPH::PreProcessor(sortedSphMarkersD, false);

        Real4_x unary_op(paramsH->rho0);
        thrust::plus<Real> binary_op;
        Ave_density = thrust::transform_reduce(sphMarkersD->rhoPresMuD.begin(), sphMarkersD->rhoPresMuD.end(), unary_op,
                                               0.0, binary_op) /
                      (numObjectsH->numFluidMarkers * paramsH->rho0);
        iteration_density++;

        double LinearSystemClock_p = clock();
        printf(" iter=%d, ", iteration_density);

        thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
        thrust::fill(b1Vector.begin(), b1Vector.end(), 0.0);
        //        thrust::fill(q_old.begin(), q_old.end(), 0.0);
        //        thrust::fill(q_new.begin(), q_new.end(), 0.0);
        Pressure_Equation<<<numBlocks, numThreads>>>(
            mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
            mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), R1CAST(b1Vector), mR3CAST(V_star_new),
            R1CAST(q_new), R1CAST(csrValFunciton), R1CAST(csrValLaplacian), mR3CAST(csrValGradient),
            R1CAST(_sumWij_inv), mR3CAST(Normals), U1CAST(csrColInd), U1CAST(Contact_i),

            mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
            U1CAST(fsiGeneralData->rigidIdentifierD),

            mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
            mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

            numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
            U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
            numAllMarkers, 0, false, paramsH->dT, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Pressure_Equation");

        if (paramsH->USE_LinearSolver) {
            ChFsiLinearSolver myLS(paramsH->LinearSolver, 0.0, paramsH->LinearSolver_Abs_Tol,
                                   paramsH->LinearSolver_Max_Iter, paramsH->Verbose_monitoring);
            if (paramsH->PPE_Solution_type != FORM_SPARSE_MATRIX) {
                printf(
                    "You should paramsH->PPE_Solution_type == FORM_SPARSE_MATRIX in order to use the "
                    "chrono_fsi linear "
                    "solvers\n");
                exit(0);
            }
            myLS.Solve(numAllMarkers, NNZ, R1CAST(AMatrix), U1CAST(Contact_i), U1CAST(csrColInd),
                       (double*)R1CAST(q_new), R1CAST(b1Vector));
            cudaCheckError();
            double Pressure_Computation = (clock() - LinearSystemClock_p) / (double)CLOCKS_PER_SEC;
            printf(" \tInner PPE: %f (sec) - Final Residual=%.3e - #Iter=%d - relative density Error=%.3e\n",
                   Pressure_Computation, myLS.GetResidual(), myLS.GetNumIterations(), Ave_density);
        } else {
            thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
            MaxRes = 100;
            Iteration = 0;
            while (MaxRes > 1e-4 && Iteration < paramsH->LinearSolver_Max_Iter) {
                Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(
                    mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(V_star_old), mR3CAST(V_star_new),
                    mR3CAST(b3Vector), R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd),
                    U1CAST(Contact_i), numAllMarkers, false, isErrorD);
                ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");

                Update_AND_Calc_Res<<<numBlocks, numThreads>>>(
                    mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old), mR3CAST(V_star_new), R1CAST(q_old),
                    R1CAST(q_new), R1CAST(Residuals), numAllMarkers, false, isErrorD);
                ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
                Iteration++;
                thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
                unsigned int position = iter - Residuals.begin();
                MaxRes = *iter;

                if (paramsH->Verbose_monitoring)
                    printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
            }
            double Pressure_Computation = (clock() - LinearSystemClock_p) / (double)CLOCKS_PER_SEC;
            printf(" \tInner PPE: %f (sec) - Final Residual=%.3e - #Iter=%d - relative density Error=%.3e\n",
                   Pressure_Computation, MaxRes, Iteration, Ave_density);
        }

        if (MaxRes < 100)
            update_r<<<numBlocks, numThreads>>>(
                mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
                mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(q_new), mR3CAST(csrValGradient), U1CAST(csrColInd),
                U1CAST(Contact_i), numAllMarkers, paramsH->dT, true);
        else {
            thrust::fill(q_old.begin(), q_old.end(), 0.0);
            thrust::fill(q_new.begin(), q_new.end(), 0.0);
            iteration_density--;
        }
    }

    //    CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);

    //==================================================Second Stage==============================================
    Iteration = 0;
    double LinearSystemClock_p = clock();
    thrust::fill(V_star_old.begin(), V_star_old.end(), mR3(0.0));
    thrust::fill(V_star_new.begin(), V_star_new.end(), mR3(0.0));
    thrust::fill(b3Vector.begin(), b3Vector.end(), mR3(0.0));
    thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
    thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
    LinearSystemClock_V = clock();
    V_star<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(b3Vector), mR3CAST(V_star_old),
        R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunciton), R1CAST(_sumWij_inv), mR3CAST(Normals),
        U1CAST(csrColInd), U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, paramsH->dT / 2, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "V_star_Predictor");

    thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
    MaxRes = 100;

    while ((MaxRes > paramsH->LinearSolver_Abs_Tol || Iteration < 3) && Iteration < paramsH->LinearSolver_Max_Iter) {
        Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix),
                                                   mR3CAST(V_star_old), mR3CAST(V_star_new), mR3CAST(b3Vector),
                                                   R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd),
                                                   U1CAST(Contact_i), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");

        Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old),
                                                       mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                       R1CAST(Residuals), numAllMarkers, true, isErrorD);
        ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
        Iteration++;
        thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
        unsigned int position = iter - Residuals.begin();
        MaxRes = *iter;

        if (paramsH->Verbose_monitoring)
            printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
    }
    V_star_Predictor = (clock() - LinearSystemClock_V) / (double)CLOCKS_PER_SEC;
    printf(" V_star_Predictor (n+1) Equation: %f (sec) - Final Residual=%.3e - #Iter=%d\n", V_star_Predictor, MaxRes,
           Iteration);

    thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
    thrust::fill(b1Vector.begin(), b1Vector.end(), 0.0);
    thrust::fill(q_old.begin(), q_old.end(), 0.0);
    thrust::fill(q_new.begin(), q_new.end(), 0.0);

    Pressure_Equation<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), R1CAST(b1Vector), mR3CAST(V_star_new), R1CAST(q_new),
        R1CAST(csrValFunciton), R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(_sumWij_inv), mR3CAST(Normals),
        U1CAST(csrColInd), U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, 0, true, paramsH->dT / 2, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Pressure_Equation");

    if (paramsH->USE_LinearSolver) {
        ChFsiLinearSolver myLS(paramsH->LinearSolver, 0.0, paramsH->LinearSolver_Abs_Tol,
                               paramsH->LinearSolver_Max_Iter, paramsH->Verbose_monitoring);
        if (paramsH->PPE_Solution_type != FORM_SPARSE_MATRIX) {
            printf(
                "You should paramsH->PPE_Solution_type == FORM_SPARSE_MATRIX in order to use the "
                "chrono_fsi linear "
                "solvers\n");
            exit(0);
        }
        myLS.Solve(numAllMarkers, NNZ, R1CAST(AMatrix), U1CAST(Contact_i), U1CAST(csrColInd), (double*)R1CAST(q_new),
                   R1CAST(b1Vector));
        cudaCheckError();
        if (myLS.GetSolverStatus()) {
            std::cout << " Linear solver converged to " << myLS.GetResidual() << " tolerance";
            std::cout << " after " << myLS.GetNumIterations() << " iterations" << std::endl;
        } else {
            std::cout << "Failed to converge after " << myLS.GetNumIterations() << " iterations";
            std::cout << " (" << myLS.GetResidual() << " final residual)" << std::endl;
        }
    } else {
        thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
        MaxRes = 100;

        while ((MaxRes > paramsH->LinearSolver_Abs_Tol || Iteration < 3) &&
               Iteration < paramsH->LinearSolver_Max_Iter) {
            Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(
                mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(AMatrix), mR3CAST(V_star_old), mR3CAST(V_star_new),
                mR3CAST(b3Vector), R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(csrColInd), U1CAST(Contact_i),
                numAllMarkers, false, isErrorD);
            ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Jacobi_SOR_Iter");

            Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_star_old),
                                                           mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                           R1CAST(Residuals), numAllMarkers, false, isErrorD);
            ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_AND_Calc_Res");
            Iteration++;
            thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
            unsigned int position = iter - Residuals.begin();
            MaxRes = *iter;

            if (paramsH->Verbose_monitoring)
                printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
        }
    }
    double Pressure_Computation = (clock() - LinearSystemClock_p) / (double)CLOCKS_PER_SEC;
    printf(" Pressure Poisson Equation: %f (sec) - Final Residual=%.3e - #Iter=%d\n", Pressure_Computation, MaxRes,
           Iteration);
    //==================================Velocity_Correction_and_update============================================
    double updateClock = clock();
    thrust::device_vector<Real3>::iterator iter =
        thrust::max_element(sortedSphMarkersD->velMasD.begin(), sortedSphMarkersD->velMasD.end(), compare_Real3_mag());
    Real MaxVel = length(*iter);
    thrust::fill(vel_vis_Sorted_D.begin(), vel_vis_Sorted_D.end(), mR3(0.0));
    Velocity_Correction<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(vel_vis_Sorted_D), mR3CAST(V_star_new), R1CAST(q_new),
        R1CAST(csrValFunciton), mR3CAST(csrValGradient), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, MaxVel,
        paramsH->dT, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Velocity_Correction_and_update");
    double updateComputation = (clock() - updateClock) / (double)CLOCKS_PER_SEC;
    printf(" Update computation: %f (sec)\n", updateComputation);
    //============================================================================================================
    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vis_vel_SPH_D, vel_vis_Sorted_D,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);

    csrValGradient.clear();
    csrValLaplacian.clear();
    csrValFunciton.clear();
    AMatrix.clear();
    Contact_i.clear();
    csrColInd.clear();
}
}  // namespace fsi
}  // namespace chrono
