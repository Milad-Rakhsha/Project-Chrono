#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <sstream>

#include "chrono_fsi/ChFsiForceXSPH.cuh"

//==========================================================================================================================================
namespace chrono {
namespace fsi {
// extern __constant__ SimParams paramsD;
// extern __constant__ NumberOfObjects numObjectsD;

struct compare_Real3_mag {
    __host__ __device__ bool operator()(Real3 lhs, Real3 rhs) { return length(lhs) < length(rhs); }
};

// double precision atomic add function
__device__ inline double datomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;

    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}
//==========================================================================================================================================
__global__ void calculate_pressure(Real4* sortedRhoPreMu, const int numAllMarkers, volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    sortedRhoPreMu[i_idx].y = Eos(sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].w);
}
//==========================================================================================================================================
__global__ void NS_RHS_Predictor(Real4* sortedPosRad,
                                 Real4* sortedRhoPreMu,
                                 Real3* sortedVelMas,
                                 Real3* NS_RHS,
                                 const Real3* A_G,
                                 const Real* A_L,
                                 const uint* csrColInd,
                                 const uint* numContacts,
                                 int numAllMarkers,
                                 //                                 const Real MaxVel,
                                 volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    Real3 grad_p_i = mR3(0.0);
    Real3 Laplacian_v_i = mR3(0.0);
    Real3 RHS = mR3(0.0);
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        grad_p_i += A_G[count] * sortedRhoPreMu[j].y;
        Laplacian_v_i += A_L[count] * sortedVelMas[j];
    }

    RHS = -1 / paramsD.rho0 * grad_p_i                  // pressure gradient
          + paramsD.mu0 / paramsD.rho0 * Laplacian_v_i  // viscous term;
          + paramsD.gravity;                            // body force

    NS_RHS[i_idx] = RHS;
    sortedPosRad[i_idx] = sortedPosRad[i_idx] + paramsD.dT / 2 * mR4(sortedVelMas[i_idx], 0.0);
    sortedVelMas[i_idx] = sortedVelMas[i_idx] + paramsD.dT / 2 * RHS;
}

//==========================================================================================================================================
__global__ void Update(Real4* PosRad_tn,
                       Real4* RhoPreMu_tn,
                       Real3* VelMas_tn,
                       Real3* VelMas_next,
                       Real4* PosRad_next,
                       Real3* fthalf,
                       int numAllMarkers,
                       // const Real MaxVel,
                       volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    bool Fluid_Marker = (RhoPreMu_tn[i_idx].w == -1.0);

    if (RhoPreMu_tn[i_idx].w <= -2) {
        return;
    }

    if (Fluid_Marker) {
        VelMas_next[i_idx] = VelMas_tn[i_idx] + paramsD.dT * fthalf[i_idx];
        PosRad_next[i_idx] =
            mR4(mR3(PosRad_tn[i_idx]) + paramsD.dT * VelMas_tn[i_idx] + 0.5 * paramsD.dT * paramsD.dT * fthalf[i_idx],
                PosRad_tn[i_idx].w);
    }
}
//==========================================================================================================================================
__global__ void Boundary_Conditions(Real4* sortedPosRad,
                                    Real4* sortedRhoPreMu,
                                    Real3* sortedVelMas,
                                    const Real3* A_G,
                                    const Real* A_L,
                                    const uint* csrColInd,
                                    const uint* numContacts,

                                    Real4* velMassRigid_fsiBodies_D,
                                    Real3* accRigid_fsiBodies_D,
                                    uint* rigidIdentifierD,

                                    Real3* pos_fsi_fea_D,
                                    Real3* vel_fsi_fea_D,
                                    Real3* acc_fsi_fea_D,
                                    uint* FlexIdentifierD,
                                    int numFlex1D,
                                    uint2* CableElementsNodes,
                                    uint4* ShellelementsNodes,

                                    int4 updatePortion,
                                    uint* gridMarkerIndexD,

                                    int numAllMarkers,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w == -1.0) {
        return;
    }
    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    Real h_i = sortedPosRad[i_idx].w;
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real3 myAcc = mR3(0);
    Real3 V_prescribed = mR3(0);
    BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D, FlexIdentifierD,
                numFlex1D, CableElementsNodes, ShellelementsNodes);
    Real pRHS1 = 0.0;
    Real pRHS2 = 0.0;

    Real den = 0.0;
    Real3 numV = mR3(0);

    for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        if (sortedRhoPreMu[j].w != -1.0)
            continue;
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real h_j = sortedPosRad[j].w;
        Real h_ij = 0.5 * (h_j + h_i);
        Real W3 = W3h(length(rij), h_ij);
        numV += W3 * sortedVelMas[j];
        den += W3;
        pRHS1 += W3 * sortedRhoPreMu[j].y;
        pRHS2 += dot(paramsD.gravity - myAcc, rij) * sortedRhoPreMu[j].x * W3;
    }

    if (abs(den) > EPSILON) {
        sortedVelMas[i_idx] = 2 * V_prescribed - numV / den;
        sortedRhoPreMu[i_idx].y = (pRHS1 + pRHS2) / den;
    } else {
        sortedVelMas[i_idx] = mR3(0);
        sortedRhoPreMu[i_idx].y = paramsD.BASEPRES;
    }
}
//==========================================================================================================================================

ChFsiForceXSPH::ChFsiForceXSPH(
    ChBce* otherBceWorker,                   ///< Pointer to the ChBce object that handles BCE markers
    SphMarkerDataD* otherSortedSphMarkersD,  ///< Information of markers in the sorted array on device
    ProximityDataD*
        otherMarkersProximityD,           ///< Pointer to the object that holds the proximity of the markers on device
    FsiGeneralData* otherFsiGeneralData,  ///< Pointer to the sph general data
    SimParams* otherParamsH,              ///< Pointer to the simulation parameters on host
    NumberOfObjects* otherNumObjects      ///< Pointer to number of objects, fluid and boundary markers, etc.
    )
    : ChFsiForce(otherBceWorker,
                 otherSortedSphMarkersD,
                 otherMarkersProximityD,
                 otherFsiGeneralData,
                 otherParamsH,
                 otherNumObjects) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}

ChFsiForceXSPH::~ChFsiForceXSPH() {}

void ChFsiForceXSPH::Finalize() {
    ChFsiForce::Finalize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH, sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH, sizeof(NumberOfObjects));
    hipMemcpyFromSymbol(paramsH, HIP_SYMBOL(paramsD), sizeof(SimParams));
    hipDeviceSynchronize();
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}
//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================
void ChFsiForceXSPH::ForceSPH(SphMarkerDataD* otherSphMarkersD,
                              FsiBodiesDataD* otherFsiBodiesD,
                              FsiMeshDataD* otherFsiMeshD) {
    std::cout << "dT in ForceImplicitSPH: " << paramsH->dT << "\n";
    CopyParams_NumberOfObjects(paramsH, numObjectsH);

    SphMarkerDataD SphMarkerDataD1 = *otherSphMarkersD;

    int numAllMarkers = numObjectsH->numAllMarkers;
    int numHelperMarkers = numObjectsH->numHelperMarkers;

    fsiCollisionSystem->ArrangeData(otherSphMarkersD);
    printf("ForceXSPH numAllMarkers:%d,numHelperMarkers=%d\n", numAllMarkers, numHelperMarkers);

    bool *isErrorH, *isErrorD, *isErrorD2;

    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    hipMalloc((void**)&isErrorD2, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(isErrorD2, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);
    thrust::device_vector<Real> _sumWij_inv;
    thrust::device_vector<uint> Contact_i;
    thrust::device_vector<Real> G_i;
    thrust::device_vector<Real> A_i;
    thrust::device_vector<Real> L_i;
    thrust::device_vector<uint> csrColInd;
    thrust::device_vector<Real> csrValLaplacian;
    thrust::device_vector<Real3> csrValGradient;
    thrust::device_vector<Real> csrValFunciton;
    thrust::device_vector<Real> AMatrix;
    thrust::device_vector<Real3> V_Star;
    thrust::device_vector<Real3> Normals;

    _sumWij_inv.resize(numAllMarkers);
    Normals.resize(numAllMarkers);
    G_i.resize(numAllMarkers * 9);
    A_i.resize(numAllMarkers * 27);
    L_i.resize(numAllMarkers * 6);
    Contact_i.resize(numAllMarkers);
    thrust::fill(Contact_i.begin(), Contact_i.end(), 0);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    thrust::fill(G_i.begin(), G_i.end(), 0);

    thrust::device_vector<Real3> ft(numAllMarkers, mR3(0));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    //============================================================================================================
    // some initialization
    // 1st call
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel");
    //============================================================================================================

    uint LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    int NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);

    //============================================================================================================
    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices   here");

    //============================================================================================================
    double A_L_Tensor_GradLaplacian = clock();
    printf("calc_A_tensor+");
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor");
    printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor");
    printf("Gradient_Laplacian_Operator: ");
    //============================================================================================================
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator");
    double Gradient_Laplacian_Operator = (clock() - A_L_Tensor_GradLaplacian) / (double)CLOCKS_PER_SEC;
    printf("%f (s)\n", Gradient_Laplacian_Operator);
    int numFlexbodies = +numObjectsH->numFlexBodies1D + numObjectsH->numFlexBodies2D;
    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int4 updatePortion =
        mI4(fsiGeneralData->referenceArray[haveGhost + haveHelper + 0].y,  // end of fluid
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1].y,  // end of boundary
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies].y,
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies + numFlexbodies].y);
    //============================================================================================================
    calculate_pressure<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calculate_pressure");
    // consider boundary condition
    //============================================================================================================
    Boundary_Conditions<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(csrValGradient), R1CAST(csrValLaplacian), U1CAST(csrColInd),
        U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Boundary_Conditions");
    //============================================================================================================
    NS_RHS_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(ft), mR3CAST(csrValGradient), R1CAST(csrValLaplacian),
        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "NS_RHS_Predictor");

    // 2nd call to calc f and v,x from predicted v and x
    CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);

    fsiCollisionSystem->ArrangeData(otherSphMarkersD);
    //============================================================================================================
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel-2");
    LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);
    //============================================================================================================
    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices-2");
    //============================================================================================================
    printf("calc_A_tensor+");
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor-2");
    printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor-2");
    printf("Gradient_Laplacian_Operator: ");
    //============================================================================================================
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator-2");

    //============================================================================================================
    // consider boundary condition
    calculate_pressure<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calculate_pressure-2");

    //============================================================================================================
    Boundary_Conditions<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(csrValGradient), R1CAST(csrValLaplacian), U1CAST(csrColInd),
        U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Boundary_Conditions-2");
    //============================================================================================================
    NS_RHS_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(ft), mR3CAST(csrValGradient), R1CAST(csrValLaplacian),
        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "NS_RHS_Predictor-2");

    thrust::device_vector<Real3> ft_unsorted(numAllMarkers);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));

    CopySortedToOriginal_NonInvasive_R3(ft_unsorted, ft, markersProximityD->gridMarkerIndexD);
    otherSphMarkersD->velMasD = SphMarkerDataD1.velMasD;
    otherSphMarkersD->posRadD = SphMarkerDataD1.posRadD;
    otherSphMarkersD->rhoPresMuD = SphMarkerDataD1.rhoPresMuD;
    // CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                     markersProximityD->gridMarkerIndexD);
    // CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
    //                                     markersProximityD->gridMarkerIndexD);
    //============================================================================================================
    Update<<<numBlocks, numThreads>>>(mR4CAST(SphMarkerDataD1.posRadD), mR4CAST(SphMarkerDataD1.rhoPresMuD),
                                      mR3CAST(SphMarkerDataD1.velMasD), mR3CAST(otherSphMarkersD->velMasD),
                                      mR4CAST(otherSphMarkersD->posRadD), mR3CAST(ft_unsorted), numAllMarkers,
                                      isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update");

    //    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_IISPH_D, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);
    //    CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);

    //    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->posRadD, sortedSphMarkersD->posRadD,
    //                                        markersProximityD->gridMarkerIndexD);
    //============================================================================================================

    _sumWij_inv.clear();
    Contact_i.clear();
    AMatrix.clear();
    csrColInd.clear();
    G_i.clear();
    A_i.clear();
    L_i.clear();
    csrValLaplacian.clear();
    csrValGradient.clear();
    ft_unsorted.clear();
    ft.clear();

}  // namespace fsi

}  // namespace fsi
}  // namespace chrono
