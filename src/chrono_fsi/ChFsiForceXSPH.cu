#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <sstream>

#include "chrono_fsi/ChFsiForceXSPH.cuh"

//==========================================================================================================================================
namespace chrono {
namespace fsi {

//==========================================================================================================================================
__global__ void calculate_pressure(Real4* sortedRhoPreMu, const int numAllMarkers, volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    sortedRhoPreMu[i_idx].y = Eos(sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].w);
}
//==========================================================================================================================================
__global__ void NS_RHS_Predictor(Real4* sortedPosRad,
                                 Real4* sortedRhoPreMu,
                                 Real3* sortedVelMas,
                                 Real3* NS_RHS,
                                 const Real3* A_G,
                                 const Real* A_L,
                                 const uint* csrColInd,
                                 const uint* numContacts,
                                 int numAllMarkers,
                                 //                                 const Real MaxVel,
                                 volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    Real3 grad_p_i = mR3(0.0);
    Real3 Laplacian_v_i = mR3(0.0);
    Real3 RHS = mR3(0.0);
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        grad_p_i += A_G[count] * sortedRhoPreMu[j].y;
        Laplacian_v_i += A_L[count] * sortedVelMas[j];
    }

    RHS = -1 / sortedRhoPreMu[i_idx].x * grad_p_i                  // pressure gradient
          + paramsD.mu0 / sortedRhoPreMu[i_idx].x * Laplacian_v_i  // viscous term;
          + paramsD.gravity;                                       // body force

    NS_RHS[i_idx] = RHS;
    if (Fluid_Marker)
        sortedPosRad[i_idx] = sortedPosRad[i_idx] + paramsD.dT / 2 * mR4(sortedVelMas[i_idx], 0.0);
    sortedVelMas[i_idx] = sortedVelMas[i_idx] + paramsD.dT / 2 * RHS;
}
//==========================================================================================================================================
__global__ void Update(Real4* PosRad_tn,
                       Real4* RhoPreMu_tn,
                       Real3* VelMas_tn,
                       Real3* VelMas_next,
                       Real4* PosRad_next,
                       Real3* fthalf,
                       int numAllMarkers,
                       // const Real MaxVel,
                       volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    bool Fluid_Marker = (RhoPreMu_tn[i_idx].w == -1.0);

    if (RhoPreMu_tn[i_idx].w <= -2) {
        return;
    }

    if (Fluid_Marker) {
        VelMas_next[i_idx] = VelMas_tn[i_idx] + paramsD.dT * fthalf[i_idx];
        PosRad_next[i_idx] =
            mR4(mR3(PosRad_tn[i_idx]) + paramsD.dT * VelMas_tn[i_idx] + 0.5 * paramsD.dT * paramsD.dT * fthalf[i_idx],
                PosRad_tn[i_idx].w);
    }
}
//==========================================================================================================================================
__global__ void Shifting_r(Real4* sortedPosRad,  // output to sortedPosRad
                           Real4* sortedRhoPreMu,
                           Real3* sortedVelMas,
                           Real3* sortedVisualVel,
                           const Real* A_f,
                           const Real3* A_G,
                           const uint* csrColInd,
                           const uint* numContacts,
                           int numAllMarkers,
                           const Real MaxVel,
                           volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    bool Fluid_Marker = (sortedRhoPreMu[i_idx].w == -1.0);

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    if (Fluid_Marker) {
        Real3 inner_sum = mR3(0.0), shift_r = mR3(0.0);
        Real mi_bar = 0.0, r0 = 0.0;
        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real h_i = sortedPosRad[i_idx].w;
        Real3 grad_i_wij = mR3(0.0);
        Real summation = 0.0;

        for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real h_j = sortedPosRad[j].w;
            Real h_ij = 0.5 * (h_j + h_i);
            Real3 rij = Distance(posRadA, posRadB);
            Real d = length(rij);
            Real m_j = pow(sortedPosRad[j].w * paramsD.MULT_INITSPACE, 3) * paramsD.rho0;
            mi_bar += m_j;
            r0 += d;
            inner_sum += m_j * rij / (d * d * d);

            Real3 grad_i_wij = GradWh(rij, h_ij);
            summation += m_j * dot((sortedVelMas[i_idx] - sortedVelMas[j]), grad_i_wij);
        }
        r0 /= (csrEndIdx - csrStartIdx - 1);
        shift_r = 0.5 * r0 * r0 * length(MaxVel) * paramsD.dT / mi_bar * inner_sum;
        sortedRhoPreMu[i_idx].x += paramsD.dT * summation;
        sortedRhoPreMu[i_idx].y = Eos(sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].w);

        Real3 grad_p = mR3(0.0);
        Real3 grad_rho = mR3(0.0);
        Real3 grad_ux = mR3(0.0);
        Real3 grad_uy = mR3(0.0);
        Real3 grad_uz = mR3(0.0);
        Real3 vis_vel = mR3(0.0);

        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            grad_rho += A_G[count] * sortedRhoPreMu[i_idx].x;
            grad_p += A_G[count] * sortedRhoPreMu[i_idx].y;
            grad_ux += A_G[count] * sortedVelMas[i_idx].x;
            grad_uy += A_G[count] * sortedVelMas[i_idx].y;
            grad_uz += A_G[count] * sortedVelMas[i_idx].z;
            vis_vel += A_f[count] * sortedVelMas[i_idx];
        }

        sortedPosRad[i_idx] += mR4(shift_r, 0.0);
        sortedRhoPreMu[i_idx].x += dot(shift_r, grad_rho);
        sortedRhoPreMu[i_idx].y += dot(shift_r, grad_p);
        sortedVelMas[i_idx].x += dot(shift_r, grad_ux);
        sortedVelMas[i_idx].y += dot(shift_r, grad_uy);
        sortedVelMas[i_idx].z += dot(shift_r, grad_uz);
        sortedVisualVel[i_idx] = vis_vel;
    }
}
//==========================================================================================================================================
__global__ void Boundary_Conditions(Real4* sortedPosRad,
                                    Real4* sortedRhoPreMu,
                                    Real3* sortedVelMas,
                                    const Real3* A_G,
                                    const Real* A_L,
                                    const uint* csrColInd,
                                    const uint* numContacts,

                                    Real4* velMassRigid_fsiBodies_D,
                                    Real3* accRigid_fsiBodies_D,
                                    uint* rigidIdentifierD,

                                    Real3* pos_fsi_fea_D,
                                    Real3* vel_fsi_fea_D,
                                    Real3* acc_fsi_fea_D,
                                    uint* FlexIdentifierD,
                                    int numFlex1D,
                                    uint2* CableElementsNodes,
                                    uint4* ShellelementsNodes,

                                    int4 updatePortion,
                                    uint* gridMarkerIndexD,

                                    int numAllMarkers,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w == -1.0) {
        return;
    }
    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    Real h_i = sortedPosRad[i_idx].w;
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real3 myAcc = mR3(0);
    Real3 V_prescribed = mR3(0);
       BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD,
       velMassRigid_fsiBodies_D,
                   accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D,
                   FlexIdentifierD, numFlex1D, CableElementsNodes, ShellelementsNodes);
    Real pRHS1 = 0.0;
    Real pRHS2 = 0.0;

    Real den = 0.0;
    Real3 numV = mR3(0);

    for (int count = csrStartIdx + 1; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        if (sortedRhoPreMu[j].w != -1.0)
            continue;
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real h_j = sortedPosRad[j].w;
        Real h_ij = 0.5 * (h_j + h_i);
        Real W3 = W3h(length(rij), h_ij);
        numV += W3 * sortedVelMas[j];
        den += W3;
        pRHS1 += W3 * sortedRhoPreMu[j].y;
        pRHS2 += dot(paramsD.gravity - myAcc, rij) * sortedRhoPreMu[j].x * W3;
    }

    if (abs(den) > EPSILON) {
        sortedVelMas[i_idx] = 2 * V_prescribed - numV / den;
        sortedRhoPreMu[i_idx].y = (pRHS1 + pRHS2) / den;
    } else {
        sortedVelMas[i_idx] = mR3(0);
        sortedRhoPreMu[i_idx].y = paramsD.BASEPRES;
    }
}

//==========================================================================================================================================
__global__ void Update_Vel_XSPH(Real4* sortedPosRad,
                                Real4* sortedRhoPreMu,
                                Real3* sortedVelMas,

                                const uint* csrColInd,
                                const uint* numContacts,
                                uint numAllMarkers,
                                volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    bool Fluid_Marker = (sortedRhoPreMu[i_idx].w == -1.0);

    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    Real3 deltaV = mR3(0);
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];
    if (Fluid_Marker) {
        Real rho_i = sortedRhoPreMu[i_idx].x;
        Real3 vel_i = sortedVelMas[i_idx];
        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real h_j = sortedPosRad[j].w;
            Real m_j = pow(h_j * paramsD.MULT_INITSPACE, 3) * paramsD.rho0;
            Real h_ij = 0.5 * (h_j + h_i);
            Real W3 = W3h(length(rij), h_ij);
            deltaV += 2.0f * m_j / (rho_i + sortedRhoPreMu[j].x) * (sortedVelMas[j] - vel_i) * W3;
        }
        sortedVelMas[i_idx] += paramsD.EPS_XSPH * deltaV;
    }
}
//==========================================================================================================================================

ChFsiForceXSPH::ChFsiForceXSPH(
    ChBce* otherBceWorker,                   ///< Pointer to the ChBce object that handles BCE markers
    SphMarkerDataD* otherSortedSphMarkersD,  ///< Information of markers in the sorted array on device
    ProximityDataD*
        otherMarkersProximityD,           ///< Pointer to the object that holds the proximity of the markers on device
    FsiGeneralData* otherFsiGeneralData,  ///< Pointer to the sph general data
    SimParams* otherParamsH,              ///< Pointer to the simulation parameters on host
    NumberOfObjects* otherNumObjects      ///< Pointer to number of objects, fluid and boundary markers, etc.
    )
    : ChFsiForce(otherBceWorker,
                 otherSortedSphMarkersD,
                 otherMarkersProximityD,
                 otherFsiGeneralData,
                 otherParamsH,
                 otherNumObjects) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}

ChFsiForceXSPH::~ChFsiForceXSPH() {}

void ChFsiForceXSPH::Finalize() {
    ChFsiForce::Finalize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH, sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH, sizeof(NumberOfObjects));
    hipMemcpyFromSymbol(paramsH, HIP_SYMBOL(paramsD), sizeof(SimParams));
    hipDeviceSynchronize();
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
}
//==========================================================================================================================================
//==========================================================================================================================================
//==========================================================================================================================================
void ChFsiForceXSPH::ForceSPH(SphMarkerDataD* otherSphMarkersD,
                              FsiBodiesDataD* otherFsiBodiesD,
                              FsiMeshDataD* otherFsiMeshD) {
    std::cout << "dT in ForceImplicitSPH: " << paramsH->dT << "\n";
    CopyParams_NumberOfObjects(paramsH, numObjectsH);

    SphMarkerDataD SphMarkerDataD1 = *otherSphMarkersD;

    int numAllMarkers = numObjectsH->numAllMarkers;
    int numHelperMarkers = numObjectsH->numHelperMarkers;

    fsiCollisionSystem->ArrangeData(otherSphMarkersD);
    printf("ForceXSPH numAllMarkers:%d,numHelperMarkers=%d\n", numAllMarkers, numHelperMarkers);

    bool *isErrorH, *isErrorD, *isErrorD2;

    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    hipMalloc((void**)&isErrorD2, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(isErrorD2, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);
    thrust::device_vector<Real> _sumWij_inv;
    thrust::device_vector<uint> Contact_i;
    thrust::device_vector<Real> G_i;
    thrust::device_vector<Real> A_i;
    thrust::device_vector<Real> L_i;
    thrust::device_vector<uint> csrColInd;
    thrust::device_vector<Real> csrValLaplacian;
    thrust::device_vector<Real3> csrValGradient;
    thrust::device_vector<Real> csrValFunciton;
    thrust::device_vector<Real> AMatrix;
    thrust::device_vector<Real3> V_Star;
    thrust::device_vector<Real3> Normals;

    _sumWij_inv.resize(numAllMarkers);
    Normals.resize(numAllMarkers);
    G_i.resize(numAllMarkers * 9);
    A_i.resize(numAllMarkers * 27);
    L_i.resize(numAllMarkers * 6);
    Contact_i.resize(numAllMarkers);
    thrust::fill(Contact_i.begin(), Contact_i.end(), 0);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    thrust::fill(G_i.begin(), G_i.end(), 0);

    thrust::device_vector<Real3> ft(numAllMarkers, mR3(0));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    //============================================================================================================
    // some initialization
    // 1st call
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel");
    //============================================================================================================

    uint LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    int NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);

    //============================================================================================================
    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices");

    //============================================================================================================
    double A_L_Tensor_GradLaplacian = clock();
    printf("calc_A_tensor+");
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor");
    printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor");
    printf("Gradient_Laplacian_Operator: ");
    //============================================================================================================
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator");
    double Gradient_Laplacian_Operator = (clock() - A_L_Tensor_GradLaplacian) / (double)CLOCKS_PER_SEC;
    printf("%f (s)\n", Gradient_Laplacian_Operator);
    int numFlexbodies = +numObjectsH->numFlexBodies1D + numObjectsH->numFlexBodies2D;
    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int4 updatePortion =
        mI4(fsiGeneralData->referenceArray[haveGhost + haveHelper + 0].y,  // end of fluid
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1].y,  // end of boundary
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies].y,
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies + numFlexbodies].y);
    //============================================================================================================
    calculate_pressure<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calculate_pressure-1");
    //============================================================================================================
    Update_Vel_XSPH<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_Vel_XSPH-1");

    // consider boundary condition
    //============================================================================================================
    Boundary_Conditions<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(csrValGradient), R1CAST(csrValLaplacian), U1CAST(csrColInd),
        U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Boundary_Conditions-1");
    //============================================================================================================
    NS_RHS_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(ft), mR3CAST(csrValGradient), R1CAST(csrValLaplacian),
        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "NS_RHS_Predictor-1");
    // 2nd call to calc f and v,x from predicted v and x
    CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);

    fsiCollisionSystem->ArrangeData(otherSphMarkersD);
    //============================================================================================================
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel-2");
    LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);
    //============================================================================================================
    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices-2");
    //============================================================================================================
    printf("calc_A_tensor+");
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor-2");
    printf("calc_L_tensor+");
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor-2");
    printf("Gradient_Laplacian_Operator: ");
    //============================================================================================================
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator-2");
    // //============================================================================================================
    // calculate_pressure<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), numAllMarkers, isErrorD);
    // ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calculate_pressure-2");
    //============================================================================================================
    Update_Vel_XSPH<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update_Vel_XSPH-2");
    // consider boundary condition
    //============================================================================================================
    Boundary_Conditions<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(csrValGradient), R1CAST(csrValLaplacian), U1CAST(csrColInd),
        U1CAST(Contact_i),

        mR4CAST(otherFsiBodiesD->velMassRigid_fsiBodies_D), mR3CAST(otherFsiBodiesD->accRigid_fsiBodies_D),
        U1CAST(fsiGeneralData->rigidIdentifierD),

        mR3CAST(otherFsiMeshD->pos_fsi_fea_D), mR3CAST(otherFsiMeshD->vel_fsi_fea_D),
        mR3CAST(otherFsiMeshD->acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),

        numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
        U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
        numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Boundary_Conditions-2");
    //============================================================================================================
    NS_RHS_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(ft), mR3CAST(csrValGradient), R1CAST(csrValLaplacian),
        U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "NS_RHS_Predictor-2");
    //============================================================================================================
    thrust::device_vector<Real3> ft_unsorted(numAllMarkers);
    thrust::fill(ft_unsorted.begin(), ft_unsorted.end(), mR3(0.0));

    CopySortedToOriginal_NonInvasive_R3(ft_unsorted, ft, markersProximityD->gridMarkerIndexD);
    // otherSphMarkersD->velMasD = SphMarkerDataD1.velMasD;
    // otherSphMarkersD->posRadD = SphMarkerDataD1.posRadD;
    //    otherSphMarkersD->rhoPresMuD = SphMarkerDataD1.rhoPresMuD;
    // CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                     markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    //============================================================================================================
    Update<<<numBlocks, numThreads>>>(mR4CAST(SphMarkerDataD1.posRadD), mR4CAST(SphMarkerDataD1.rhoPresMuD),
                                      mR3CAST(SphMarkerDataD1.velMasD), mR3CAST(otherSphMarkersD->velMasD),
                                      mR4CAST(otherSphMarkersD->posRadD), mR3CAST(ft_unsorted), numAllMarkers,
                                      isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Update");

    thrust::device_vector<Real3>::iterator iter =
        thrust::max_element(otherSphMarkersD->velMasD.begin(), otherSphMarkersD->velMasD.end(), compare_Real3_mag());
    Real MaxVel = length(*iter);

    fsiCollisionSystem->ArrangeData(otherSphMarkersD);
    //3rd call
    //============================================================================================================
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), U1CAST(Contact_i), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel-3");
    LastVal = Contact_i[numAllMarkers - 1];
    thrust::exclusive_scan(Contact_i.begin(), Contact_i.end(), Contact_i.begin());
    Contact_i.push_back(LastVal + Contact_i[numAllMarkers - 1]);
    NNZ = Contact_i[numAllMarkers];
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunciton.resize(NNZ);
    AMatrix.resize(NNZ);
    csrColInd.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunciton.begin(), csrValFunciton.end(), 0.0);
    thrust::fill(csrColInd.begin(), csrColInd.end(), 0.0);
    //============================================================================================================
    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals), U1CAST(csrColInd),
        U1CAST(Contact_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calcNormalizedRho_Gi_fillInMatrixIndices-3");
    printf("calc_A_tensor+");
    //============================================================================================================
    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_A_tensor-3");
    //============================================================================================================
    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "calc_L_tensor-3");
    printf("Gradient_Laplacian_Operator: ");
    //============================================================================================================
    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i), R1CAST(csrValLaplacian),
        mR3CAST(csrValGradient), R1CAST(csrValFunciton), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Gradient_Laplacian_Operator-3");

    //============================================================================================================
    thrust::device_vector<Real3> vel_vis(numAllMarkers);
    thrust::fill(vel_vis.begin(), vel_vis.end(), mR3(0.0));
    Shifting_r<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
                                          mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(vel_vis), R1CAST(csrValFunciton),
                                          mR3CAST(csrValGradient), U1CAST(csrColInd), U1CAST(Contact_i), numAllMarkers,
                                          MaxVel, isErrorD);
    ChDeviceUtils::Sync_CheckError(isErrorH, isErrorD, "Shifting_r");

    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_XSPH_D, vel_vis, markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);
    //    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_IISPH_D, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);
    //    CopySortedToOriginal_NonInvasive_R3(otherSphMarkersD->velMasD, sortedSphMarkersD->velMasD,
    //                                        markersProximityD->gridMarkerIndexD);

    //    CopySortedToOriginal_NonInvasive_R4(otherSphMarkersD->posRadD, sortedSphMarkersD->posRadD,
    //                                        markersProximityD->gridMarkerIndexD);
    //============================================================================================================

    _sumWij_inv.clear();
    Contact_i.clear();
    AMatrix.clear();
    csrColInd.clear();
    G_i.clear();
    A_i.clear();
    L_i.clear();
    csrValLaplacian.clear();
    csrValGradient.clear();
    ft_unsorted.clear();
    ft.clear();

}  // namespace fsi

}  // namespace fsi
}  // namespace chrono
