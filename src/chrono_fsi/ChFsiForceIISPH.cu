#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
//
// Base class for processing sph force in fsi system.//
// =============================================================================
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include "chrono_fsi/ChDeviceUtils.cuh"
#include "chrono_fsi/ChFsiForceIISPH.cuh"
#include "chrono_fsi/ChFsiForceParallel.cuh"
#include "chrono_fsi/ChSphGeneral.cuh"
#include "chrono_fsi/solver6x6.cuh"

//==========================================================================================================================================
namespace chrono {
namespace fsi {

//--------------------------------------------------------------------------------------------------------------------------------
__device__ void grad_scalar(int i_idx,
                            Real4* sortedPosRad,  // input: sorted positions
                            Real4* sortedRhoPreMu,
                            Real* sumWij_inv,
                            Real* G_i,
                            Real4* Scalar,
                            Real3& myGrad,
                            uint* cellStart,
                            uint* cellEnd) {
    // Note that this function only calculates the gradient of the first element of the Scalar;
    // This is hard coded like this for now because usually rho appears in Real4 structure
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    int3 gridPos = calcGridPos(posRadA);

    //    printf("update G[%d]= %f,%f,%f  %f,%f,%f, %f,%f,%f\n", i_idx, G_i[i_idx * 9 + 0], G_i[i_idx * 9 + 1],
    //           G_i[i_idx * 9 + 2], G_i[i_idx * 9 + 3], G_i[i_idx * 9 + 4], G_i[i_idx * 9 + 5], G_i[i_idx * 9 + 6],
    //           G_i[i_idx * 9 + 7], G_i[i_idx * 9 + 8]);

    // This is the elements of inverse of G
    Real mGi[9];
    for (int n = 0; n < 9; n++)
        mGi[n] = G_i[i_idx * 9 + n];

    Real3 grad_si = mR3(0.);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;

                        Real h_j = sortedPosRad[j].w;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real W3 = W3h(d, h_ij);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        Real V_j = sumWij_inv[j];
                        Real3 common_part = mR3(0);
                        common_part.x = grad_i_wij.x * mGi[0] + grad_i_wij.y * mGi[1] + grad_i_wij.z * mGi[2];
                        common_part.y = grad_i_wij.x * mGi[3] + grad_i_wij.y * mGi[4] + grad_i_wij.z * mGi[5];
                        common_part.z = grad_i_wij.x * mGi[6] + grad_i_wij.y * mGi[7] + grad_i_wij.z * mGi[8];
                        grad_si += common_part * (Scalar[j].x - Scalar[i_idx].x) * V_j;
                    }
                }
            }
        }
    }
    myGrad = grad_si;
    //    printf("grad_scalar[%d]= %f,%f,%f\n", i_idx, myGrad.x, myGrad.y, myGrad.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ void grad_vector(int i_idx,
                            Real4* sortedPosRad,  // input: sorted positions
                            Real4* sortedRhoPreMu,
                            Real* sumWij_inv,
                            Real* G_i,
                            Real3* Vector,
                            Real3& myGradx,
                            Real3& myGrady,
                            Real3& myGradz,
                            uint* cellStart,
                            uint* cellEnd) {
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    int3 gridPos = calcGridPos(posRadA);

    // This is the elements of inverse of G
    Real mGi[9];
    for (int n = 0; n < 9; n++)
        mGi[n] = G_i[i_idx * 9 + n];

    Real3 common_part = mR3(0.);
    Real3 grad_Vx = mR3(0.);
    Real3 grad_Vy = mR3(0.);
    Real3 grad_Vz = mR3(0.);

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;

                        Real h_j = sortedPosRad[j].w;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real W3 = W3h(d, h_ij);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        Real V_j = sumWij_inv[j];
                        common_part.x = grad_i_wij.x * mGi[0] + grad_i_wij.y * mGi[1] + grad_i_wij.z * mGi[2];
                        common_part.y = grad_i_wij.x * mGi[3] + grad_i_wij.y * mGi[4] + grad_i_wij.z * mGi[5];
                        common_part.z = grad_i_wij.x * mGi[6] + grad_i_wij.y * mGi[7] + grad_i_wij.z * mGi[8];
                        grad_Vx += common_part * (Vector[i_idx].x - Vector[j].x) * V_j;
                        grad_Vy += common_part * (Vector[i_idx].y - Vector[j].y) * V_j;
                        grad_Vz += common_part * (Vector[i_idx].z - Vector[j].z) * V_j;
                    }
                }
            }
        }
    }
    myGradx = grad_Vx;
    myGrady = grad_Vy;
    myGradz = grad_Vz;
    //    printf("grad_vector[%d]= %f,%f,%f  %f,%f,%f, %f,%f,%f\n", i_idx, myGradx.x, myGradx.y, myGradx.z, myGrady.x,
    //           myGrady.y, myGrady.z, myGradz.x, myGradz.y, myGradz.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calc_A_tensor(Real* A_tensor,
                              Real* G_tensor,
                              Real4* sortedPosRad,
                              Real4* sortedRhoPreMu,
                              Real* sumWij_inv,
                              uint* cellStart,
                              uint* cellEnd,
                              const int numAllMarkers,
                              volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    // Remember : we want to solve 6x6 system Bi*l=-[1 0 0 1 0 1]'
    // elements of matrix B depends on tensor A
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;
    Real sum_mW = 0;
    Real A_ijk[27] = {0.0};

    Real Gi[9] = {0.0};
    for (int i = 0; i < 9; i++)
        Gi[i] = G_tensor[i_idx * 9 + i];

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posRadA, posRadB);
                        Real d = length(rij);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_ij = GradWh(rij, h_ij);
                        Real V_j = sumWij_inv[j];
                        Real com_part = 0;
                        com_part = (Gi[0] * grad_ij.x + Gi[1] * grad_ij.y + Gi[2] * grad_ij.z) * V_j;
                        A_ijk[0] += rij.x * rij.x * com_part;  // 111
                        A_ijk[1] += rij.x * rij.y * com_part;  // 112
                        A_ijk[2] += rij.x * rij.z * com_part;  // 113
                        A_ijk[3] += rij.y * rij.x * com_part;  // 121
                        A_ijk[4] += rij.y * rij.y * com_part;  // 122
                        A_ijk[5] += rij.y * rij.z * com_part;  // 123
                        A_ijk[6] += rij.z * rij.x * com_part;  // 131
                        A_ijk[7] += rij.z * rij.y * com_part;  // 132
                        A_ijk[8] += rij.z * rij.z * com_part;  // 133
                        com_part = (Gi[3] * grad_ij.x + Gi[4] * grad_ij.y + Gi[5] * grad_ij.z) * V_j;
                        A_ijk[9] += rij.x * rij.x * com_part;   // 211
                        A_ijk[10] += rij.x * rij.y * com_part;  // 212
                        A_ijk[11] += rij.x * rij.z * com_part;  // 213
                        A_ijk[12] += rij.y * rij.x * com_part;  // 221
                        A_ijk[13] += rij.y * rij.y * com_part;  // 222
                        A_ijk[14] += rij.y * rij.z * com_part;  // 223
                        A_ijk[15] += rij.z * rij.x * com_part;  // 231
                        A_ijk[16] += rij.z * rij.y * com_part;  // 232
                        A_ijk[17] += rij.z * rij.z * com_part;  // 233
                        com_part = (Gi[6] * grad_ij.x + Gi[7] * grad_ij.y + Gi[8] * grad_ij.z) * V_j;
                        A_ijk[18] += rij.x * rij.x * com_part;  // 311
                        A_ijk[19] += rij.x * rij.y * com_part;  // 312
                        A_ijk[20] += rij.x * rij.z * com_part;  // 313
                        A_ijk[21] += rij.y * rij.x * com_part;  // 321
                        A_ijk[22] += rij.y * rij.y * com_part;  // 322
                        A_ijk[23] += rij.y * rij.z * com_part;  // 323
                        A_ijk[24] += rij.z * rij.x * com_part;  // 331
                        A_ijk[25] += rij.z * rij.y * com_part;  // 332
                        A_ijk[26] += rij.z * rij.z * com_part;  // 333
                    }
                }
            }
        }
    }

    for (int i = 0; i < 27; i++)
        A_tensor[i_idx * 9 + i] = A_ijk[i];

    //    printf("A_tensor[%d]= %f,%f,%f,%f,%f,%f,%f,%f,%f, %f,%f,%f,%f,%f,%f,%f,%f,%f, %f,%f,%f,%f,%f,%f,%f,%f,%f\n",
    //    i_idx,
    //           A_ijk[0], A_ijk[1], A_ijk[2], A_ijk[3], A_ijk[4], A_ijk[5], A_ijk[6], A_ijk[7], A_ijk[8], A_ijk[9 + 0],
    //           A_ijk[9 + 1], A_ijk[9 + 2], A_ijk[9 + 3], A_ijk[9 + 4], A_ijk[9 + 5], A_ijk[9 + 6], A_ijk[9 + 7],
    //           A_ijk[9 + 8], A_ijk[18 + 0], A_ijk[18 + 1], A_ijk[18 + 2], A_ijk[18 + 3], A_ijk[18 + 4], A_ijk[18 + 5],
    //           A_ijk[18 + 6], A_ijk[18 + 7], A_ijk[18 + 8]);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calc_L_tensor(Real* A_tensor,
                              Real* L_tensor,
                              Real* G_tensor,
                              Real4* sortedPosRad,
                              Real4* sortedRhoPreMu,
                              Real* sumWij_inv,
                              uint* cellStart,
                              uint* cellEnd,
                              const int numAllMarkers,
                              volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    if (sortedRhoPreMu[i_idx].w != -1) {
        return;
    }

    // Remember : we want to solve 6x6 system Bi*l=-[1 0 0 1 0 1]'
    // elements of matrix B depends on tensor A
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;
    Real B[36] = {0.0};

    Real Gi[9] = {0.0};
    for (int i = 0; i < 9; i++)
        Gi[i] = G_tensor[i_idx * 9 + i];

    Real A_ijk[27] = {0.0};
    for (int i = 0; i < 27; i++)
        A_ijk[i] = A_tensor[i_idx * 27 + i];

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posRadA, posRadB);
                        Real d = length(rij);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real3 eij = rij / d;

                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_ij = GradWh(rij, h_ij);
                        Real V_j = sumWij_inv[j];
                        Real com_part = 0;
                        // mn=11

                        Real XX = (eij.x * grad_ij.x);
                        Real XY = (eij.x * grad_ij.y + eij.y * grad_ij.x);
                        Real XZ = (eij.x * grad_ij.z + eij.z * grad_ij.x);
                        Real YY = (eij.y * grad_ij.y);
                        Real YZ = (eij.y * grad_ij.z + eij.z * grad_ij.y);
                        Real ZZ = (eij.z * grad_ij.z);

                        com_part = (A_ijk[0] * eij.x + A_ijk[9] * eij.y + A_ijk[18] * eij.z + rij.x * eij.x) * V_j;
                        B[6 * 0 + 0] += com_part * XX;  // 11
                        B[6 * 0 + 1] += com_part * XY;  // 12
                        B[6 * 0 + 2] += com_part * XZ;  // 13
                        B[6 * 0 + 3] += com_part * YY;  // 14
                        B[6 * 0 + 4] += com_part * YZ;  // 15
                        B[6 * 0 + 5] += com_part * ZZ;  // 15
                        // mn=12
                        com_part = (A_ijk[1] * eij.x + A_ijk[10] * eij.y + A_ijk[19] * eij.z + rij.x * eij.y) * V_j;
                        B[6 * 1 + 0] += com_part * XX;  // 21
                        B[6 * 1 + 1] += com_part * XY;  // 22
                        B[6 * 1 + 2] += com_part * XZ;  // 23
                        B[6 * 1 + 3] += com_part * YY;  // 24
                        B[6 * 1 + 4] += com_part * YZ;  // 25
                        B[6 * 1 + 5] += com_part * ZZ;  // 25

                        // mn=13
                        com_part = (A_ijk[2] * eij.x + A_ijk[11] * eij.y + A_ijk[20] * eij.z + rij.x * eij.z) * V_j;
                        B[6 * 2 + 0] += com_part * XX;  // 31
                        B[6 * 2 + 1] += com_part * XY;  // 32
                        B[6 * 2 + 2] += com_part * XZ;  // 33
                        B[6 * 2 + 3] += com_part * YY;  // 34
                        B[6 * 2 + 4] += com_part * YZ;  // 35
                        B[6 * 2 + 5] += com_part * ZZ;  // 36

                        // Note that we skip mn=21 since it is similar to mn=12
                        // mn=22
                        com_part = (A_ijk[4] * eij.x + A_ijk[13] * eij.y + A_ijk[22] * eij.z + rij.y * eij.y) * V_j;
                        B[6 * 3 + 0] += com_part * XX;  // 41
                        B[6 * 3 + 1] += com_part * XY;  // 42
                        B[6 * 3 + 2] += com_part * XZ;  // 43
                        B[6 * 3 + 3] += com_part * YY;  // 44
                        B[6 * 3 + 4] += com_part * YZ;  // 45
                        B[6 * 3 + 5] += com_part * ZZ;  // 46

                        // mn=23
                        com_part = (A_ijk[5] * eij.x + A_ijk[14] * eij.y + A_ijk[23] * eij.z + rij.y * eij.z) * V_j;
                        B[6 * 4 + 0] += com_part * XX;  // 51
                        B[6 * 4 + 1] += com_part * XY;  // 52
                        B[6 * 4 + 2] += com_part * XZ;  // 53
                        B[6 * 4 + 3] += com_part * YY;  // 54
                        B[6 * 4 + 4] += com_part * YZ;  // 55
                        B[6 * 4 + 5] += com_part * ZZ;  // 56
                        // mn=33
                        com_part = (A_ijk[8] * eij.x + A_ijk[17] * eij.y + A_ijk[26] * eij.z + rij.z * eij.z) * V_j;
                        B[6 * 5 + 0] += com_part * XX;  // 61
                        B[6 * 5 + 1] += com_part * XY;  // 62
                        B[6 * 5 + 2] += com_part * XZ;  // 63
                        B[6 * 5 + 3] += com_part * YY;  // 64
                        B[6 * 5 + 4] += com_part * YZ;  // 65
                        B[6 * 5 + 5] += com_part * ZZ;  // 66
                    }
                }
            }
        }
    }

    inv6xdelta_mn(B, &L_tensor[6 * i_idx]);
    //    printf("L[%d]=%f,%f,%f,%f,%f,%f\n", i_idx, L_tensor[6 * i_idx + 0], L_tensor[6 * i_idx + 1],
    //           L_tensor[6 * i_idx + 2], L_tensor[6 * i_idx + 3], L_tensor[6 * i_idx + 4], L_tensor[6 * i_idx + 5]);
    //    for (uint j = 0; j < 6; j++)
    //        printf("B[%d,[%d]]=%f,%f,%f,%f,%f,%f\n", i_idx, j, B[6 * j + 0], B[6 * j + 1], B[6 * j + 2], B[6 * j + 3],
    //               B[6 * j + 4], B[6 * j + 5]);

    //    printf(
    //        "B[%d]=%f,%f,%f,%f,%f,%f,%f,%f,%f, %f,%f,%f,%f,%f,%f,%f,%f,%f, %f,%f,%f,%f,%f,%f,%f,%f,%f, "
    //        "%f,%f,%f,%f,%f,%f,%f,%f,%f, --- %f,%f,%f,%f,%f,%f\n",
    //        i_idx, B[0], B[1], B[2], B[3], B[4], B[5], B[6], B[7], B[8], B[9], B[10], B[11], B[12], B[13], B[14],
    //        B[15], B[16], B[17], B[18], B[19], B[20], B[21], B[22], B[23], B[24], B[25], B[26], B[27], B[28],
    //        B[29], B[30], B[31], B[32], B[33], 1, B[35], L_tensor[6 * i_idx + 0], L_tensor[6 * i_idx + 1],
    //        L_tensor[6 * i_idx
    //        + 2], L_tensor[6 * i_idx + 3], L_tensor[6 * i_idx + 4], L_tensor[6 * i_idx + 5]);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calcRho_kernel(Real4* sortedPosRad,  // input: sorted positionsmin(
                               Real4* sortedRhoPreMu,
                               Real* sumWij_inv,
                               uint* cellStart,
                               uint* cellEnd,
                               const int numAllMarkers,
                               volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;
    Real sum_mW = 0;
    Real sum_W = 0.0;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real W3 = W3h(d, 0.5 * (h_j + h_i));
                        //                        Real W3 = 0.5 * (W3h(d, h_i) + W3h(d, h_j));
                        sum_mW += m_j * W3;
                        sum_W += W3;
                    }
                }
            }
        }
    }

    // Adding neighbor contribution is done!
    sumWij_inv[i_idx] = m_i / sum_mW;
    sortedRhoPreMu[i_idx].x = sum_mW;

    //    if (sumWij_inv[i_idx] > 1e-5 && sortedRhoPreMu[i_idx].w > -2)
    //        printf("sum_mW=%f,  sumWij_inv[i_idx]=%.4e\n", sum_mW, sumWij_inv[i_idx]);
    //
    //    //    sortedRhoPreMu[i_idx].x = sum_mW;
    //
    //    if ((sortedRhoPreMu[i_idx].x > 2 * paramsD.rho0 || sortedRhoPreMu[i_idx].x < 0) && sortedRhoPreMu[i_idx].w
    //    == -1)
    //        printf("(calcRho_kernel)too large/small density marker %d, rho=%f\n", i_idx, sortedRhoPreMu[i_idx].x);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calcNormalizedRho_kernel(Real4* sortedPosRad,  // input: sorted positions
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* sumWij_inv,
                                         Real* G_i,
                                         Real* dxi_over_Vi,
                                         Real* Color,
                                         uint* cellStart,
                                         uint* cellEnd,
                                         const int numAllMarkers,
                                         volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers || sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }
    //    Real3 gravity = paramsD.gravity;
    Real RHO_0 = paramsD.rho0;
    Real IncompressibilityFactor = paramsD.IncompressibilityFactor;
    //    dxi_over_Vi[i_idx] = 1e10;

    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;
    Real sum_mW = 0;
    Real sum_W_sumWij_inv = 0;
    Real C = 0;
    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    // This is the elements of inverse of G
    Real mGi[9] = {0.0};

    //  /// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);
    //
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                                                 // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real3 dv3 = Distance(sortedVelMas[i_idx], sortedVelMas[j]);
                        Real d = length(dist3);
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        C += m_j * Color[i_idx] / sortedRhoPreMu[i_idx].x * W3h(d, 0.5 * (h_j + h_i));
                        //                        Real particle_particle_n_CFL = abs(dot(dv3, dist3)) / d;
                        //                        Real particle_particle = length(dv3);
                        //                        Real particle_n_CFL = abs(dot(sortedVelMas[i_idx], dist3)) / d;
                        //                        Real particle_CFL = length(sortedVelMas[i_idx]);

                        //                        if (i_idx != j)
                        //                            dxi_over_Vi[i_idx] = fminf(d / particle_CFL,
                        //                            dxi_over_Vi[i_idx]);

                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real W3 = W3h(d, h_ij);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        Real V_j = sumWij_inv[j];
                        mGi[0] -= dist3.x * grad_i_wij.x * V_j;
                        mGi[1] -= dist3.x * grad_i_wij.y * V_j;
                        mGi[2] -= dist3.x * grad_i_wij.z * V_j;
                        mGi[3] -= dist3.y * grad_i_wij.x * V_j;
                        mGi[4] -= dist3.y * grad_i_wij.y * V_j;
                        mGi[5] -= dist3.y * grad_i_wij.z * V_j;
                        mGi[6] -= dist3.z * grad_i_wij.x * V_j;
                        mGi[7] -= dist3.z * grad_i_wij.y * V_j;
                        mGi[8] -= dist3.z * grad_i_wij.z * V_j;
                        sum_mW += m_j * W3;
                        sum_W_sumWij_inv += sumWij_inv[j] * W3;
                    }
                }
            }
        }
    }

    Real Det = (mGi[0] * mGi[4] * mGi[8] - mGi[0] * mGi[5] * mGi[7] - mGi[1] * mGi[3] * mGi[8] +
                mGi[1] * mGi[5] * mGi[6] + mGi[2] * mGi[3] * mGi[7] - mGi[2] * mGi[4] * mGi[6]);
    G_i[i_idx * 9 + 0] = (mGi[4] * mGi[8] - mGi[5] * mGi[7]) / Det;
    G_i[i_idx * 9 + 1] = -(mGi[1] * mGi[8] - mGi[2] * mGi[7]) / Det;
    G_i[i_idx * 9 + 2] = (mGi[1] * mGi[5] - mGi[2] * mGi[4]) / Det;
    G_i[i_idx * 9 + 3] = -(mGi[3] * mGi[8] - mGi[5] * mGi[6]) / Det;
    G_i[i_idx * 9 + 4] = (mGi[0] * mGi[8] - mGi[2] * mGi[6]) / Det;
    G_i[i_idx * 9 + 5] = -(mGi[0] * mGi[5] - mGi[2] * mGi[3]) / Det;
    G_i[i_idx * 9 + 6] = (mGi[3] * mGi[7] - mGi[4] * mGi[6]) / Det;
    G_i[i_idx * 9 + 7] = -(mGi[0] * mGi[7] - mGi[1] * mGi[6]) / Det;
    G_i[i_idx * 9 + 8] = (mGi[0] * mGi[4] - mGi[1] * mGi[3]) / Det;

    //    printf("update G[%d]= %f,%f,%f  %f,%f,%f, %f,%f,%f\n", i_idx, G_i[i_idx * 9 + 0], G_i[i_idx * 9 + 1],
    //           G_i[i_idx * 9 + 2], G_i[i_idx * 9 + 3], G_i[i_idx * 9 + 4], G_i[i_idx * 9 + 5], G_i[i_idx * 9 + 6],
    //           G_i[i_idx * 9 + 7], G_i[i_idx * 9 + 8]);

    if (sortedRhoPreMu[i_idx].x > RHO_0)
        IncompressibilityFactor = 1;

    sortedRhoPreMu[i_idx].x = (sum_mW / sum_W_sumWij_inv - RHO_0) * IncompressibilityFactor + RHO_0;

    //    if (sortedRhoPreMu[i_idx].x < EPSILON)
    if ((sortedRhoPreMu[i_idx].x > 5 * RHO_0 || sortedRhoPreMu[i_idx].x < RHO_0 / 5) && sortedRhoPreMu[i_idx].w > -2)
        printf(
            "calcNormalizedRho_kernel-- sortedRhoPreMu[i_idx].w=%f, h=%f, sum_mW=%f, "
            "sum_W_sumWij_inv=%.4e, sortedRhoPreMu[i_idx].x=%.4e\n",
            sortedRhoPreMu[i_idx].w, sortedPosRad[i_idx].w, sum_mW, sum_W_sumWij_inv, sortedRhoPreMu[i_idx].x);

    //    sortedRhoPreMu[i_idx].x = (sum_mW - RHO_0) * IncompressibilityFactor + RHO_0;
    //
    //    if (sortedRhoPreMu[i_idx].x < EPSILON) {
    //        printf("My density is %f, index= %d\n", sortedRhoPreMu[i_idx].x, i_idx);
    //
    //        printf("My position = [%f %f %f]\n", sortedPosRad[i_idx].x, sortedPosRad[i_idx].y,
    //        sortedPosRad[i_idx].z);
    //
    //        *isErrorD = true;
    //        return;
    //    }

    //  if (sortedRhoPreMu[i_idx].w > -1) {
    //    sortedRhoPreMu[i_idx].x = RHO_0;
    //  }
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void V_i_np__AND__d_ii_kernel(Real4* sortedPosRad,  // input: sorted positions
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real3* d_ii,
                                         Real3* V_i_np,
                                         Real* sumWij_inv,
                                         Real* G_tensor,
                                         Real* L_tensor,
                                         uint* cellStart,
                                         uint* cellEnd,
                                         const int numAllMarkers,
                                         volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers || sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }
    //    sortedRhoPreMu[i_idx].x = sortedRhoPreMu[i_idx].x / sumWij_inv[i_idx];
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real mu_0 = paramsD.mu0;
    Real epsilon = paramsD.epsMinMarkersDis;
    Real dT = paramsD.dT;
    Real3 gravity = paramsD.gravity;
    Real RHO_0 = paramsD.rho0;
    if (sortedRhoPreMu[i_idx].x < EPSILON) {
        printf("My density is %f,ref density= %f\n", sortedRhoPreMu[i_idx].x, RHO_0);
    }

    Real3 posi = mR3(sortedPosRad[i_idx]);
    Real3 Veli = sortedVelMas[i_idx];
    Real Rhoi = sortedRhoPreMu[i_idx].x;
    Real3 My_d_ii = mR3(0);
    Real3 My_F_i_np = mR3(0);

    Real3 myGradvx = mR3(0);
    Real3 myGradvy = mR3(0);
    Real3 myGradvz = mR3(0);
    Real Gi[9] = {0.0};
    Real Li[6] = {0.0};
    Real3 LaplacainVi = mR3(0.0);
    for (int i = 0; i < 9; i++)
        Gi[i] = G_tensor[i_idx * 9 + i];
    for (int i = 0; i < 6; i++)
        Li[i] = L_tensor[i_idx * 6 + i];

    grad_vector(i_idx, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_tensor, sortedVelMas, myGradvx, myGradvy, myGradvz,
                cellStart, cellEnd);

    // get address in grid
    int3 gridPos = calcGridPos(posi);

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posj = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posi, posj);
                        Real d = length(rij);

                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        Real3 eij = rij / d;

                        Real3 Velj = sortedVelMas[j];
                        Real Rhoj = sortedRhoPreMu[j].x;
                        Real h_j = sortedPosRad[j].w;

                        if (Rhoj == 0) {
                            printf("Bug F_i_np__AND__d_ii_kernel i=%d j=%d, hi=%f, hj=%f\n", i_idx, j, h_i, h_j);
                        }

                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_ij = GradWh(rij, h_ij);
                        My_d_ii += m_j * (-(dT * dT) / (Rhoi * Rhoi)) * grad_ij;

                        Real Rho_bar = (Rhoj + Rhoi) * 0.5;
                        Real3 V_ij = (Veli - Velj);
                        //                        Real nu = mu_0 * paramsD.HSML * 320 / Rho_bar;
                        //            Real3 muNumerator = nu * fmin(0.0, dot(rij, V_ij)) * grad_i_wij;
                        Real3 muNumerator = 2 * mu_0 * dot(rij, grad_ij) * V_ij;
                        Real muDenominator = (Rho_bar * Rho_bar) * (d * d + h_ij * h_ij * epsilon);
                        My_F_i_np += m_j * muNumerator / muDenominator;

                        Real Wd = W3h(d, h_ij);
                        My_F_i_np -= paramsD.kappa / m_i * m_j * Wd * rij;

                        Real Vj = sumWij_inv[j];
                        Real commonterm =
                            2 * Vj *
                            (Li[0] * eij.x * grad_ij.x + Li[1] * eij.x * grad_ij.y + Li[2] * eij.x * grad_ij.z +
                             Li[1] * eij.y * grad_ij.x + Li[3] * eij.y * grad_ij.y + Li[4] * eij.y * grad_ij.z +
                             Li[2] * eij.z * grad_ij.x + Li[4] * eij.z * grad_ij.y + Li[5] * eij.z * grad_ij.z);

                        LaplacainVi.x += commonterm * (V_ij.x / d - dot(eij, myGradvx));
                        LaplacainVi.y += commonterm * (V_ij.y / d - dot(eij, myGradvy));
                        LaplacainVi.z += commonterm * (V_ij.y / d - dot(eij, myGradvz));
                    }
                }
            }
        }
    }

    My_F_i_np = mu_0 * LaplacainVi;

    My_F_i_np *= m_i;

    My_F_i_np += m_i * gravity;
    d_ii[i_idx] = My_d_ii;
    V_i_np[i_idx] = (My_F_i_np * dT + Veli);  // This does not contain m_0?
}
//--------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Rho_np_AND_a_ii_AND_sum_m_GradW(Real4* sortedPosRad,
                                                Real4* sortedRhoPreMu,
                                                Real* rho_np,  // Write
                                                Real* a_ii,    // Write
                                                Real* p_old,   // Write
                                                Real3* V_np,   // Read
                                                Real3* d_ii,   // Read
                                                Real3* sum_m_GradW,
                                                uint* cellStart,
                                                uint* cellEnd,
                                                const int numAllMarkers,
                                                volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers || sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real3 posi = mR3(sortedPosRad[i_idx]);
    Real3 Veli_np = V_np[i_idx];
    Real Rho_i = sortedRhoPreMu[i_idx].x;
    Real3 my_d_ii = d_ii[i_idx];
    Real rho_temp = 0;
    Real my_a_ii = 0;
    Real3 My_sum_m_gradW = mR3(0);
    Real dT = paramsD.dT;
    // get address in gridj
    int3 gridPos = calcGridPos(posi);

    //
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posj = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posi, posj);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 Velj_np = V_np[j];
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        rho_temp += m_j * dot((Veli_np - Velj_np), grad_i_wij);
                        Real3 d_ji = m_i * (-(dT * dT) / (Rho_i * Rho_i)) * (-grad_i_wij);
                        my_a_ii += m_j * dot((my_d_ii - d_ji), grad_i_wij);
                        My_sum_m_gradW += m_j * grad_i_wij;
                    }
                }
            }
        }
    }
    rho_np[i_idx] = dT * rho_temp + sortedRhoPreMu[i_idx].x;

    a_ii[i_idx] = my_a_ii;
    sum_m_GradW[i_idx] = My_sum_m_gradW;

    p_old[i_idx] = sortedRhoPreMu[i_idx].y;  // = 1000;  // Note that this is outside of the for loop
}
//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Calc_dij_pj(Real3* dij_pj,  // write
                            Real3* F_p,     // Write
                            Real3* d_ii,    // Read
                            Real4* sortedPosRad,
                            Real3* sortedVelMas,
                            Real4* sortedRhoPreMu,
                            Real* p_old,
                            uint* cellStart,
                            uint* cellEnd,
                            const int numAllMarkers,
                            volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers || sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real3 my_F_p = mR3(0);
    Real p_i_old = p_old[i_idx];
    Real3 pos_i = mR3(sortedPosRad[i_idx]);
    Real Rho_i = sortedRhoPreMu[i_idx].x;
    if (sortedRhoPreMu[i_idx].x < EPSILON) {
        printf("(Calc_dij_pj) My density is %f in Calc_dij_pj\n", sortedRhoPreMu[i_idx].x);
    }
    Real dT = paramsD.dT;

    Real3 My_dij_pj = mR3(0);
    int3 gridPos = calcGridPos(pos_i);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 pos_j = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(pos_i, pos_j);
                        Real d = length(dist3);
                        ////CHECK THIS CONDITION!!!
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        Real Rho_j = sortedRhoPreMu[j].x;
                        Real p_j_old = p_old[j];
                        My_dij_pj += m_j * (-(dT * dT) / (Rho_j * Rho_j)) * grad_i_wij * p_j_old;
                        my_F_p += m_j * ((p_i_old / (Rho_i * Rho_i)) + (p_j_old / (Rho_j * Rho_j))) * grad_i_wij;
                    }
                }
            }
        }
    }
    dij_pj[i_idx] = My_dij_pj;
    F_p[i_idx] = -m_i * my_F_p;
}
////--------------------------------------------------------------------------------------------------------------------------------

__device__ void BCE_Vel_Acc(int i_idx,
                            Real3& myAcc,
                            Real3& V_prescribed,

                            Real4* sortedPosRad,
                            int4 updatePortion,
                            uint* gridMarkerIndexD,

                            Real4* velMassRigid_fsiBodies_D,
                            Real3* accRigid_fsiBodies_D,
                            uint* rigidIdentifierD,

                            Real3* pos_fsi_fea_D,
                            Real3* vel_fsi_fea_D,
                            Real3* acc_fsi_fea_D,
                            uint* FlexIdentifierD,
                            const int numFlex1D,
                            uint2* CableElementsNodes,
                            uint4* ShellelementsNodes) {
    int Original_idx = gridMarkerIndexD[i_idx];

    //  Real MASS;

    // See if this belongs to boundary
    if (Original_idx >= updatePortion.x && Original_idx < updatePortion.y) {
        myAcc = mR3(0.0);
        V_prescribed = mR3(0.0);
        if (paramsD.Apply_BC_U)
            V_prescribed = user_BC_U(mR3(sortedPosRad[i_idx]));

    } else if (Original_idx >= updatePortion.y && Original_idx < updatePortion.z) {
        int rigidIndex = rigidIdentifierD[Original_idx - updatePortion.y];
        V_prescribed = mR3(velMassRigid_fsiBodies_D[rigidIndex].x, velMassRigid_fsiBodies_D[rigidIndex].y,
                           velMassRigid_fsiBodies_D[rigidIndex].z);
        myAcc = mR3(accRigid_fsiBodies_D[rigidIndex].x, accRigid_fsiBodies_D[rigidIndex].y,
                    accRigid_fsiBodies_D[rigidIndex].z);

        // Or not, Flexible bodies for sure
    } else if (Original_idx >= updatePortion.z && Original_idx < updatePortion.w) {
        int FlexIndex = FlexIdentifierD[Original_idx - updatePortion.z];

        if (FlexIndex < numFlex1D) {
            int nA = CableElementsNodes[FlexIndex].y;
            int nB = CableElementsNodes[FlexIndex].x;

            Real3 pos_fsi_fea_D_nA = pos_fsi_fea_D[nA];
            Real3 pos_fsi_fea_D_nB = pos_fsi_fea_D[nB];

            Real3 vel_fsi_fea_D_nA = vel_fsi_fea_D[nA];
            Real3 vel_fsi_fea_D_nB = vel_fsi_fea_D[nB];

            Real3 acc_fsi_fea_D_nA = acc_fsi_fea_D[nA];
            Real3 acc_fsi_fea_D_nB = acc_fsi_fea_D[nB];

            Real3 dist3 = mR3(sortedPosRad[Original_idx]) - pos_fsi_fea_D_nA;
            Real3 x_dir = (pos_fsi_fea_D_nB - pos_fsi_fea_D_nA);
            Real Cable_x = length(x_dir);
            x_dir = x_dir / length(x_dir);
            Real dx = dot(dist3, x_dir);

            Real2 N_shell = Cables_ShapeFunctions(dx / Cable_x);
            Real NA = N_shell.x;
            Real NB = N_shell.y;

            V_prescribed = NA * vel_fsi_fea_D_nA + NB * vel_fsi_fea_D_nB;
            myAcc = NA * acc_fsi_fea_D_nA + NB * acc_fsi_fea_D_nB;
        }
        if (FlexIndex >= numFlex1D) {
            int nA = ShellelementsNodes[FlexIndex - numFlex1D].x;
            int nB = ShellelementsNodes[FlexIndex - numFlex1D].y;
            int nC = ShellelementsNodes[FlexIndex - numFlex1D].z;
            int nD = ShellelementsNodes[FlexIndex - numFlex1D].w;

            Real3 pos_fsi_fea_D_nA = pos_fsi_fea_D[nA];
            Real3 pos_fsi_fea_D_nB = pos_fsi_fea_D[nB];
            Real3 pos_fsi_fea_D_nC = pos_fsi_fea_D[nC];
            Real3 pos_fsi_fea_D_nD = pos_fsi_fea_D[nD];

            Real3 vel_fsi_fea_D_nA = vel_fsi_fea_D[nA];
            Real3 vel_fsi_fea_D_nB = vel_fsi_fea_D[nB];
            Real3 vel_fsi_fea_D_nC = vel_fsi_fea_D[nC];
            Real3 vel_fsi_fea_D_nD = vel_fsi_fea_D[nD];

            Real3 acc_fsi_fea_D_nA = acc_fsi_fea_D[nA];
            Real3 acc_fsi_fea_D_nB = acc_fsi_fea_D[nB];
            Real3 acc_fsi_fea_D_nC = acc_fsi_fea_D[nC];
            Real3 acc_fsi_fea_D_nD = acc_fsi_fea_D[nD];

            Real3 Shell_center = 0.25 * (pos_fsi_fea_D_nA + pos_fsi_fea_D_nB + pos_fsi_fea_D_nC + pos_fsi_fea_D_nD);
            Real3 dist3 = mR3(sortedPosRad[Original_idx]) - Shell_center;
            Real Shell_x =
                0.25 * (length(pos_fsi_fea_D_nB - pos_fsi_fea_D_nA) + length(pos_fsi_fea_D_nD - pos_fsi_fea_D_nC));
            Real Shell_y =
                0.25 * (length(pos_fsi_fea_D_nD - pos_fsi_fea_D_nA) + length(pos_fsi_fea_D_nC - pos_fsi_fea_D_nB));
            Real2 FlexSPH_MeshPos_Natural = mR2(dist3.x / Shell_x, dist3.y / Shell_y);

            Real4 N_shell = Shells_ShapeFunctions(FlexSPH_MeshPos_Natural.x, FlexSPH_MeshPos_Natural.y);
            Real NA = N_shell.x;
            Real NB = N_shell.y;
            Real NC = N_shell.z;
            Real ND = N_shell.w;
            V_prescribed =
                NA * vel_fsi_fea_D_nA + NB * vel_fsi_fea_D_nB + NC * vel_fsi_fea_D_nC + ND * vel_fsi_fea_D_nD;
            myAcc = NA * acc_fsi_fea_D_nA + NB * acc_fsi_fea_D_nB + NC * acc_fsi_fea_D_nC + ND * acc_fsi_fea_D_nD;
        }
    } else {
        printf("i_idx=%d, Original_idx:%d was not found \n\n", i_idx, Original_idx);
    }
}
////--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcNumber_Contacts(uint* numContacts,
                                    Real4* sortedPosRad,
                                    Real4* sortedRhoPreMu,
                                    uint* cellStart,
                                    uint* cellEnd,
                                    const int numAllMarkers,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        numContacts[i_idx] = 1;
        return;
    }

    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    int myType = sortedRhoPreMu[i_idx].w;
    Real3 pos_i = mR3(sortedPosRad[i_idx]);

    uint numCol[800];
    int counter = 1;
    numCol[0] = i_idx;  // The first one is always the idx of the marker itself

    int3 gridPos = calcGridPos(pos_i);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {
                    // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 pos_j = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(pos_i, pos_j);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        bool AlreadyHave = false;
                        for (uint findCol = 1; findCol <= counter; findCol++) {
                            if (numCol[findCol] == j) {
                                AlreadyHave = true;
                                continue;
                            }
                        }

                        // Room for improvment ...
                        if (!AlreadyHave) {
                            numCol[counter] = j;
                            counter++;
                            if (myType >= 0 && sortedRhoPreMu[j].w >= 0)  // Do not count BCE-BCE interactions...
                                counter--;
                        }

                        if (myType != -1)  // For BCE no need to go deeper than this...
                            continue;

                        Real h_j = sortedPosRad[j].w;
                        int3 gridPosJ = calcGridPos(pos_j);

                        for (int zz = -1; zz <= 1; zz++) {
                            for (int yy = -1; yy <= 1; yy++) {
                                for (int xx = -1; xx <= 1; xx++) {
                                    int3 neighbourPosJ = gridPosJ + mI3(xx, yy, zz);
                                    uint gridHashJ = calcGridHash(neighbourPosJ);
                                    uint startIndexJ = cellStart[gridHashJ];
                                    if (startIndexJ != 0xffffffff) {  // cell is not empty
                                        uint endIndexJ = cellEnd[gridHashJ];
                                        for (uint k = startIndexJ; k < endIndexJ; k++) {
                                            Real3 pos_k = mR3(sortedPosRad[k]);
                                            Real3 dist3jk = Distance(pos_j, pos_k);
                                            Real djk = length(dist3jk);
                                            if (djk > RESOLUTION_LENGTH_MULT * h_j || k == j || k == i_idx ||
                                                sortedRhoPreMu[k].w <= -2)
                                                continue;
                                            bool AlreadyHave2 = false;
                                            for (uint findCol = 1; findCol <= counter; findCol++) {
                                                if (numCol[findCol] == k) {
                                                    AlreadyHave2 = true;
                                                    continue;
                                                }
                                            }
                                            if (!AlreadyHave2) {
                                                numCol[counter] = k;
                                                counter++;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                        ///////////////////////////////
                    }
                }
            }
        }
    }

    numContacts[i_idx] = counter + 10;
}

////--------------------------------------------------------------------------------------------------------------------------------

__global__ void Calc_summGradW(Real3* summGradW,  // write
                               Real4* sortedPosRad,
                               Real4* sortedRhoPreMu,
                               uint* cellStart,
                               uint* cellEnd,
                               const int numAllMarkers,
                               volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }
    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real3 pos_i = mR3(sortedPosRad[i_idx]);
    Real3 My_summgradW = mR3(0);
    //    Real dT = paramsD.dT;
    int3 gridPos = calcGridPos(pos_i);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 pos_j = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(pos_i, pos_j);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        My_summgradW += m_j * grad_i_wij;
                    }
                }
            }
        }
    }
    summGradW[i_idx] = My_summgradW;
}

////--------------------------------------------------------------------------------------------------------------------------------
__device__ void Calc_BC_aij_Bi(const uint i_idx,
                               Real* csrValA,
                               uint* csrColIndA,
                               unsigned long int* GlobalcsrColIndA,
                               uint* numContacts,
                               ///> The above 4 vectors are used for CSR form.
                               Real* a_ii,  // write
                               Real* B_i,
                               Real4* sortedPosRad,
                               Real3* sortedVelMas,
                               const Real4* sortedRhoPreMu,
                               Real3* V_new,
                               Real* p_old,

                               Real4* velMassRigid_fsiBodies_D,
                               Real3* accRigid_fsiBodies_D,
                               uint* rigidIdentifierD,

                               Real3* pos_fsi_fea_D,
                               Real3* vel_fsi_fea_D,
                               Real3* acc_fsi_fea_D,
                               uint* FlexIdentifierD,
                               const int numFlex1D,
                               uint2* CableElementsNodes,
                               uint4* ShellelementsNodes,

                               int4 updatePortion,
                               uint* gridMarkerIndexD,
                               const uint* cellStart,
                               const uint* cellEnd,
                               const int numAllMarkers,
                               bool IsSPARSE) {
    uint csrStartIdx = numContacts[i_idx] + 1;
    uint csrEndIdx = numContacts[i_idx + 1];

    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real3 gravity = paramsD.gravity;
    //  if (bceIndex >= numObjectsD.numRigid_SphMarkers) {
    //    return;
    //  }

    //  int Original_idx = gridMarkerIndexD[i_idx];
    Real3 myAcc;
    Real3 V_prescribed;

    if (!(sortedRhoPreMu[i_idx].w >= 0 && sortedRhoPreMu[i_idx].w <= 2))
        printf("type of marker is %f\n", sortedRhoPreMu[i_idx].w);

    BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D, FlexIdentifierD,
                numFlex1D, CableElementsNodes, ShellelementsNodes);

    for (int c = csrStartIdx; c < csrEndIdx; c++) {
        csrValA[c] = 0;
        csrColIndA[c] = i_idx;
        GlobalcsrColIndA[c] = i_idx + numAllMarkers * i_idx;
    }

    // if ((csrEndIdx - csrStartIdx) != uint(0)) {
    Real3 numeratorv = mR3(0);
    Real denumenator = 0;
    Real pRHS = 0;
    //  Real Rho_i = sortedRhoPreMu[i_idx].x;
    Real3 pos_i = mR3(sortedPosRad[i_idx]);
    // get address in grid
    int3 gridPos = calcGridPos(pos_i);

    uint counter = 0;
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 pos_j = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(pos_i, pos_j);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || sortedRhoPreMu[j].w != -1)
                            continue;
                        Real3 Vel_j = sortedVelMas[j];
                        Real h_j = sortedPosRad[j].w;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real Wd = W3h(d, h_ij);
                        numeratorv += Vel_j * Wd;
                        pRHS += dot(gravity - myAcc, dist3) * sortedRhoPreMu[j].x * Wd;
                        denumenator += Wd;
                        csrValA[counter + csrStartIdx] = -Wd;
                        csrColIndA[counter + csrStartIdx] = j;
                        GlobalcsrColIndA[counter + csrStartIdx] = j + numAllMarkers * i_idx;
                        counter++;
                    }
                }
            }
        }
    }

    //  if (abs(Scaling) < EPSILON)
    //    Scaling = -paramsD.HSML;

    if (abs(denumenator) < EPSILON) {
        V_new[i_idx] = 2 * V_prescribed;
        B_i[i_idx] = 0;

        csrValA[csrStartIdx - 1] = a_ii[i_idx];
        csrColIndA[csrStartIdx - 1] = i_idx;
        GlobalcsrColIndA[csrStartIdx - 1] = i_idx + numAllMarkers * i_idx;
    } else {
        Real Scaling = a_ii[i_idx] / denumenator;

        V_new[i_idx] = 2 * V_prescribed - numeratorv / denumenator;
        B_i[i_idx] = pRHS;
        csrValA[csrStartIdx - 1] = denumenator;
        csrColIndA[csrStartIdx - 1] = i_idx;
        GlobalcsrColIndA[csrStartIdx - 1] = i_idx + numAllMarkers * i_idx;

        for (int i = csrStartIdx - 1; i < csrEndIdx; i++)
            csrValA[i] *= Scaling;
        B_i[i_idx] *= Scaling;
    }

    sortedVelMas[i_idx] = V_new[i_idx];
}
//--------------------------------------------------------------------------------------------------------------------------------

////--------------------------------------------------------------------------------------------------------------------------------
__device__ void Calc_fluid_aij_Bi(const uint i_idx,
                                  Real* csrValA,
                                  uint* csrColIndA,
                                  unsigned long int* GlobalcsrColIndA,
                                  uint* numContacts,
                                  ///> The above 4 vectors are used for CSR form.
                                  Real* B_i,
                                  Real3* d_ii,   // Read
                                  Real* a_ii,    // Read
                                  Real* rho_np,  // Read
                                  Real3* summGradW,
                                  Real4* sortedPosRad,
                                  Real4* sortedRhoPreMu,
                                  uint* cellStart,
                                  uint* cellEnd,
                                  const int numAllMarkers,
                                  bool IsSPARSE) {
    Real3 pos_i = mR3(sortedPosRad[i_idx]);
    Real dT = paramsD.dT;

    int counter = 0;  // There is always one non-zero at each row- The marker itself
    B_i[i_idx] = paramsD.rho0 - rho_np[i_idx];

    uint csrStartIdx = numContacts[i_idx] + 1;  // Reserve the starting index for the A_ii
    uint csrEndIdx = numContacts[i_idx + 1];

    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    //  for (int c = csrStartIdx; c < csrEndIdx; c++) {
    //    csrValA[c] = a_ii[i_idx];
    //    csrColIndA[c] = i_idx;
    //    GlobalcsrColIndA[c] = i_idx + numAllMarkers * i_idx;
    //  }

    int3 gridPos = calcGridPos(pos_i);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    // iterate over particles in this cell
                    uint endIndex = cellEnd[gridHash];
                    //          Real Rho_i = sortedRhoPreMu[i_idx].x;

                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 pos_j = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(pos_i, pos_j);
                        Real d = length(dist3);
                        if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_i_wij = GradWh(dist3, h_ij);
                        Real Rho_j = sortedRhoPreMu[j].x;
                        Real m_j = h_j * h_j * h_j * paramsD.rho0;
                        Real3 d_it = m_j * (-(dT * dT) / (Rho_j * Rho_j)) * grad_i_wij;
                        Real My_a_ij_1 = m_j * dot(d_it, summGradW[i_idx]);
                        Real My_a_ij_2 = m_j * dot(d_ii[j], grad_i_wij);
                        Real My_a_ij_12 = My_a_ij_1 - My_a_ij_2;
                        bool DONE1 = false;

                        for (uint findCol = csrStartIdx; findCol < csrEndIdx; findCol++) {
                            if (csrColIndA[findCol] == j) {
                                csrValA[findCol] += My_a_ij_12;
                                csrColIndA[findCol] = j;
                                GlobalcsrColIndA[findCol] = j + numAllMarkers * i_idx;
                                DONE1 = true;
                                continue;
                            }
                        }
                        if (!DONE1) {
                            csrValA[counter + csrStartIdx] += My_a_ij_12;
                            csrColIndA[counter + csrStartIdx] = j;
                            GlobalcsrColIndA[counter + csrStartIdx] = j + numAllMarkers * i_idx;
                            counter++;
                        }
                        int3 gridPosJ = calcGridPos(pos_j);
                        for (int zz = -1; zz <= 1; zz++) {
                            for (int yy = -1; yy <= 1; yy++) {
                                for (int xx = -1; xx <= 1; xx++) {
                                    int3 neighbourPosJ = gridPosJ + mI3(xx, yy, zz);
                                    uint gridHashJ = calcGridHash(neighbourPosJ);
                                    uint startIndexJ = cellStart[gridHashJ];
                                    if (startIndexJ != 0xffffffff) {  // cell is not empty
                                        uint endIndexJ = cellEnd[gridHashJ];
                                        for (uint k = startIndexJ; k < endIndexJ; k++) {
                                            Real3 pos_k = mR3(sortedPosRad[k]);
                                            Real3 dist3jk = Distance(pos_j, pos_k);
                                            Real djk = length(dist3jk);
                                            if (djk > RESOLUTION_LENGTH_MULT * h_j || k == j || k == i_idx ||
                                                sortedRhoPreMu[k].w <= -2)
                                                continue;
                                            Real h_k = sortedPosRad[j].w;
                                            Real h_jk = 0.5 * (h_j + h_k);
                                            Real3 grad_j_wjk = GradWh(dist3jk, h_jk);
                                            Real m_k = pow(sortedPosRad[k].w, 3) * paramsD.rho0;
                                            Real Rho_k = sortedRhoPreMu[k].x;
                                            Real3 d_jk = m_k * (-(dT * dT) / (Rho_k * Rho_k)) * grad_j_wjk;
                                            Real My_a_ij_3 = m_j * dot(d_jk, grad_i_wij);
                                            bool DONE2 = false;

                                            for (uint findCol = csrStartIdx; findCol < csrEndIdx; findCol++) {
                                                if (csrColIndA[findCol] == k) {
                                                    csrValA[findCol] -= My_a_ij_3;
                                                    csrColIndA[findCol] = k;
                                                    GlobalcsrColIndA[findCol] = k + numAllMarkers * i_idx;
                                                    DONE2 = true;
                                                    continue;
                                                }
                                            }
                                            if (!DONE2) {
                                                csrValA[counter + csrStartIdx] -= My_a_ij_3;
                                                csrColIndA[counter + csrStartIdx] = k;
                                                GlobalcsrColIndA[counter + csrStartIdx] = k + numAllMarkers * i_idx;
                                                counter++;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                        ///////////////////////////////
                    }
                }
            }
        }
    }

    for (int myIdx = csrStartIdx; myIdx < csrEndIdx; myIdx++) {
        if (csrColIndA[myIdx] == i_idx)
            csrValA[myIdx] = a_ii[i_idx];
    }

    csrValA[csrStartIdx - 1] = a_ii[i_idx];
    csrColIndA[csrStartIdx - 1] = i_idx;
    GlobalcsrColIndA[csrStartIdx - 1] = i_idx + numAllMarkers * i_idx;

    if (sortedRhoPreMu[i_idx].x < 0.999 * paramsD.rho0) {
        csrValA[csrStartIdx - 1] = a_ii[i_idx];
        for (int myIdx = csrStartIdx; myIdx < csrEndIdx; myIdx++) {
            csrValA[myIdx] = 0.0;
            B_i[i_idx] = 0.0;
        }
    }

    Real RHS = B_i[i_idx];
    B_i[i_idx] = RHS;  // fminf(0.0, RHS);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void FormAXB(Real* csrValA,
                        uint* csrColIndA,
                        unsigned long int* GlobalcsrColIndA,

                        uint* numContacts,
                        ///> The above 4 vectors are used for CSR form.
                        Real* a_ij,   // write
                        Real* B_i,    // write
                        Real3* d_ii,  // Read
                        Real* a_ii,   // Read
                        Real3* summGradW,
                        Real4* sortedPosRad,
                        Real3* sortedVelMas,
                        Real4* sortedRhoPreMu,
                        Real3* V_new,
                        Real* p_old,
                        Real* rho_np,

                        Real4* velMassRigid_fsiBodies_D,
                        Real3* accRigid_fsiBodies_D,
                        uint* rigidIdentifierD,

                        Real3* pos_fsi_fea_D,
                        Real3* vel_fsi_fea_D,
                        Real3* acc_fsi_fea_D,
                        uint* FlexIdentifierD,
                        const int numFlex1D,
                        uint2* CableElementsNodes,
                        uint4* ShellelementsNodes,

                        int4 updatePortion,
                        uint* gridMarkerIndexD,
                        uint* cellStart,
                        uint* cellEnd,
                        const int numAllMarkers,
                        bool IsSPARSE,
                        volatile bool* isError) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    //    Real m_0 = paramsD.markerMass;
    //    Real RHO_0 = paramsD.rho0;
    //    Real dT = paramsD.dT;
    //    Real3 gravity = paramsD.gravity;

    int TYPE_OF_NARKER = sortedRhoPreMu[i_idx].w;

    if (TYPE_OF_NARKER <= -2) {
        B_i[i_idx] = 0;
        uint csrStartIdx = numContacts[i_idx];
        // This needs to be check to see if it messes up the condition number of the matrix
        csrValA[csrStartIdx] = 1.0;
        csrColIndA[csrStartIdx] = i_idx;
        GlobalcsrColIndA[csrStartIdx] = i_idx + numAllMarkers * i_idx;
    } else if (TYPE_OF_NARKER == -1) {
        Calc_fluid_aij_Bi(i_idx, csrValA, csrColIndA, GlobalcsrColIndA, numContacts, B_i, d_ii, a_ii, rho_np, summGradW,
                          sortedPosRad, sortedRhoPreMu, cellStart, cellEnd, numAllMarkers, true);

    } else if (TYPE_OF_NARKER > -1)
        Calc_BC_aij_Bi(i_idx, csrValA, csrColIndA, GlobalcsrColIndA, numContacts, a_ii, B_i, sortedPosRad, sortedVelMas,

                       sortedRhoPreMu, V_new, p_old,

                       velMassRigid_fsiBodies_D, accRigid_fsiBodies_D, rigidIdentifierD,

                       pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D, FlexIdentifierD, numFlex1D, CableElementsNodes,
                       ShellelementsNodes,

                       updatePortion, gridMarkerIndexD, cellStart, cellEnd, numAllMarkers, true);
}

//--------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Calc_Pressure_AXB_USING_CSR(Real* csrValA,
                                            Real* a_ii,
                                            uint* csrColIndA,
                                            uint* numContacts,
                                            Real4* sortedRhoPreMu,
                                            Real* sumWij_inv,
                                            Real3* sortedVelMas,
                                            Real3* V_new,
                                            Real* p_old,
                                            Real* B_i,  // Read
                                            const int numAllMarkers,
                                            volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    //    Real RHO_0 = paramsD.rho0;
    //    bool ClampPressure = paramsD.ClampPressure;
    //    Real Max_Pressure = paramsD.Max_Pressure;
    uint startIdx = numContacts[i_idx] + 1;  // numContacts[i_idx] is the diagonal itself
    uint endIdx = numContacts[i_idx + 1];

    Real aij_pj = 0;
    //  Real error = aij_pj + sortedRhoPreMu[i_idx].y * csrValA[startIdx - 1] - B_i[i_idx];

    for (int myIdx = startIdx; myIdx < endIdx; myIdx++) {
        if (csrColIndA[myIdx] != i_idx)
            aij_pj += csrValA[myIdx] * p_old[csrColIndA[myIdx]];
    }
    Real RHS = B_i[i_idx];
    sortedRhoPreMu[i_idx].y = (RHS - aij_pj) / csrValA[startIdx - 1];

    //  sortedRhoPreMu[i_idx].y = (ClampPressure && sortedRhoPreMu[i_idx].y < 0.0) ? 0.0 : sortedRhoPreMu[i_idx].y;

    if (!isfinite(aij_pj)) {
        printf("a_ij *p_j became Nan in Calc_Pressure_AXB_USING_CSR ");
    }
}

////--------------------------------------------------------------------------------------------------------------------------------
__global__ void Calc_Pressure(Real* a_ii,     // Read
                              Real3* d_ii,    // Read
                              Real3* dij_pj,  // Read
                              Real* rho_np,   // Read
                              Real* rho_p,    // Write
                              Real3* F_p,
                              Real4* sortedPosRad,
                              Real3* sortedVelMas,
                              Real4* sortedRhoPreMu,

                              Real4* velMassRigid_fsiBodies_D,
                              Real3* accRigid_fsiBodies_D,
                              uint* rigidIdentifierD,

                              Real3* pos_fsi_fea_D,
                              Real3* vel_fsi_fea_D,
                              Real3* acc_fsi_fea_D,
                              uint* FlexIdentifierD,
                              const int numFlex1D,
                              uint2* CableElementsNodes,
                              uint4* ShellelementsNodes,

                              int4 updatePortion,
                              uint* gridMarkerIndexD,

                              Real* p_old,
                              Real3* V_new,
                              uint* cellStart,
                              uint* cellEnd,
                              const int numAllMarkers,
                              volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real RHO_0 = paramsD.rho0;
    Real dT = paramsD.dT;
    Real3 gravity = paramsD.gravity;
    bool ClampPressure = paramsD.ClampPressure;

    if (sortedRhoPreMu[i_idx].x < EPSILON) {
        printf("(Calc_Pressure)My density is %f in Calc_Pressure\n", sortedRhoPreMu[i_idx].x);
    }
    int myType = sortedRhoPreMu[i_idx].w;
    Real Rho_i = sortedRhoPreMu[i_idx].x;
    Real p_i = p_old[i_idx];
    Real3 pos_i = mR3(sortedPosRad[i_idx]);
    Real p_new = 0;
    Real my_rho_p = 0;
    Real3 F_i_p = F_p[i_idx];

    if (myType == -1) {
        if (Rho_i < 0.95 * RHO_0) {
            p_new = 0;
        } else {
            Real3 my_dij_pj = dij_pj[i_idx];
            Real sum_dij_pj = 0;  // This is the first summation  term in the expression for the pressure.
            Real sum_djj_pj = 0;  // This is the second summation term in the expression for the pressure.
            Real sum_djk_pk = 0;  // This is the last summation term in the expression for the pressure.
            int3 gridPosI = calcGridPos(pos_i);
            for (int z = -1; z <= 1; z++) {
                for (int y = -1; y <= 1; y++) {
                    for (int x = -1; x <= 1; x++) {
                        int3 neighbourPosI = gridPosI + mI3(x, y, z);
                        uint gridHashI = calcGridHash(neighbourPosI);
                        // get start of bucket for this cell
                        uint startIndexI = cellStart[gridHashI];
                        if (startIndexI != 0xffffffff) {
                            uint endIndexI = cellEnd[gridHashI];
                            for (uint j = startIndexI; j < endIndexI; j++) {
                                Real3 pos_j = mR3(sortedPosRad[j]);
                                Real3 dist3ij = Distance(pos_i, pos_j);
                                Real dij = length(dist3ij);
                                if (dij > RESOLUTION_LENGTH_MULT * paramsD.HSML || i_idx == j ||
                                    sortedRhoPreMu[j].w <= -2)
                                    continue;
                                //                Real Rho_j = sortedRhoPreMu[j].x;
                                Real p_j_old = p_old[j];
                                Real h_j = sortedPosRad[j].w;
                                Real m_j = h_j * h_j * h_j * paramsD.rho0;

                                Real3 djj = d_ii[j];
                                Real3 F_j_p = F_p[j];

                                Real h_ij = 0.5 * (h_j + h_i);
                                Real3 grad_i_wij = GradWh(dist3ij, h_ij);
                                Real3 d_ji = m_i * (-(dT * dT) / (Rho_i * Rho_i)) * (-grad_i_wij);
                                Real3 djk_pk = dij_pj[j] - d_ji * p_i;
                                sum_dij_pj += m_j * dot(my_dij_pj, grad_i_wij);
                                sum_djj_pj += m_j * dot(djj, grad_i_wij) * p_j_old;
                                sum_djk_pk += m_j * dot(djk_pk, grad_i_wij);
                                my_rho_p += (dT * dT) * m_j * dot((F_i_p / m_i - F_j_p / m_j), grad_i_wij);
                            }
                        }
                    }
                }
            }

            Real RHS = fminf(0.0, RHO_0 - rho_np[i_idx]);
            //      Real RHS = RHO_0 - rho_np[i_idx];

            Real aij_pj = +sum_dij_pj - sum_djj_pj - sum_djk_pk;
            p_new = (RHS - aij_pj) / a_ii[i_idx];
            //      sortedRhoPreMu[i_idx].x = aij_pj + p_new * a_ii[i_idx] + RHO_0 - RHS;
        }
    } else {  // Do Adami BC

        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);
        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, velMassRigid_fsiBodies_D,
                    accRigid_fsiBodies_D, rigidIdentifierD, pos_fsi_fea_D, vel_fsi_fea_D, acc_fsi_fea_D,
                    FlexIdentifierD, numFlex1D, CableElementsNodes, ShellelementsNodes);

        Real3 numeratorv = mR3(0);
        Real denumenator = 0;
        Real numeratorp = 0;
        Real3 Vel_i;

        // get address in grid
        int3 gridPos = calcGridPos(pos_i);
        for (int z = -1; z <= 1; z++) {
            for (int y = -1; y <= 1; y++) {
                for (int x = -1; x <= 1; x++) {
                    int3 neighbourPos = gridPos + mI3(x, y, z);
                    uint gridHash = calcGridHash(neighbourPos);
                    // get start of bucket for this cell
                    uint startIndex = cellStart[gridHash];
                    if (startIndex != 0xffffffff) {  // cell is not empty
                        uint endIndex = cellEnd[gridHash];
                        for (uint j = startIndex; j < endIndex; j++) {
                            Real3 pos_j = mR3(sortedPosRad[j]);
                            Real3 dist3 = Distance(pos_i, pos_j);
                            Real d = length(dist3);
                            if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML || sortedRhoPreMu[j].w != -1)
                                continue;
                            // OLD VELOCITY IS SHOULD BE OBDATED NOT THE NEW ONE!!!!!
                            Real3 Vel_j = sortedVelMas[j];
                            Real p_j = p_old[j];
                            Real3 F_j_p = F_p[j];
                            Real h_j = sortedPosRad[j].w;
                            Real m_j = h_j * h_j * h_j * paramsD.rho0;
                            Real h_ij = 0.5 * (h_j + h_i);
                            Real Wd = W3h(d, h_ij);
                            numeratorv += Vel_j * Wd;
                            numeratorp += p_j * Wd + dot(gravity - myAcc, dist3) * sortedRhoPreMu[j].x * Wd;
                            denumenator += Wd;
                            Real3 TobeUsed = (F_i_p / m_i - F_j_p / m_j);
                            my_rho_p += (dT * dT) * m_j * dot(TobeUsed, GradWh(dist3, h_ij));

                            if (isnan(numeratorp))
                                printf("Something is wrong here..., %f\n", numeratorp);
                        }
                    }
                }
            }
        }
        if (abs(denumenator) < EPSILON) {
            p_new = 0;
            Vel_i = 2 * V_prescribed;
        } else {
            Vel_i = 2 * V_prescribed - numeratorv / denumenator;
            p_new = numeratorp / denumenator;
        }

        V_new[i_idx] = Vel_i;
    }
    if (ClampPressure && p_new < 0.0)
        p_new = 0.0;
    rho_p[i_idx] = my_rho_p;
    sortedRhoPreMu[i_idx].y = p_new;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Initialize_Variables(Real4* sortedRhoPreMu,
                                     Real* p_old,
                                     Real3* sortedVelMas,
                                     Real3* V_new,
                                     const int numAllMarkers,
                                     volatile bool* isErrorD) {
    const uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    p_old[i_idx] = sortedRhoPreMu[i_idx].y;  // This needs consistency p_old is old but v_new is new !!
    if (sortedRhoPreMu[i_idx].w > -1) {
        sortedVelMas[i_idx] = V_new[i_idx];
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Update_AND_Calc_Res(Real3* sortedVelMas,
                                    Real4* sortedRhoPreMu,
                                    Real* p_old,
                                    Real3* V_new,
                                    Real* rho_p,
                                    Real* rho_np,
                                    Real* Residuals,
                                    const int numAllMarkers,
                                    const int Iteration,
                                    Real params_relaxation,
                                    bool IsSPARSE,
                                    volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }

    //  p_i = (1 - relax) * p_old_i + relax * p_i;
    sortedRhoPreMu[i_idx].y = (1 - params_relaxation) * p_old[i_idx] + params_relaxation * sortedRhoPreMu[i_idx].y;
    //  Real AbsRes = abs(sortedRhoPreMu[i_idx].y - p_old[i_idx]);

    //  Real Updated_rho = rho_np[i_idx] + rho_p[i_idx];
    //  Real rho_res = abs(1000 - sortedRhoPreMu[i_idx].x);  // Hard-coded for now
    Real p_res = 0;
    //  p_res = abs(sortedRhoPreMu[i_idx].y - p_old[i_idx]) / (abs(p_old[i_idx]) + 0.00001);
    p_res = abs(sortedRhoPreMu[i_idx].y - p_old[i_idx]);

    Residuals[i_idx] = p_res;
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcForces(Real3* new_vel,  // Write
                           Real4* derivVelRhoD,
                           Real4* sortedPosRad,  // Read
                           Real3* sortedVelMas,  // Read
                           Real4* sortedRhoPreMu,
                           Real* sumWij_inv,
                           Real* G_tensor,
                           Real* L_tensor,
                           Real3* r_shift,
                           uint* cellStart,
                           uint* cellEnd,
                           uint numAllMarkers,
                           volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        sortedRhoPreMu[i_idx].x = 0;
        sortedRhoPreMu[i_idx].y = 0;
        sortedRhoPreMu[i_idx].z = 0;
        return;
    }

    if (sortedRhoPreMu[i_idx].w > -1) {
        return;
    }

    Real mu_0 = paramsD.mu0;
    Real h_i = sortedPosRad[i_idx].w;
    Real m_i = h_i * h_i * h_i * paramsD.rho0;

    Real dT = paramsD.dT;
    Real3 gravity = paramsD.gravity;
    Real epsilon = paramsD.epsMinMarkersDis;
    Real3 posi = mR3(sortedPosRad[i_idx]);
    Real3 Veli = sortedVelMas[i_idx];
    Real p_i = sortedRhoPreMu[i_idx].y;
    Real rho_i = sortedRhoPreMu[i_idx].x;
    Real3 F_i_mu = mR3(0);
    Real3 F_i_surface_tension = mR3(0);
    Real3 F_i_p = mR3(0);
    if ((sortedRhoPreMu[i_idx].x > 3 * paramsD.rho0 || sortedRhoPreMu[i_idx].x < 0) && sortedRhoPreMu[i_idx].w < 0)
        printf("too large/small density marker %d, type=%f\n", i_idx, sortedRhoPreMu[i_idx].w);

    Real r0 = 0;
    int Ni = 0;
    Real mi_bar = 0;
    Real3 inner_sum = mR3(0);

    int3 gridPos = calcGridPos(posi);
    Real3 myGradvx = mR3(0);
    Real3 myGradvy = mR3(0);
    Real3 myGradvz = mR3(0);
    Real Gi[9] = {0.0};
    Real Li[6] = {0.0};
    Real3 LaplacainVi = mR3(0.0);
    for (int i = 0; i < 9; i++)
        Gi[i] = G_tensor[i_idx * 9 + i];
    for (int i = 0; i < 6; i++)
        Li[i] = L_tensor[i_idx * 6 + i];

    grad_vector(i_idx, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_tensor, sortedVelMas, myGradvx, myGradvy, myGradvz,
                cellStart, cellEnd);

    // get address in grid
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    Real3 posj = mR3(sortedPosRad[j]);
                    Real3 rij = Distance(posi, posj);
                    Real d = length(rij);
                    if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 || i_idx == j)
                        continue;
                    Real3 eij = rij / d;
                    Real h_j = sortedPosRad[j].w;
                    Real m_j = h_j * h_j * h_j * paramsD.rho0;

                    mi_bar += m_j;
                    Ni++;
                    r0 += d;
                    inner_sum += m_j * rij / (d * d * d);

                    Real h_ij = 0.5 * (h_j + h_i);
                    Real Wd = m_j * W3h(d, h_ij);
                    Real3 grad_ij = GradWh(rij, h_ij);

                    Real3 Velj = sortedVelMas[j];
                    Real p_j = sortedRhoPreMu[j].y;
                    Real rho_j = sortedRhoPreMu[j].x;

                    Real3 V_ij = (Veli - Velj);
                    // Only Consider (fluid-fluid + fluid-solid) or Solid-Fluid Interaction
                    if (sortedRhoPreMu[i_idx].w < 0 || (sortedRhoPreMu[i_idx].w > 0 && sortedRhoPreMu[j].w < 0))
                        F_i_p += -m_j * ((p_i / (rho_i * rho_i)) + (p_j / (rho_j * rho_j))) * grad_ij;

                    Real Rho_bar = (rho_j + rho_i) * 0.5;
                    //                    Real nu = mu_0 * paramsD.HSML * 320 / Rho_bar;
                    //          Real3 muNumerator = nu * fminf(0.0, dot(rij, V_ij)) * grad_ij;
                    Real3 muNumerator = 2 * mu_0 * dot(rij, grad_ij) * V_ij;

                    Real muDenominator = (Rho_bar * Rho_bar) * (d * d + paramsD.HSML * paramsD.HSML * epsilon);
                    // Only Consider (fluid-fluid + fluid-solid) or Solid-Fluid Interaction
                    if (sortedRhoPreMu[i_idx].w < 0 || (sortedRhoPreMu[i_idx].w > 0 && sortedRhoPreMu[j].w < 0))
                        F_i_mu += m_j * muNumerator / muDenominator;
                    if (!isfinite(length(F_i_mu))) {
                        printf("F_i_np in CalcForces returns Nan or Inf");
                    }

                    F_i_surface_tension += m_j * Wd * rij;

                    Real Vj = sumWij_inv[j];
                    Real commonterm =
                        2 * Vj *
                        (Li[0] * eij.x * grad_ij.x + Li[1] * eij.x * grad_ij.y + Li[2] * eij.x * grad_ij.z +
                         Li[1] * eij.y * grad_ij.x + Li[3] * eij.y * grad_ij.y + Li[4] * eij.y * grad_ij.z +
                         Li[2] * eij.z * grad_ij.x + Li[4] * eij.z * grad_ij.y + Li[5] * eij.z * grad_ij.z);

                    LaplacainVi.x += commonterm * (V_ij.x / d - dot(eij, myGradvx));
                    LaplacainVi.y += commonterm * (V_ij.y / d - dot(eij, myGradvy));
                    LaplacainVi.z += commonterm * (V_ij.y / d - dot(eij, myGradvz));

                    if (!isfinite(length(LaplacainVi))) {
                        printf("LaplacainVi in CalcForces returns Nan or Inf");
                    }
                }
            }
        }
    }

    r0 /= Ni;
    r_shift[i_idx] = 0.5 * r0 * r0 * paramsD.v_Max * dT / mi_bar * inner_sum;

    F_i_surface_tension = -F_i_surface_tension * paramsD.kappa / m_i;
    // Forces are per unit mass at this point.
    //    derivVelRhoD[i_idx] = mR4((F_i_p + F_i_mu + F_i_surface_tension) * m_i);
    derivVelRhoD[i_idx] = mR4((F_i_p + mu_0 * LaplacainVi + F_i_surface_tension) * m_i);

    // Add the gravity forces only to the fluid markers
    if ((int)sortedRhoPreMu[i_idx].w == -1)
        derivVelRhoD[i_idx] = derivVelRhoD[i_idx] + mR4(gravity) * m_i;

    new_vel[i_idx] = Veli + dT * mR3(derivVelRhoD[i_idx]) / m_i + r_shift[i_idx] / dT;
    sortedVelMas[i_idx] = new_vel[i_idx];
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void UpdateDensity(Real3* vis_vel,
                              Real3* new_vel,       // Write
                              Real4* sortedPosRad,  // Read
                              Real4* sortedRhoPreMu,
                              Real* sumWij_inv,
                              uint* cellStart,
                              uint* cellEnd,
                              uint numAllMarkers,
                              volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        sortedRhoPreMu[i_idx].x = 0;
        sortedRhoPreMu[i_idx].y = 0;
        sortedRhoPreMu[i_idx].z = 0;
        return;
    }
    Real dT = paramsD.dT;
    Real rho_plus = 0;
    Real3 Vel_i = new_vel[i_idx];
    Real3 posi = mR3(sortedPosRad[i_idx]);
    if ((sortedRhoPreMu[i_idx].x > 2 * paramsD.rho0 || sortedRhoPreMu[i_idx].x < 0) && sortedRhoPreMu[i_idx].w < 0)
        printf("(UpdateDensity-0)too large/small density marker %d, type=%f\n", i_idx, sortedRhoPreMu[i_idx].w);
    Real h_i = sortedPosRad[i_idx].w;
    //    Real m_i = h_i * h_i * h_i * paramsD.rho0;
    int3 gridPos = calcGridPos(posi);

    Real3 normalizedV_n = mR3(0);
    Real normalizedV_d = 0.0;

    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    Real3 posj = mR3(sortedPosRad[j]);
                    Real3 dist3 = Distance(posi, posj);
                    Real d = length(dist3);
                    if (d > RESOLUTION_LENGTH_MULT * h_i || sortedRhoPreMu[j].w <= -2 ||
                        (sortedRhoPreMu[i_idx].w > 0 && sortedRhoPreMu[j].w > 0))
                        continue;
                    Real3 Vel_j = new_vel[j];
                    Real h_j = sortedPosRad[j].w;
                    Real m_j = h_j * h_j * h_j * paramsD.rho0;
                    Real h_ij = 0.5 * (h_j + h_i);
                    Real3 grad_i_wij = GradWh(dist3, h_ij);
                    rho_plus += m_j * dot((Vel_i - Vel_j), grad_i_wij) * sumWij_inv[j];
                    Real Wd = W3h(d, h_ij);
                    if (sortedRhoPreMu[j].w == -1) {
                        normalizedV_n += Vel_j * Wd * m_j / sortedRhoPreMu[j].x;
                        normalizedV_d += Wd * m_j / sortedRhoPreMu[j].x;
                    }
                }
            }
        }
    }
    if (normalizedV_d > EPSILON && sortedRhoPreMu[i_idx].w == -1)
        vis_vel[i_idx] = normalizedV_n / normalizedV_d;
    sortedRhoPreMu[i_idx].x += rho_plus * dT;
    if ((sortedRhoPreMu[i_idx].x > 2 * paramsD.rho0 || sortedRhoPreMu[i_idx].x < 0) && sortedRhoPreMu[i_idx].w < 0)
        printf("(UpdateDensity-1)too large/small density marker %d, type=%f\n", i_idx, sortedRhoPreMu[i_idx].w);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void FinalizePressure(Real4* sortedPosRad,  // Read
                                 Real4* sortedRhoPreMu,
                                 Real* p_old,
                                 Real3* F_p,  // Write
                                 uint* cellStart,
                                 uint* cellEnd,
                                 uint numAllMarkers,
                                 Real p_shift,
                                 volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[i_idx].w <= -2) {
        return;
    }
    //  Real m_0 = paramsD.markerMass;
    //  Real3 posi = sortedPosRad[i_idx];
    //  Real Rho_i = sortedRhoPreMu[i_idx].x;
    //  Real3 my_F_p = mR3(0);
    //  Real p_i = sortedRhoPreMu[i_idx].y;
    //  Real sum_pW = 0;
    //  Real sum_W = 0;

    // if (p_old[i_idx] > 0)
    // sortedRhoPreMu[i_idx].y = p_old[i_idx];
    // else
    // sortedRhoPreMu[i_idx].y = 0;

    //  if (p_shift < 0)
    sortedRhoPreMu[i_idx].y = p_old[i_idx] + ((paramsD.ClampPressure) ? paramsD.BASEPRES : 0.0);  //- p_shift;

    //  if (p_old[i_idx] < 0)
    //    sortedRhoPreMu[i_idx].y = (p_old[i_idx] > 0) ? p_old[i_idx] : 0.0;

    //  if (sortedRhoPreMu[i_idx].y > paramsD.Max_Pressure)
    //    sortedRhoPreMu[i_idx].y = paramsD.Max_Pressure;

    //  // get address in grid
    //  int3 gridPos = calcGridPos(posi);
    //  for (int z = -1; z <= 1; z++) {
    //    for (int y = -1; y <= 1; y++) {
    //      for (int x = -1; x <= 1; x++) {
    //        int3 neighbourPos = gridPos + mI3(x, y, z);
    //        uint gridHash = calcGridHash(neighbourPos);
    //        // get start of bucket for this cell
    //        uint startIndex = cellStart[gridHash];
    //        uint endIndex = cellEnd[gridHash];
    //
    //        for (uint j = startIndex; j < endIndex; j++) {
    //          Real3 posJ = sortedPosRad[j];
    //          Real3 dist3 = Distance(posi, posJ);
    //          Real d = length(dist3);
    //          if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML || i_idx == j)
    //            continue;
    //          Real3 grad_i_wij = GradW(dist3);
    //          Real p_j = p_old[j];
    //          Real Rho_j = sortedRhoPreMu[j].x;
    //          my_F_p += -m_0 * m_0 * ((p_old[i_idx] / (Rho_i * Rho_i)) + (p_old[j] / (Rho_j * Rho_j))) *
    //          grad_i_wij;
    //        }
    //      }
    //    }
    //  }
    //  F_p[i_idx] = my_F_p;

    //  if (abs(sum_W) < EPSILON) {
    //    sortedRhoPreMu[i_idx].y = 0;
    //  } else {
    //    sortedRhoPreMu[i_idx].y = sum_pW / sum_W;
    //  }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Calc_HelperMarkers_normals(Real4* sortedPosRad,
                                           Real4* sortedRhoPreMu,
                                           Real3* helpers_normal,
                                           int* myType,
                                           uint* cellStart,
                                           uint* cellEnd,
                                           uint* gridMarkerIndexD,
                                           int numAllMarkers,
                                           volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    myType[i_idx] = (int)round(sortedRhoPreMu[i_idx].w);

    if (sortedRhoPreMu[i_idx].w != -3) {
        return;
    }
    int Original_idx = gridMarkerIndexD[i_idx];
    Real3 posi = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;

    int j_pre = i_idx;
    int num_samples = 0;
    Real3 my_normal = mR3(0.1);

    //    int3 gridPos = calcGridPos(posi);
    //    for (int z = -2; z <= 2; z++) {
    //        for (int y = -2; y <= 2; y++) {
    //            for (int x = -2; x <= 2; x++) {
    //                int3 neighbourPos = gridPos + mI3(x, y, z);
    //                uint gridHash = calcGridHash(neighbourPos);
    //                uint startIndex = cellStart[gridHash];
    //                uint endIndex = cellEnd[gridHash];
    //                for (uint j = startIndex; j < endIndex; j++) {
    //                    if (sortedRhoPreMu[j].w != -3) {
    //                        continue;
    //                    }
    //                    if (j_pre == i_idx) {
    //                        j_pre = j;
    //                        continue;
    //                    } else {
    //                        Real3 posj = mR3(sortedPosRad[j]);
    //                        Real3 posjpre = mR3(sortedPosRad[j_pre]);
    //                        Real3 temp = cross(posj - posi, posjpre - posi);
    //                        temp = temp / length(temp);
    //                        num_samples++;
    //                        my_normal += temp;
    //                        j_pre = i_idx;
    //                    }
    //                }
    //            }
    //        }
    //    }

    Real3 cent = mR3(-0.005, 0, 0.195);
    my_normal = posi - cent;
    my_normal = mR3(my_normal.x, 0.0, my_normal.z);
    //    Real3 test = (posi - cent);
    //    test.y = 0;

    //    if (posi.y > 0.038 && r < 0.098)
    //        my_normal = mR3(0, 1, 0);
    //    else if (posi.y > 0.038 && r > 0.098)
    //        my_normal = normalize(my_normal + mR3(0, 0.05, 0));
    //    else if (posi.y < -0.038 && length(my_normal) < 0.098)
    //        my_normal = mR3(0, -1, 0);
    //    else if (posi.y < -0.038 && r > 0.098)
    //        my_normal = normalize(my_normal + mR3(0, -0.05, 0));

    //    Real3 cent = mR3(-0.005, 0, 0.195);
    //    my_normal = posi - cent;
    //    normalize(my_normal);

    helpers_normal[Original_idx] = my_normal / length(my_normal);

    //    printf("Original_idx=%d, p=(%f,%f,%f), n1=%f, n2=%f, n3=%f\n", Original_idx, posi.x, posi.y, posi.z,
    //           helpers_normal[Original_idx].x, helpers_normal[Original_idx].y, helpers_normal[Original_idx].z);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Calc_Splits_and_Merges(Real4* sortedPosRad,
                                       Real4* sortedRhoPreMu,
                                       Real3* sortedVelMas,
                                       Real3* helpers_normal,
                                       Real* sumWij_inv,
                                       Real* G_i,
                                       uint* splitMe,
                                       uint* MergeMe,
                                       int* myType,
                                       uint* cellStart,
                                       uint* cellEnd,
                                       uint* gridMarkerIndexD,
                                       Real fineResolution,
                                       Real coarseResolution,
                                       int numAllMarkers,
                                       int limit,
                                       volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Original_idx = gridMarkerIndexD[i_idx];
    if (i_idx >= numAllMarkers) {
        return;
    }

    if (sortedRhoPreMu[i_idx].w != -3) {
        return;
    }

    Real3 posi = mR3(sortedPosRad[i_idx]);

    //    MergingMarkers1[Original_idx] = mI4(i_idx);
    //    MergingMarkers2[Original_idx] = mI4(i_idx);
    Real3 normal = helpers_normal[Original_idx];
    //    Real3 normal = posi - mR3(-0.005, 0.0, 0.195);

    sortedRhoPreMu[i_idx].x = normal.x;
    sortedRhoPreMu[i_idx].y = normal.y;
    sortedRhoPreMu[i_idx].z = normal.z;

    int3 gridPos = calcGridPos(posi);

    //    printf("Original_idx=%d, p=(%f,%f,%f), n1=%f, n2=%f, n3=%f\n", Original_idx, posi.x, posi.y, posi.z,
    //           helpers_normal[Original_idx].x, helpers_normal[Original_idx].y, helpers_normal[Original_idx].z);
    //    uint N1, N2, N3, N4, N5, N6, N7, N8;
    uint N1 = i_idx;
    uint N2 = i_idx;
    uint N3 = i_idx;
    uint N4 = i_idx;
    uint N5 = i_idx;
    uint N6 = i_idx;
    uint N7 = i_idx;
    uint N8 = i_idx;
    uint mySplits = 0;
    uint myMerges = 0;
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                //                int x = 0, y = 0, z = 0;
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (sortedRhoPreMu[j].w != -1 || j == i_idx)
                        continue;

                    Real3 posj = mR3(sortedPosRad[j]);
                    Real3 dist3 = Distance(posj, posi);
                    Real d = length(dist3);
                    Real3 velj = sortedVelMas[j];
                    Real cosT = dot(dist3, normal) / (length(dist3) * length(normal));
                    Real cosTv = dot(velj, normal) / (length(velj) * length(normal));

                    //                    Real y2 = d * d * cosT * cosT;
                    //                    Real b2 = coarseResolution * coarseResolution;
                    Real x2 = d * d * (1 - cosT * cosT);
                    //                    Real a2 = fineResolution * fineResolution;
                    //                    y2 / b2 + x2 / a2 < 1

                    ////////////////////////////////////////////////////////////////
                    ////////////////////////////////////////////////////////////////
                    /*x2 < 9 / 4 * fineResolution * fineResolution &&*/
                    /*|| (d * cosT < fineResolution * 1.5 && dot(velj, normal) > 0))*/
                    /*||
                         (d * cosT < fineResolution * 1 && dot(velj, normal) > 0.8 * length(velj)))*/

                    //                    if (abs(sortedPosRad[j].w - fineResolution) < EPSILON && (MergeMe[j] ==
                    //                    i_idx)
                    //                    &&
                    //                        (MergeMe[j] == i_idx) && myType[j] == -1 && dot(dist3, normal) > 0 &&
                    //                        x2 < fineResolution * fineResolution) {
                    //////////////////////////////////////////////////////////////////
                    //////////////////////////////////////////////////////////////////
                    //                    if (sortedRhoPreMu[j].w == -1 && abs(sortedPosRad[j].w - fineResolution) <
                    //                    EPSILON &&
                    //                        myType[j] == -1 && MergeMe[j] == 0 && splitMe[j] == 0 && d <
                    //                        coarseResolution && cosTv > 0.2) {
                    //        (dot(dist3, normal) > 0 || (d < 2 * fineResolution && cosTv > 0.2)) &&
                    if (abs(sortedPosRad[j].w - fineResolution) < EPSILON && (MergeMe[j] == 0) &&
                        (sortedRhoPreMu[j].w == -1) && posj.x > 0 && dot(dist3, normal) > 0) {
                        //                        if (x != 0 || y != 0 || z != 0)
                        //                            continue;
                        uint p = 9;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N8]))) || N8 == i_idx)
                            p = 8;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N7]))) || N7 == i_idx)
                            p = 7;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N6]))) || N6 == i_idx)
                            p = 6;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N5]))) || N5 == i_idx)
                            p = 5;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N4]))) || N4 == i_idx)
                            p = 4;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N3]))) || N3 == i_idx)
                            p = 3;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N2]))) || N2 == i_idx)
                            p = 2;
                        if (d < length(Distance(posi, mR3(sortedPosRad[N1]))) || N1 == i_idx)
                            p = 1;

                        //            if (abs(sortedPosRad[j].w - coarseResolution) < EPSILON &&
                        //            length(sortedPosRad[j]-sortedPosRad[j])<fineResolution)

                        // Release the last one if
                        if (p < 9) {
                            //                            atomicCAS(&MergeMe[j], 0, i_idx);
                            if (MergeMe[j] != i_idx)
                                continue;
                            // Release the last one since it is going to be replaced
                            //                            if (N8 != i_idx)
                            //                                atomicExch(&MergeMe[N8], 0);
                        }
                        if (p == 8) {
                            N8 = j;
                        } else if (p == 7) {
                            N8 = N7;
                            N7 = j;
                        } else if (p == 6) {
                            N8 = N7;
                            N7 = N6;
                            N6 = j;
                        } else if (p == 5) {
                            N8 = N7;
                            N7 = N6;
                            N6 = N5;
                            N5 = j;
                        } else if (p == 4) {
                            N8 = N7;
                            N7 = N6;
                            N6 = N5;
                            N5 = N4;
                            N4 = j;
                        } else if (p == 3) {
                            N8 = N7;
                            N7 = N6;
                            N6 = N5;
                            N5 = N4;
                            N4 = N3;
                            N3 = j;
                        } else if (p == 2) {
                            N8 = N7;
                            N7 = N6;
                            N6 = N5;
                            N5 = N4;
                            N4 = N3;
                            N3 = N2;
                            N2 = j;
                        } else if (p == 1) {
                            N8 = N7;
                            N7 = N6;
                            N6 = N5;
                            N5 = N4;
                            N4 = N3;
                            N3 = N2;
                            N2 = N1;
                            N1 = j;
                        }
                    } else if (abs(sortedPosRad[j].w - coarseResolution) < EPSILON && sortedRhoPreMu[j].w == -1 &&
                               splitMe[j] == 0 && myType[j] == -1 && sortedRhoPreMu[j].w == -1 &&
                               d < 2 * coarseResolution && cosTv < 0.0 && dot(dist3, normal) < 0 && posj.x < 0) {
                        //                    } else if (abs(sortedPosRad[j].w - coarseResolution) < EPSILON &&
                        //                    splitMe[j] == 0 &&
                        //                               MergeMe[j] == 0 && myType[j] == -1 && sortedRhoPreMu[j].w
                        //                               == -1
                        //                               && d < 1 * coarseResolution && cosTv < -0.5) {
                        //                        atomicCAS(&splitMe[j], 0, i_idx);
                    }
                }
            }
        }  // namespace fsi
    }      // namespace chrono

    // note that this can cause race condition if two helper markers try to merge the same marker Ni
    if (N8 != i_idx && N7 != i_idx && N6 != i_idx && N5 != i_idx && N4 != i_idx && N3 != i_idx && N2 != i_idx &&
        N1 != i_idx) {
        if (MergeMe[N1] != i_idx || MergeMe[N2] != i_idx || MergeMe[N3] != i_idx || MergeMe[N4] != i_idx ||
            MergeMe[N5] != i_idx || MergeMe[N6] != i_idx || MergeMe[N7] != i_idx || MergeMe[N8] != i_idx) {
            printf("RACE CONDITION in merging! Please revise the spacing or the merging scheme.\n");
            *isErrorD = true;
        }

        //        printf("idx=%d merging %d,%d,%d,%d,%d,%d,%d,%d\n", i_idx, N1, N2, N3, N4, N5, N6, N7, N8);
        if (sortedPosRad[N1].w != fineResolution || sortedPosRad[N2].w != fineResolution ||
            sortedPosRad[N3].w != fineResolution || sortedPosRad[N4].w != fineResolution ||
            sortedPosRad[N5].w != fineResolution || sortedPosRad[N6].w != fineResolution ||
            sortedPosRad[N7].w != fineResolution || sortedPosRad[N8].w != fineResolution) {
            printf("ops something went wrong!.\n");
            *isErrorD = true;
        }

        Real4 center = 0.125 * (sortedPosRad[N1] + sortedPosRad[N2] + sortedPosRad[N3] + sortedPosRad[N4] +
                                sortedPosRad[N5] + sortedPosRad[N6] + sortedPosRad[N7] + sortedPosRad[N8]);

        Real3 myGrad[8];
        grad_scalar(N1, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[0], cellStart, cellEnd);
        grad_scalar(N2, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[1], cellStart, cellEnd);
        grad_scalar(N3, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[2], cellStart, cellEnd);
        grad_scalar(N4, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[3], cellStart, cellEnd);
        grad_scalar(N5, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[4], cellStart, cellEnd);
        grad_scalar(N6, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[5], cellStart, cellEnd);
        grad_scalar(N7, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[6], cellStart, cellEnd);
        grad_scalar(N8, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[7], cellStart, cellEnd);
        sortedRhoPreMu[N1].x -= dot(myGrad[0], mR3(sortedPosRad[N1] - center));
        sortedRhoPreMu[N2].x -= dot(myGrad[1], mR3(sortedPosRad[N2] - center));
        sortedRhoPreMu[N3].x -= dot(myGrad[2], mR3(sortedPosRad[N3] - center));
        sortedRhoPreMu[N4].x -= dot(myGrad[3], mR3(sortedPosRad[N4] - center));
        sortedRhoPreMu[N5].x -= dot(myGrad[4], mR3(sortedPosRad[N5] - center));
        sortedRhoPreMu[N6].x -= dot(myGrad[5], mR3(sortedPosRad[N6] - center));
        sortedRhoPreMu[N7].x -= dot(myGrad[6], mR3(sortedPosRad[N7] - center));
        sortedRhoPreMu[N8].x -= dot(myGrad[7], mR3(sortedPosRad[N8] - center));

        Real4 rpmt = 0.125 * (sortedRhoPreMu[N1] + sortedRhoPreMu[N2] + sortedRhoPreMu[N3] + sortedRhoPreMu[N4] +
                              sortedRhoPreMu[N5] + sortedRhoPreMu[N6] + sortedRhoPreMu[N7] + sortedRhoPreMu[N8]);

        if (rpmt.x > 1.0 * paramsD.rho0)
            return;

        sortedRhoPreMu[N1] = rpmt;

        Real3 myGradx[8];
        Real3 myGrady[8];
        Real3 myGradz[8];
        grad_vector(N1, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[0], myGrady[0], myGradz[0],
                    cellStart, cellEnd);
        grad_vector(N2, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[1], myGrady[1], myGradz[1],
                    cellStart, cellEnd);
        grad_vector(N3, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[2], myGrady[2], myGradz[2],
                    cellStart, cellEnd);
        grad_vector(N4, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[3], myGrady[3], myGradz[3],
                    cellStart, cellEnd);
        grad_vector(N5, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[4], myGrady[4], myGradz[4],
                    cellStart, cellEnd);
        grad_vector(N6, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[5], myGrady[5], myGradz[5],
                    cellStart, cellEnd);
        grad_vector(N7, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[6], myGrady[6], myGradz[6],
                    cellStart, cellEnd);
        grad_vector(N8, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[7], myGrady[7], myGradz[7],
                    cellStart, cellEnd);

        sortedVelMas[N1].x -= dot(myGradx[0], mR3(sortedPosRad[N1] - center));
        sortedVelMas[N2].x -= dot(myGradx[1], mR3(sortedPosRad[N2] - center));
        sortedVelMas[N3].x -= dot(myGradx[2], mR3(sortedPosRad[N3] - center));
        sortedVelMas[N4].x -= dot(myGradx[3], mR3(sortedPosRad[N4] - center));
        sortedVelMas[N5].x -= dot(myGradx[4], mR3(sortedPosRad[N5] - center));
        sortedVelMas[N6].x -= dot(myGradx[5], mR3(sortedPosRad[N6] - center));
        sortedVelMas[N7].x -= dot(myGradx[6], mR3(sortedPosRad[N7] - center));
        sortedVelMas[N8].x -= dot(myGradx[7], mR3(sortedPosRad[N8] - center));

        sortedVelMas[N1].y -= dot(myGrady[0], mR3(sortedPosRad[N1] - center));
        sortedVelMas[N2].y -= dot(myGrady[1], mR3(sortedPosRad[N2] - center));
        sortedVelMas[N3].y -= dot(myGrady[2], mR3(sortedPosRad[N3] - center));
        sortedVelMas[N4].y -= dot(myGrady[3], mR3(sortedPosRad[N4] - center));
        sortedVelMas[N5].y -= dot(myGrady[4], mR3(sortedPosRad[N5] - center));
        sortedVelMas[N6].y -= dot(myGrady[5], mR3(sortedPosRad[N6] - center));
        sortedVelMas[N7].y -= dot(myGrady[6], mR3(sortedPosRad[N7] - center));
        sortedVelMas[N8].y -= dot(myGrady[7], mR3(sortedPosRad[N8] - center));

        sortedVelMas[N1].z -= dot(myGradz[0], mR3(sortedPosRad[N1] - center));
        sortedVelMas[N2].z -= dot(myGradz[1], mR3(sortedPosRad[N2] - center));
        sortedVelMas[N3].z -= dot(myGradz[2], mR3(sortedPosRad[N3] - center));
        sortedVelMas[N4].z -= dot(myGradz[3], mR3(sortedPosRad[N4] - center));
        sortedVelMas[N5].z -= dot(myGradz[4], mR3(sortedPosRad[N5] - center));
        sortedVelMas[N6].z -= dot(myGradz[5], mR3(sortedPosRad[N6] - center));
        sortedVelMas[N7].z -= dot(myGradz[6], mR3(sortedPosRad[N7] - center));
        sortedVelMas[N8].z -= dot(myGradz[7], mR3(sortedPosRad[N8] - center));

        sortedVelMas[N1] = 0.125 * (sortedVelMas[N1] + sortedVelMas[N2] + sortedVelMas[N3] + sortedVelMas[N4] +
                                    sortedVelMas[N5] + sortedVelMas[N6] + sortedVelMas[N7] + sortedVelMas[N8]);
        sortedPosRad[N1] = center;

        if (sortedPosRad[N1].w != fineResolution || sortedRhoPreMu[N1].w != -1) {
            printf("ops something went wrong!!!!\n");
            *isErrorD = true;
        }
        sortedPosRad[N1].w = coarseResolution;
        sortedRhoPreMu[N1].w = -1;
        sortedRhoPreMu[N2].w = -2;
        sortedRhoPreMu[N3].w = -2;
        sortedRhoPreMu[N4].w = -2;
        sortedRhoPreMu[N5].w = -2;
        sortedRhoPreMu[N6].w = -2;
        sortedRhoPreMu[N7].w = -2;
        sortedRhoPreMu[N8].w = -2;
        sortedPosRad[N2] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N3] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N4] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N5] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N6] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N7] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedPosRad[N8] = mR4(mR3(0.0, 0.0, -0.8), coarseResolution);
        sortedVelMas[N2] = mR3(0.0);
        sortedVelMas[N3] = mR3(0.0);
        sortedVelMas[N4] = mR3(0.0);
        sortedVelMas[N5] = mR3(0.0);
        sortedVelMas[N6] = mR3(0.0);
        sortedVelMas[N7] = mR3(0.0);
        sortedVelMas[N8] = mR3(0.0);
    }
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Split(Real4* sortedPosRad,
                      Real4* sortedRhoPreMu,
                      Real3* sortedVelMas,
                      Real3* helpers_normal,
                      Real* sumWij_inv,
                      Real* G_i,
                      Real* L_i,
                      uint* splitMe,
                      uint* MergeMe,
                      int* myType,
                      uint* cellStart,
                      uint* cellEnd,
                      uint* gridMarkerIndexD,
                      Real fineResolution,
                      Real coarseResolution,
                      int numAllMarkers,
                      int limit,
                      volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Original_idx = gridMarkerIndexD[i_idx];
    if (i_idx >= numAllMarkers) {
        return;
    }
    if (splitMe[i_idx] == 0)
        return;

    Real3 posi = mR3(sortedPosRad[i_idx]);
    Real3 veli = sortedVelMas[i_idx];

    if (abs(sortedPosRad[i_idx].w - coarseResolution) < EPSILON && sortedRhoPreMu[i_idx].w == -1 &&
        splitMe[i_idx] != 0 && myType[i_idx] == -1) {
        uint temp = 0;

        MergeMe[i_idx] = -1;
        uint childMarkers[7] = {0};
        temp = 0;
        uint k = 0;
        int numpass = 0;
        Real4 RhoPreMu = sortedRhoPreMu[i_idx];

        while (k < 7) {
            //            atomicCAS(&myType[temp], -2, i_idx);
            if (myType[temp] == i_idx) {
                childMarkers[k] = temp;
                k++;
            }
            if (temp > numAllMarkers) {
                if (numpass < 2) {
                    temp = 0;
                    continue;
                }
                break;
                *isErrorD = true;
            }
            temp++;
        }
        for (int n = 0; n < 7; n++) {
            sortedVelMas[childMarkers[n]] = veli;
            sortedRhoPreMu[childMarkers[n]] = RhoPreMu;
            sumWij_inv[childMarkers[n]] = sumWij_inv[i_idx];

            for (int l = 0; l < 9; l++)
                G_i[childMarkers[n] * 9 + l] = G_i[i_idx * 9 + l];

            for (int l = 0; l < 6; l++)
                L_i[childMarkers[n] * 6 + l] = L_i[i_idx * 6 + l];
        }
        Real3 center = mR3(sortedPosRad[i_idx]);
        Real h = fineResolution;

        int N1 = childMarkers[0];
        int N2 = childMarkers[1];
        int N3 = childMarkers[2];
        int N4 = childMarkers[3];
        int N5 = childMarkers[4];
        int N6 = childMarkers[5];
        int N7 = childMarkers[6];
        int N8 = i_idx;
        Real d = 0.5 * fineResolution;
        // This is recommended for our confuguration from Vacondio et. al
        //                Real d = 0.72 * coarseResolution / 1.7321;
        sortedPosRad[N1] = mR4(center + d * mR3(-1, -1, -1), h);
        sortedPosRad[N2] = mR4(center + d * mR3(+1, -1, -1), h);
        sortedPosRad[N3] = mR4(center + d * mR3(+1, +1, -1), h);
        sortedPosRad[N4] = mR4(center + d * mR3(-1, +1, -1), h);
        sortedPosRad[N5] = mR4(center + d * mR3(-1, -1, +1), h);
        sortedPosRad[N6] = mR4(center + d * mR3(+1, -1, +1), h);
        sortedPosRad[N7] = mR4(center + d * mR3(+1, +1, +1), h);
        sortedPosRad[N8] = mR4(center + d * mR3(-1, +1, +1), h);

        Real3 myGrad[8] = {0};
        Real rho0 = +sortedRhoPreMu[N8].x;
        grad_scalar(N1, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[0], cellStart, cellEnd);
        grad_scalar(N2, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[1], cellStart, cellEnd);
        grad_scalar(N3, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[2], cellStart, cellEnd);
        grad_scalar(N4, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[3], cellStart, cellEnd);
        grad_scalar(N5, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[4], cellStart, cellEnd);
        grad_scalar(N6, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[5], cellStart, cellEnd);
        grad_scalar(N7, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[6], cellStart, cellEnd);
        grad_scalar(N8, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedRhoPreMu, myGrad[7], cellStart, cellEnd);
        sortedRhoPreMu[N1].x += dot(myGrad[0], mR3(sortedPosRad[N1]) - center);
        sortedRhoPreMu[N2].x += dot(myGrad[1], mR3(sortedPosRad[N2]) - center);
        sortedRhoPreMu[N3].x += dot(myGrad[2], mR3(sortedPosRad[N3]) - center);
        sortedRhoPreMu[N4].x += dot(myGrad[3], mR3(sortedPosRad[N4]) - center);
        sortedRhoPreMu[N5].x += dot(myGrad[4], mR3(sortedPosRad[N5]) - center);
        sortedRhoPreMu[N6].x += dot(myGrad[5], mR3(sortedPosRad[N6]) - center);
        sortedRhoPreMu[N7].x += dot(myGrad[6], mR3(sortedPosRad[N7]) - center);
        sortedRhoPreMu[N8].x += dot(myGrad[7], mR3(sortedPosRad[N8]) - center);

        Real3 myGradx[8] = {0};
        Real3 myGrady[8] = {0};
        Real3 myGradz[8] = {0};
        grad_vector(N1, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[0], myGrady[0], myGradz[0],
                    cellStart, cellEnd);
        grad_vector(N2, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[1], myGrady[1], myGradz[1],
                    cellStart, cellEnd);
        grad_vector(N3, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[2], myGrady[2], myGradz[2],
                    cellStart, cellEnd);
        grad_vector(N4, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[3], myGrady[3], myGradz[3],
                    cellStart, cellEnd);
        grad_vector(N5, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[4], myGrady[4], myGradz[4],
                    cellStart, cellEnd);
        grad_vector(N6, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[5], myGrady[5], myGradz[5],
                    cellStart, cellEnd);
        grad_vector(N7, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[6], myGrady[6], myGradz[6],
                    cellStart, cellEnd);
        grad_vector(N8, sortedPosRad, sortedRhoPreMu, sumWij_inv, G_i, sortedVelMas, myGradx[7], myGrady[7], myGradz[7],
                    cellStart, cellEnd);

        sortedVelMas[N1].x += dot(myGradx[0], mR3(sortedPosRad[N1]) - center);
        sortedVelMas[N2].x += dot(myGradx[1], mR3(sortedPosRad[N2]) - center);
        sortedVelMas[N3].x += dot(myGradx[2], mR3(sortedPosRad[N3]) - center);
        sortedVelMas[N4].x += dot(myGradx[3], mR3(sortedPosRad[N4]) - center);
        sortedVelMas[N5].x += dot(myGradx[4], mR3(sortedPosRad[N5]) - center);
        sortedVelMas[N6].x += dot(myGradx[5], mR3(sortedPosRad[N6]) - center);
        sortedVelMas[N7].x += dot(myGradx[6], mR3(sortedPosRad[N7]) - center);
        sortedVelMas[N8].x += dot(myGradx[7], mR3(sortedPosRad[N8]) - center);

        sortedVelMas[N1].y += dot(myGrady[0], mR3(sortedPosRad[N1]) - center);
        sortedVelMas[N2].y += dot(myGrady[1], mR3(sortedPosRad[N2]) - center);
        sortedVelMas[N3].y += dot(myGrady[2], mR3(sortedPosRad[N3]) - center);
        sortedVelMas[N4].y += dot(myGrady[3], mR3(sortedPosRad[N4]) - center);
        sortedVelMas[N5].y += dot(myGrady[4], mR3(sortedPosRad[N5]) - center);
        sortedVelMas[N6].y += dot(myGrady[5], mR3(sortedPosRad[N6]) - center);
        sortedVelMas[N7].y += dot(myGrady[6], mR3(sortedPosRad[N7]) - center);
        sortedVelMas[N8].y += dot(myGrady[7], mR3(sortedPosRad[N8]) - center);

        sortedVelMas[N1].z += dot(myGradz[0], mR3(sortedPosRad[N1]) - center);
        sortedVelMas[N2].z += dot(myGradz[1], mR3(sortedPosRad[N2]) - center);
        sortedVelMas[N3].z += dot(myGradz[2], mR3(sortedPosRad[N3]) - center);
        sortedVelMas[N4].z += dot(myGradz[3], mR3(sortedPosRad[N4]) - center);
        sortedVelMas[N5].z += dot(myGradz[4], mR3(sortedPosRad[N5]) - center);
        sortedVelMas[N6].z += dot(myGradz[5], mR3(sortedPosRad[N6]) - center);
        sortedVelMas[N7].z += dot(myGradz[6], mR3(sortedPosRad[N7]) - center);
        sortedVelMas[N8].z += dot(myGradz[7], mR3(sortedPosRad[N8]) - center);

        //        printf("extrapolate vel of marker %d with %f\n", N1, dot(myGradx[0], mR3(sortedPosRad[N1])
        //        - center));

        if (temp > numAllMarkers)
            printf(
                "Reached the limit of the ghost markers. Please increase the number of ghost "
                "markers.\n");
    }
}

//--------------------------------------------------------------------------------------------------------------------------------

// ChFsiForceI2SPH::ChFsiForceI2SPH() {}

// ChFsiForceIISPH::ChFsiForceIISPH() {}
ChFsiForceIISPH::~ChFsiForceIISPH() {}
ChFsiForceIISPH::ChFsiForceIISPH(
    ChBce* otherBceWorker,                   ///< Pointer to the ChBce object that handles BCE markers
    SphMarkerDataD* otherSortedSphMarkersD,  ///< Information of markers in the sorted array on device
    ProximityDataD*
        otherMarkersProximityD,           ///< Pointer to the object that holds the proximity of the markers on device
    FsiGeneralData* otherFsiGeneralData,  ///< Pointer to the sph general data
    SimParams* otherParamsH,              ///< Pointer to the simulation parameters on host
    NumberOfObjects* otherNumObjects      ///< Pointer to number of objects, fluid and boundary markers, etc.
    )
    : ChFsiForceParallel(otherBceWorker,
                         otherSortedSphMarkersD,
                         otherMarkersProximityD,
                         otherFsiGeneralData,
                         otherParamsH,
                         otherNumObjects) {
    printf("ChFsiForceIISPH::ChFsiForceIISPH constructor\n");
}

void ChFsiForceIISPH::calcPressureIISPH(thrust::device_vector<Real4> velMassRigid_fsiBodies_D,
                                        thrust::device_vector<Real3> accRigid_fsiBodies_D,
                                        thrust::device_vector<Real3> pos_fsi_fea_D,
                                        thrust::device_vector<Real3> vel_fsi_fea_D,
                                        thrust::device_vector<Real3> acc_fsi_fea_D,
                                        thrust::device_vector<Real>& sumWij_inv,
                                        thrust::device_vector<Real>& G_i,
                                        thrust::device_vector<Real>& L_i,
                                        thrust::device_vector<Real>& Color) {
    //    Real RES = paramsH->PPE_res;

    PPE_SolutionType mySolutionType = paramsH->PPE_Solution_type;

    double total_step_timeClock = clock();
    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    //------------------------------------------------------------------------
    // thread per particle
    uint numThreads, numBlocks;
    int numAllMarkers = numObjectsH->numAllMarkers;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);

    printf("numBlocks: %d, numThreads: %d, numAllMarker:%d \n", numBlocks, numThreads, numAllMarkers);

    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    //    calcRho_kernel<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(sumWij_inv),
    //        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    //
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed after calcRho_kernel!\n");
    //    }
    //
    //    thrust::device_vector<Real> dxi_over_Vi(numAllMarkers);
    //    thrust::fill(dxi_over_Vi.begin(), dxi_over_Vi.end(), 0);
    //
    //    calcNormalizedRho_kernel<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
    //        mR4CAST(sortedSphMarkersD->rhoPresMuD),  // input: sorted velocities
    //        R1CAST(sumWij_inv), R1CAST(G_i), R1CAST(dxi_over_Vi), R1CAST(Color),
    //        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed after calcNormalizedRho_kernel!\n");
    //    }
    /*
        if (paramsH->Adaptive_time_stepping) {
          int position = thrust::min_element(dxi_over_Vi.begin(), dxi_over_Vi.end()) - dxi_over_Vi.begin();
          Real min_dxi_over_Vi = dxi_over_Vi[position];

          Real dt = paramsH->Co_number * min_dxi_over_Vi;
          printf("Min dxi_over_Vi of fluid particles to boundary is: %f. Time step based on Co=%f is %f\n",
       min_dxi_over_Vi,
                 paramsH->Co_number, dt);
          // I am doing this to prevent very low time steps (when it requires it to save data at the current
       step)
          // Because if will have to do two time steps either way
          if (dt / paramsH->dT_Max > 0.7 && dt / paramsH->dT_Max < 1)
            paramsH->dT = paramsH->dT_Max * 0.5;
          else
            paramsH->dT = fminf((float)dt, paramsH->dT_Max);

          std::cout << "time step is set to min(dt_Co,dT_Max)= " << paramsH->dT << "\n";
        }
        */
    thrust::device_vector<Real3> d_ii(numAllMarkers);
    thrust::device_vector<Real3> V_np(numAllMarkers);
    thrust::fill(d_ii.begin(), d_ii.end(), mR3(0.0));
    thrust::fill(V_np.begin(), V_np.end(), mR3(0.0));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    V_i_np__AND__d_ii_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(d_ii), mR3CAST(V_np), R1CAST(sumWij_inv), R1CAST(G_i),
        R1CAST(L_i), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers,
        isErrorD);

    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after F_i_np__AND__d_ii_kernel!\n");
    }

    thrust::device_vector<Real> a_ii(numAllMarkers);
    thrust::device_vector<Real> rho_np(numAllMarkers);
    thrust::device_vector<Real> p_old(numAllMarkers);
    thrust::fill(a_ii.begin(), a_ii.end(), 0.0);
    thrust::fill(rho_np.begin(), rho_np.end(), 0.0);
    thrust::fill(p_old.begin(), p_old.end(), 0.0);
    thrust::device_vector<Real3> summGradW(numAllMarkers);

    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    Rho_np_AND_a_ii_AND_sum_m_GradW<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(rho_np), R1CAST(a_ii),
        R1CAST(p_old), mR3CAST(V_np), mR3CAST(d_ii), mR3CAST(summGradW), U1CAST(markersProximityD->cellStartD),
        U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);

    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after F_i_np__AND__d_ii_kernel!\n");
    }

    thrust::device_vector<Real3> V_new(numAllMarkers);
    thrust::fill(V_new.begin(), V_new.end(), mR3(0.0));
    thrust::device_vector<Real> a_ij;
    thrust::device_vector<Real> B_i(numAllMarkers);
    thrust::device_vector<uint> csrColIndA;
    thrust::device_vector<uint> numContacts(numAllMarkers);
    thrust::device_vector<unsigned long int> GlobalcsrColIndA;
    thrust::device_vector<Real> csrValA;

    double durationFormAXB;

    int numFlexbodies = +numObjectsH->numFlexBodies1D + numObjectsH->numFlexBodies2D;

    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;

    int4 updatePortion =
        mI4(fsiGeneralData->referenceArray[haveGhost + haveHelper + 0].y,  // end of fluid
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1].y,  // end of boundary
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies].y,
            fsiGeneralData->referenceArray[haveGhost + haveHelper + 1 + numObjectsH->numRigidBodies + numFlexbodies].y);

    uint NNZ;
    if (mySolutionType == FORM_SPARSE_MATRIX) {
        thrust::fill(a_ij.begin(), a_ij.end(), 0.0);
        thrust::fill(B_i.begin(), B_i.end(), 0.0);
        thrust::fill(summGradW.begin(), summGradW.end(), mR3(0.0));
        thrust::fill(numContacts.begin(), numContacts.end(), 0.0);
        //------------------------------------------------------------------------
        //------------- MatrixJacobi
        //------------------------------------------------------------------------

        bool SPARSE_FLAG = true;
        double FormAXBClock = clock();
        thrust::device_vector<Real> Residuals(numAllMarkers);
        thrust::fill(Residuals.begin(), Residuals.end(), 1.0);
        thrust::device_vector<Real> rho_p(numAllMarkers);
        thrust::fill(rho_p.begin(), rho_p.end(), 0.0);

        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
        CalcNumber_Contacts<<<numBlocks, numThreads>>>(
            U1CAST(numContacts), mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
            U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);

        hipDeviceSynchronize();
        cudaCheckError();
        hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
        if (*isErrorH == true) {
            throw std::runtime_error("Error! program crashed after CalcNumber_Contacts!\n");
        }
        uint MAX_CONTACT = thrust::reduce(numContacts.begin(), numContacts.end(), 0, thrust::maximum<Real>());
        std::cout << "Max contact between SPH particles: " << MAX_CONTACT << std::endl;

        uint LastVal = numContacts[numAllMarkers - 1];
        thrust::exclusive_scan(numContacts.begin(), numContacts.end(), numContacts.begin());
        numContacts.push_back(LastVal + numContacts[numAllMarkers - 1]);
        NNZ = numContacts[numAllMarkers];

        csrValA.resize(NNZ);
        csrColIndA.resize(NNZ);
        GlobalcsrColIndA.resize(NNZ);

        thrust::fill(csrValA.begin(), csrValA.end(), 0.0);
        thrust::fill(GlobalcsrColIndA.begin(), GlobalcsrColIndA.end(), 0.0);
        thrust::fill(csrColIndA.begin(), csrColIndA.end(), 0.0);

        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

        std::cout << "updatePortion of  BC: " << updatePortion.x << " " << updatePortion.y << " " << updatePortion.z
                  << " " << updatePortion.w << "\n ";

        FormAXB<<<numBlocks, numThreads>>>(
            R1CAST(csrValA), U1CAST(csrColIndA), LU1CAST(GlobalcsrColIndA), U1CAST(numContacts), R1CAST(a_ij),
            R1CAST(B_i), mR3CAST(d_ii), R1CAST(a_ii), mR3CAST(summGradW), mR4CAST(sortedSphMarkersD->posRadD),
            mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(V_new), R1CAST(p_old),
            R1CAST(rho_np),

            mR4CAST(velMassRigid_fsiBodies_D), mR3CAST(accRigid_fsiBodies_D), U1CAST(fsiGeneralData->rigidIdentifierD),

            mR3CAST(pos_fsi_fea_D), mR3CAST(vel_fsi_fea_D), mR3CAST(acc_fsi_fea_D),
            U1CAST(fsiGeneralData->FlexIdentifierD), numObjectsH->numFlexBodies1D,
            U2CAST(fsiGeneralData->CableElementsNodes), U4CAST(fsiGeneralData->ShellElementsNodes),

            updatePortion, U1CAST(markersProximityD->gridMarkerIndexD), U1CAST(markersProximityD->cellStartD),
            U1CAST(markersProximityD->cellEndD), numAllMarkers, SPARSE_FLAG, isErrorD);

        hipDeviceSynchronize();
        cudaCheckError();
        hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
        if (*isErrorH == true) {
            throw std::runtime_error("Error! program crashed after F_i_np__AND__d_ii_kernel!\n");
        }

        durationFormAXB = (clock() - FormAXBClock) / (double)CLOCKS_PER_SEC;
    }
    //------------------------------------------------------------------------
    //------------- Iterative loop
    //------------------------------------------------------------------------
    int Iteration = 0;
    Real MaxRes = 100;
    thrust::device_vector<Real> Residuals(numAllMarkers);
    thrust::fill(Residuals.begin(), Residuals.end(), 1.0);
    thrust::device_vector<Real3> dij_pj(numAllMarkers);
    thrust::fill(dij_pj.begin(), dij_pj.end(), mR3(0.0));
    thrust::device_vector<Real3> F_p(numAllMarkers);
    thrust::fill(F_p.begin(), F_p.end(), mR3(0.0));
    thrust::device_vector<Real> rho_p(numAllMarkers);
    thrust::fill(rho_p.begin(), rho_p.end(), 0.0);

    double LinearSystemClock = clock();

    ChFsiLinearSolver myLS(paramsH->LinearSolver, paramsH->LinearSolver_Rel_Tol, paramsH->LinearSolver_Abs_Tol,
                           paramsH->LinearSolver_Max_Iter, paramsH->Verbose_monitoring);
    if (paramsH->USE_LinearSolver) {
        if (paramsH->PPE_Solution_type != FORM_SPARSE_MATRIX) {
            printf(
                "You should paramsH->PPE_Solution_type == FORM_SPARSE_MATRIX in order to use the "
                "chrono_fsi linear "
                "solvers\n");
            exit(0);
        }

        myLS.Solve(numAllMarkers, NNZ, R1CAST(csrValA), U1CAST(numContacts), U1CAST(csrColIndA), (double*)R1CAST(p_old),
                   R1CAST(B_i));
        cudaCheckError();
    }

    while ((MaxRes > paramsH->LinearSolver_Rel_Tol || Iteration < 3) && paramsH->USE_Iterative_solver &&
           Iteration < paramsH->LinearSolver_Max_Iter) {
        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
        Initialize_Variables<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(p_old),
                                                        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(V_new),
                                                        numAllMarkers, isErrorD);
        hipDeviceSynchronize();
        cudaCheckError();
        hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
        if (*isErrorH == true) {
            throw std::runtime_error("Error! program crashed after Initialize_Variables!\n");
        }

        if (mySolutionType == MATRIX_FREE) {
            *isErrorH = false;
            hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
            Calc_dij_pj<<<numBlocks, numThreads>>>(
                mR3CAST(dij_pj), mR3CAST(F_p), mR3CAST(d_ii), mR4CAST(sortedSphMarkersD->posRadD),
                mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(p_old),
                U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
            hipDeviceSynchronize();
            cudaCheckError();
            hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
            if (*isErrorH == true) {
                throw std::runtime_error("Error! program crashed after Calc_dij_pj!\n");
            }

            *isErrorH = false;
            hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
            Calc_Pressure<<<numBlocks, numThreads>>>(
                R1CAST(a_ii), mR3CAST(d_ii), mR3CAST(dij_pj), R1CAST(rho_np), R1CAST(rho_p), mR3CAST(F_p),
                mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
                mR4CAST(sortedSphMarkersD->rhoPresMuD), mR4CAST(velMassRigid_fsiBodies_D),
                mR3CAST(accRigid_fsiBodies_D), U1CAST(fsiGeneralData->rigidIdentifierD), mR3CAST(pos_fsi_fea_D),
                mR3CAST(vel_fsi_fea_D), mR3CAST(acc_fsi_fea_D), U1CAST(fsiGeneralData->FlexIdentifierD),
                numObjectsH->numFlexBodies1D, U2CAST(fsiGeneralData->CableElementsNodes),
                U4CAST(fsiGeneralData->ShellElementsNodes), updatePortion, U1CAST(markersProximityD->gridMarkerIndexD),
                R1CAST(p_old), mR3CAST(V_new), U1CAST(markersProximityD->cellStartD),
                U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);

            hipDeviceSynchronize();
            cudaCheckError();
            hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
            if (*isErrorH == true) {
                throw std::runtime_error("Error! program crashed after Calc_Pressure!\n");
            }
        }

        if (mySolutionType == FORM_SPARSE_MATRIX) {
            *isErrorH = false;
            hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
            Calc_Pressure_AXB_USING_CSR<<<numBlocks, numThreads>>>(
                R1CAST(csrValA), R1CAST(a_ii), U1CAST(csrColIndA), U1CAST(numContacts),
                mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(sumWij_inv), mR3CAST(sortedSphMarkersD->velMasD),
                mR3CAST(V_new), R1CAST(p_old), R1CAST(B_i), numAllMarkers, isErrorD);
            hipDeviceSynchronize();
            cudaCheckError();
            hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
            if (*isErrorH == true) {
                throw std::runtime_error("Error! program crashed after Iterative_pressure_update!\n");
            }
        }
        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

        Update_AND_Calc_Res<<<numBlocks, numThreads>>>(
            mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(p_old), mR3CAST(V_new),
            R1CAST(rho_p), R1CAST(rho_np), R1CAST(Residuals), numAllMarkers, Iteration, paramsH->PPE_relaxation, false,
            isErrorD);
        hipDeviceSynchronize();
        cudaCheckError();
        hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
        if (*isErrorH == true) {
            throw std::runtime_error("Error! program crashed after Iterative_pressure_update!\n");
        }

        Iteration++;

        thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
        unsigned int position = iter - Residuals.begin();
        MaxRes = *iter;

        //    Real PMAX = thrust::reduce(p_old.begin(), p_old.end(), 0.0, thrust::maximum<Real>());
        //    MaxRes = thrust::reduce(Residuals.begin(), Residuals.end(), 0.0, thrust::plus<Real>()) /
        //    numObjectsH->numAllMarkers;
        //    MaxRes = thrust::reduce(Residuals.begin(), Residuals.end(), 0.0, thrust::maximum<Real>());
        //      Real R_np = thrust::reduce(rho_np.begin(), rho_np.end(), 0.0, thrust::plus<Real>()) /
        //      rho_np.size();
        //      Real R_p = thrust::reduce(rho_p.begin(), rho_p.end(), 0.0, thrust::plus<Real>()) /
        //      rho_p.size();
        //
        if (paramsH->Verbose_monitoring)
            printf("Iter= %d, Res= %f\n", Iteration, MaxRes);
    }

    //  thrust::device_vector<Real>::iterator iter = thrust::min_element(p_old.begin(), p_old.end());
    //  unsigned int position = iter - p_old.begin();
    //  Real shift_p = *iter;
    Real shift_p = 0;

    // This must be run if cusp solver is used
    if (paramsH->ClampPressure || paramsH->USE_LinearSolver) {
        //    printf("Shifting pressure values by %f\n", -shift_p);
        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
        FinalizePressure<<<numBlocks, numThreads>>>(
            mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(p_old), mR3CAST(F_p),
            U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, shift_p,
            isErrorD);
        hipDeviceSynchronize();
        cudaCheckError();
        hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
        if (*isErrorH == true) {
            throw std::runtime_error("Error! program crashed after FinalizePressure!\n");
        }
    }
    double durationLinearSystem = (clock() - LinearSystemClock) / (double)CLOCKS_PER_SEC;
    double durationtotal_step_time = (clock() - total_step_timeClock) / (double)CLOCKS_PER_SEC;

    printf("---------------IISPH CLOCK-------------------\n");
    printf(" Total: %f \n FormAXB: %f\n Linear System: %f \n", durationtotal_step_time, durationFormAXB,
           durationLinearSystem);
    if (!paramsH->USE_LinearSolver)
        printf(" Iter (Jacobi+SOR)# = %d, to Res= %f \n", Iteration, MaxRes);
    if (paramsH->USE_LinearSolver)
        if (myLS.GetSolverStatus()) {
            std::cout << " Solver converged to " << myLS.GetResidual() << " tolerance";
            std::cout << " after " << myLS.GetNumIterations() << " iterations" << std::endl;
        } else {
            std::cout << "Failed to converge after " << myLS.GetIterationLimit() << " iterations";
            std::cout << " (" << myLS.GetResidual() << " final residual)" << std::endl;
        }

    //------------------------------------------------------------------------
    //------------------------------------------------------------------------
    hipFree(isErrorD);
    free(isErrorH);
    // dxi_over_Vi.clear();
    p_old.clear();
    d_ii.clear();
    V_np.clear();
    V_new.clear();
    a_ii.clear();
    rho_np.clear();
    rho_p.clear();
    F_p.clear();
    a_ii.clear();
    a_ij.clear();
    B_i.clear();
    dij_pj.clear();
    summGradW.clear();
    csrColIndA.clear();
    GlobalcsrColIndA.clear();
    csrValA.clear();
    V_new.clear();
    Residuals.clear();
    numContacts.clear();
}

void ChFsiForceIISPH::ForceIISPH(SphMarkerDataD* otherSphMarkersD,
                                 FsiBodiesDataD* otherFsiBodiesD,
                                 FsiMeshDataD* otherFsiMeshD) {
    std::cout << "dT in ForceSPH before calcPressure: " << paramsH->dT << "\n";

    sphMarkersD = otherSphMarkersD;
    int numAllMarkers = numObjectsH->numAllMarkers;
    int numHelperMarkers = numObjectsH->numHelperMarkers;
    fsiCollisionSystem->ArrangeData(sphMarkersD);

    bool *isErrorH, *isErrorD, *isErrorD2;

    //    printf("block=%d, threads=%d for helpers\n", numBlocks1, numThreads1);
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    hipMalloc((void**)&isErrorD2, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(isErrorD2, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    //    uint numThreads1, numBlocks1;
    //    computeGridSize(numHelperMarkers, 256, numBlocks1, numThreads1);
    uint numThreads, numBlocks;
    computeGridSize(numAllMarkers, 256, numBlocks, numThreads);

    thrust::device_vector<Real> Color(numAllMarkers);
    thrust::fill(Color.begin(), Color.end(), 1.0e10);
    thrust::device_vector<Real> _sumWij_inv(numAllMarkers);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::device_vector<Real> G_i(numAllMarkers * 9);
    thrust::fill(G_i.begin(), G_i.end(), 0);
    thrust::device_vector<Real> A_i(numAllMarkers * 27);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::device_vector<Real> L_i(numAllMarkers * 6);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    //    calcRho_kernel<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
    //        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    //
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed after calcRho_kernel!\n");
    //    }
    //
    //    thrust::device_vector<Real> dxi_over_Vi(numAllMarkers);
    //    thrust::fill(dxi_over_Vi.begin(), dxi_over_Vi.end(), 0);
    //
    //    calcNormalizedRho_kernel<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
    //        mR4CAST(sortedSphMarkersD->rhoPresMuD),  // input: sorted velocities
    //        R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(dxi_over_Vi), R1CAST(Color),
    //        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed after calcNormalizedRho_kernel!\n");
    //    }
    //
    //    thrust::device_vector<uint> SplitMe(numAllMarkers);
    //    thrust::device_vector<uint> MergeMe(numAllMarkers);
    //    thrust::device_vector<uint> myType(numAllMarkers);
    //    thrust::device_vector<Real3> helpers_normal(numHelperMarkers);
    //
    //    thrust::fill(SplitMe.begin(), SplitMe.end(), 0);
    //    thrust::fill(MergeMe.begin(), MergeMe.end(), 0);
    //    thrust::fill(myType.begin(), myType.end(), 0);
    //    thrust::fill(helpers_normal.begin(), helpers_normal.end(), mR3(0));
    //    hipDeviceSynchronize();
    //
    //    Calc_HelperMarkers_normals<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(helpers_normal),
    //        I1CAST(myType), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
    //        U1CAST(markersProximityD->gridMarkerIndexD), numAllMarkers, isErrorD2);
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed in Calc_HelperMarkers_normals!\n");
    //    }
    //    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    //    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    //    int limit = numObjectsH->numFluidMarkers + numObjectsH->numGhostMarkers + numObjectsH->numHelperMarkers;
    //
    //    Calc_Splits_and_Merges<<<numBlocks, numThreads>>>(
    //        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
    //        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(helpers_normal), R1CAST(_sumWij_inv), R1CAST(G_i),
    //        U1CAST(SplitMe), U1CAST(MergeMe), I1CAST(myType), U1CAST(markersProximityD->cellStartD),
    //        U1CAST(markersProximityD->cellEndD), U1CAST(markersProximityD->gridMarkerIndexD), paramsH->HSML / 2.0,
    //        paramsH->HSML, numAllMarkers, limit, isErrorD2);
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed in Calc_Split_and_Merges!\n");
    //    }
    //
    //    Split<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->posRadD),
    //    mR4CAST(sortedSphMarkersD->rhoPresMuD),
    //                                     mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(helpers_normal),
    //                                     R1CAST(_sumWij_inv), R1CAST(G_i), U1CAST(SplitMe), U1CAST(MergeMe),
    //                                     I1CAST(myType), U1CAST(markersProximityD->cellStartD),
    //                                     U1CAST(markersProximityD->cellEndD),
    //                                     U1CAST(markersProximityD->gridMarkerIndexD), paramsH->HSML / 2.0,
    //                                     paramsH->HSML, numAllMarkers, limit, isErrorD2);
    //    hipDeviceSynchronize();
    //    cudaCheckError();
    //    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    //    if (*isErrorH == true) {
    //        throw std::runtime_error("Error! program crashed in Calc_Split_and_Merges!\n");
    //    }
    //

    printf(
        "calling test calcRho_kernel "
        "sortedSphMarkersD->posRadD.size()=%d,sortedSphMarkersD->rhoPresMuD.size()=%d,_sumWij_inv.size()=%d,"
        "numAllMarkers=%d\n",
        sortedSphMarkersD->posRadD.size(), sortedSphMarkersD->rhoPresMuD.size(), _sumWij_inv.size(), numAllMarkers);

    printf("sizeof(int)=%d,sizeof(double)=%d\n", sizeof(int), sizeof(Real));
    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after calcRho_kernel!\n");
    }

    calcNormalizedRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(_sumWij_inv), R1CAST(Color),
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after calcNormalizedRho_kernel!\n");
    }
    //

    calc_A_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);

    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after calcRho_kernel!\n");
    }

    calc_L_tensor<<<numBlocks, numThreads>>>(R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(sortedSphMarkersD->posRadD),
                                             mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv),
                                             U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                             numAllMarkers, isErrorD);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after calcRho_kernel!\n");
    }

    calcPressureIISPH(otherFsiBodiesD->velMassRigid_fsiBodies_D, otherFsiBodiesD->accRigid_fsiBodies_D,
                      otherFsiMeshD->pos_fsi_fea_D, otherFsiMeshD->vel_fsi_fea_D, otherFsiMeshD->acc_fsi_fea_D,
                      _sumWij_inv, G_i, L_i, Color);

    //------------------------------------------------------------------------
    // thread per particle
    //  std::cout << "dT in ForceSPH after calcPressure: " << paramsH->dT << "\n";
    double CalcForcesClock = clock();

    thrust::fill(vel_IISPH_Sorted_D.begin(), vel_IISPH_Sorted_D.end(), mR3(0.0));
    thrust::fill(derivVelRhoD_Sorted_D.begin(), derivVelRhoD_Sorted_D.end(), mR4(0.0));
    thrust::fill(vel_XSPH_Sorted_D.begin(), vel_XSPH_Sorted_D.end(), mR3(0.0));
    thrust::device_vector<Real3> dr_shift(numAllMarkers);
    thrust::fill(dr_shift.begin(), dr_shift.end(), mR3(0.0));

    CalcForces<<<numBlocks, numThreads>>>(mR3CAST(vel_XSPH_Sorted_D), mR4CAST(derivVelRhoD_Sorted_D),
                                          mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD),
                                          mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i),
                                          R1CAST(L_i), mR3CAST(dr_shift), U1CAST(markersProximityD->cellStartD),
                                          U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);
    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in CalcForces!\n");
    }
    double calcforce = (clock() - CalcForcesClock) / (double)CLOCKS_PER_SEC;
    printf(" Force Computation: %f \n", calcforce);
    double UpdateClock = clock();

    thrust::device_vector<uint> SplitMe(numAllMarkers);
    thrust::device_vector<uint> MergeMe(numAllMarkers);
    thrust::device_vector<uint> myType(numAllMarkers);
    thrust::device_vector<Real3> helpers_normal(numHelperMarkers);
    //
    thrust::fill(SplitMe.begin(), SplitMe.end(), 0);
    thrust::fill(MergeMe.begin(), MergeMe.end(), 0);
    thrust::fill(myType.begin(), myType.end(), 0);
    thrust::fill(helpers_normal.begin(), helpers_normal.end(), mR3(0));
    hipDeviceSynchronize();

    Calc_HelperMarkers_normals<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(helpers_normal),
        I1CAST(myType), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
        U1CAST(markersProximityD->gridMarkerIndexD), numAllMarkers, isErrorD2);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in Calc_HelperMarkers_normals!\n");
    }
    double CalcSplitMerge = clock();

    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int limit = numObjectsH->numFluidMarkers + numObjectsH->numGhostMarkers + numObjectsH->numHelperMarkers;

    Calc_Splits_and_Merges<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
        mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(helpers_normal), R1CAST(_sumWij_inv), R1CAST(G_i), U1CAST(SplitMe),
        U1CAST(MergeMe), I1CAST(myType), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
        U1CAST(markersProximityD->gridMarkerIndexD), paramsH->HSML / 2.0, paramsH->HSML, numAllMarkers, limit,
        isErrorD2);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in Calc_Split_and_Merges!\n");
    }

    Split<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
                                     mR3CAST(sortedSphMarkersD->velMasD), mR3CAST(helpers_normal), R1CAST(_sumWij_inv),
                                     R1CAST(G_i), R1CAST(L_i), U1CAST(SplitMe), U1CAST(MergeMe), I1CAST(myType),
                                     U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
                                     U1CAST(markersProximityD->gridMarkerIndexD), paramsH->HSML / 2.0, paramsH->HSML,
                                     numAllMarkers, limit, isErrorD2);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(isErrorH, isErrorD2, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in Calc_Split_and_Merges!\n");
    }

    UpdateDensity<<<numBlocks, numThreads>>>(
        mR3CAST(vel_IISPH_Sorted_D), mR3CAST(vel_XSPH_Sorted_D), mR4CAST(sortedSphMarkersD->posRadD),
        mR4CAST(sortedSphMarkersD->rhoPresMuD), R1CAST(_sumWij_inv), U1CAST(markersProximityD->cellStartD),
        U1CAST(markersProximityD->cellEndD), numAllMarkers, isErrorD);

    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in CalcForces!\n");
    }
    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_XSPH_D, vel_XSPH_Sorted_D,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_IISPH_D, vel_IISPH_Sorted_D,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->posRadD, sortedSphMarkersD->posRadD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R3(sphMarkersD->velMasD, sortedSphMarkersD->velMasD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(sphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                        markersProximityD->gridMarkerIndexD);
    CopySortedToOriginal_NonInvasive_R4(fsiGeneralData->derivVelRhoD, derivVelRhoD_Sorted_D,
                                        markersProximityD->gridMarkerIndexD);
    printf(" Update information: %f \n", (clock() - UpdateClock) / (double)CLOCKS_PER_SEC);
    printf("----------------------------------------------\n");

    _sumWij_inv.clear();
    helpers_normal.clear();
    Color.clear();
    //
}

}  // namespace fsi
}  // namespace chrono
